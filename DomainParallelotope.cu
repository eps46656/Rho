#include "define.cuh"
#include "DomainParallelotope.cuh"

#define RHO__throw__local(desc) RHO__throw(DomainParallelotope, __func__, desc)

namespace rho {

DomainParallelotope::DomainParallelotope(Space* ref): DomainSole(ref) {}

#///////////////////////////////////////////////////////////////////////////////

bool DomainParallelotope::Refresh() const {
	if (!this->ref()->RefreshSelf()) { return false; }

	ContainFlag flag(0);
	ContainFlag flag_end(1);
	flag_end <<= this->dim_s();

	this->tod_matrix_.Resize(flag_end);

	for (; flag != flag_end; ++flag) {
		const Num* a_i(this->ref()->root_axis());
		Num* m_i(this->tod_matrix_[flag]);

		for (ContainFlag reader(1); reader != flag_end;
			 reader <<= 1, a_i += RHO__max_dim) {
			if (!(flag & reader)) {
				Copy(this->dim_r(), m_i, a_i);
				m_i += RHO__max_dim;
			}
		}

		this->tod_matrix_[flag].set_dim(
			(m_i - this->tod_matrix_[flag]) / RHO__max_dim, this->dim_r());
		Tod::TanMatrix(this->tod_matrix_[flag]);
	}

	return true;
}

#///////////////////////////////////////////////////////////////////////////////

bool DomainParallelotope::Contain_s(const Num* point) const {
	for (size_t i(0); i != this->dim_s(); ++i) {
		if (point[i].lt<-1>() || point[i].gt<1>()) { return false; }
	}

	return true;
}

#///////////////////////////////////////////////////////////////////////////////

bool DomainParallelotope::RayCastB(const Ray& ray) const {
	RayCastTemp rct;
	if (!this->RayCast_(ray, rct)) { return false; }

	if (rct.t[0].eq<0>()) { return rct.t[1].ne<0>() && rct.t[1].lt<1>(); }

	return rct.t[0].lt<1>();
}

RayCastData DomainParallelotope::RayCast(const Ray& ray) const {
	RayCastTemp rct;

	if (this->RayCast_(ray, rct)) {
		if (rct.t[0].ne<0>()) {
			auto rcd(New<RayCastDataCore_>());
			rcd->domain = this;
			rcd->t = rct.t[0];
			rcd->phase.set(false, rct.t[0] != rct.t[1]);
			rcd->contain_flag = rct.contain_flag[0];

			return RayCastData(rcd);
		}

		if (rct.t[1].ne<0>()) {
			auto rcd(New<RayCastDataCore_>());
			rcd->domain = this;
			rcd->t = rct.t[1];
			rcd->phase.set(true, false);
			rcd->contain_flag = rct.contain_flag[1];

			return RayCastData(rcd);
		}
	}

	return RayCastData();
}

void DomainParallelotope::RayCastForRender(RayCastDataPair& rcdp,
										   ComponentCollider* cmpt_collider,
										   const Ray& ray) const {
	RayCastTemp rct;
	if (!this->RayCast_(ray, rct)) { return; }

	if (rct.t[0].ne<0>()) {
		if (rcdp[1] < rct.t[0]) { return; }

		auto rcd(New<RayCastDataCore_>());
		rcd->cmpt_collider = cmpt_collider;
		rcd->domain = this;
		rcd->t = rct.t[0];
		rcd->phase.set(false, rct.t[0] != rct.t[1]);
		rcd->contain_flag = rct.contain_flag[0];

		if (rct.t[0] < rcdp[0]) {
			rcdp[1] = Move(rcdp[0]);
			rcdp[0] = rcd;
		} else {
			rcdp[1] = rcd;
			return;
		}
	}

	if (rct.t[0] != rct.t[1] && rct.t[1] < rcdp[1]) {
		auto rcd(New<RayCastDataCore_>());
		rcd->cmpt_collider = cmpt_collider;
		rcd->domain = this;
		rcd->t = rct.t[1];
		rcd->phase.set(true, false);
		rcd->contain_flag = rct.contain_flag[1];

		if (rct.t[1] < rcdp[0]) {
			rcdp[1] = Move(rcdp[0]);
			rcdp[0] = rcd;
		} else {
			rcdp[1] = rcd;
		}
	}
}

bool DomainParallelotope::RayCastFull(RayCastDataVector& dst,
									  const Ray& ray) const {
	RayCastTemp rct;

	if (this->RayCast_(ray, rct)) {
		if (rct.t[0].ne<0>()) {
			auto rcd(New<RayCastDataCore_>());
			rcd->domain = this;
			rcd->t = rct.t[0];
			rcd->phase.set(false, rct.t[0] != rct.t[1]);
			rcd->contain_flag = rct.contain_flag[0];

			dst.Push(rcd);
		}

		if (rct.t[0] != rct.t[1]) {
			auto rcd(New<RayCastDataCore_>());
			rcd->domain = this;
			rcd->t = rct.t[1];
			rcd->phase.set(true, false);
			rcd->contain_flag = rct.contain_flag[1];

			dst.Push(rcd);
		}
	}

	return false;
}

bool DomainParallelotope::RayCast_(const Ray& ray, RayCastTemp& rct) const {
	rct.t[0] = 0;
	rct.t[1] = RHO__inf;
	rct.contain_flag[0] = rct.contain_flag[1] = 0;

	Vec origin;
	Vec direct;

	this->ref()->MapPointFromRoot_rr(origin, ray.origin);
	this->ref()->MapVectorFromRoot_rr(direct, ray.direct);

#///////////////////////////////////////////////////////////////////////////////

	for (dim_t i(this->dim_s()); i != this->dim_r(); ++i) {
		if (direct[i].eq<0>()) {
			if (origin[i].eq<0>()) { continue; }
			return false;
		}

		Num t(-origin[i] / direct[i]);
		if (t < rct.t[0] || rct.t[1] < t) { return false; }
		rct.t[0] = rct.t[1] = t;
	}

#///////////////////////////////////////////////////////////////////////////////

	for (dim_t i(0); i != this->dim_s(); ++i) {
		if (direct[i].eq<0>()) {
			if (origin[i].lt<-1>() || origin[i].gt<1>()) { return false; }
			continue;
		}

		Num t[]{ (-1 - origin[i]) / direct[i], (1 - origin[i]) / direct[i] };

		if (t[1] < t[0]) { Swap(t[0], t[1]); }
		if (t[1] < rct.t[0] || rct.t[1] < t[0]) { return false; }

		if (rct.t[0] < t[0]) {
			rct.t[0] = t[0];
			rct.contain_flag[0] = ContainFlag(1) << i;
		} else if (rct.t[0] == t[0]) {
			rct.contain_flag[0] |= ContainFlag(1) << i;
		}

		if (t[1] < rct.t[1]) {
			rct.t[1] = t[1];
			rct.contain_flag[1] = ContainFlag(1) << i;
		} else if (t[1] == rct.t[1]) {
			rct.contain_flag[1] |= ContainFlag(1) << i;
		}
	}

	return true;
}

#///////////////////////////////////////////////////////////////////////////////

void DomainParallelotope::GetTodTan(Num* dst, const RayCastData& rcd,
									const Num* root_direct) const {
	dot(this->dim_r(), this->dim_r(), dst, root_direct,
		this->tod_matrix_[rcd.Get<RayCastDataCore_*>()->contain_flag]);
}

#////////////////////////////////////////////////

size_t DomainParallelotope::Complexity() const {
	return 10 * this->dim_s() + 5 * this->dim_cr();
}

}