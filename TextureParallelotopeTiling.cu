#include "hip/hip_runtime.h"
#include "define.cuh"
#include "TextureParallelotopeTiling.cuh"

#define RHO__throw__local(description)                                         \
	RHO__throw(TextureParallelotopeTiling, __func__, description);

namespace rho {

size_t TextureParallelotopeTiling::dim() const { return this->dim_; }

size_t TextureParallelotopeTiling::size(size_t index) const {
	RHO__debug_if(this->dim_ <= index) { RHO__throw__local("index error"); }
	return this->size_[index].first;
}

cntr::Vector<Texture::Data>& TextureParallelotopeTiling::data() {
	return this->data_;
}

const cntr::Vector<Texture::Data>& TextureParallelotopeTiling::data() const {
	return this->data_;
}

#////////////////////////////////////////////////

TextureParallelotopeTiling& TextureParallelotopeTiling::set_size(size_t* size) {
	for (size_t i(0); i != this->dim_; ++i) {
		this->size_[i].second = (this->size_[i].first = size[i]);
	}

	return *this;
}

TextureParallelotopeTiling& TextureParallelotopeTiling::set_size(size_t index,
																 size_t size) {
	RHO__debug_if(this->dim_ <= index) RHO__throw__local("index error");

	this->size_[index].second = (this->size_[index].first = size);

	return *this;
}

const Space* TextureParallelotopeTiling::ref() const { return this->ref_; }

void TextureParallelotopeTiling::set_ref(const Space* ref) { this->ref_ = ref; }

#////////////////////////////////////////////////

TextureParallelotopeTiling::TextureParallelotopeTiling(size_t dim): dim_(dim) {}

TextureParallelotopeTiling::~TextureParallelotopeTiling() {}

#////////////////////////////////////////////////

bool TextureParallelotopeTiling::Refresh() const {
	return this->ref_->Refresh();
}

#////////////////////////////////////////////////

Texture::Data TextureParallelotopeTiling::GetData(const Num* root_point,
												  const Num* tod_tan) const {
	Vec point;

	this->ref_->MapPointFromRoot_rs(point, root_point);

	size_t index(0);
	Num a;

	for (size_t i(0); i != this->dim_; ++i) {
		a = (point[i] + 1) / 2;
		a -= floor(a);

		(index *= this->size_[i].first) += size_t(this->size_[i].second * a);
	}

	return this->data_[index];
}

}