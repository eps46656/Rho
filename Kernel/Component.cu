#include "define.cuh"
#include "Kernel.cuh"

namespace rho {

bool Component::PriorityCmp::operator()(const Component* x,
										const Component* y) {
	return x->active_ && (!y->active_ || x->priority_ > y->priority_);
}

#///////////////////////////////////////////////////////////////////////////////

bool Component::active() const { return this->active_; }
bool Component::latest() const { return this->latest_; }

priority_t Component::priority() const { return this->priority_; }

Space* Component::root() const { return this->object_->root(); }
Object* Component::object() const { return this->object_; }

size_t Component::dim_r() const { return this->object_->root()->dim_r(); }

#///////////////////////////////////////////////////////////////////////////////

Component::Component(Type type, Object* object):
	type(type), active_(true), latest_(false), object_(object) {
	if (this->object_) { this->object_->AddCmpt_(this); }
}

Component::~Component() {
	if (this->object_) { this->object_->SubCmpt_(this); }
}

#///////////////////////////////////////////////////////////////////////////////

void Component::SetLatestFalse_() { this->latest_ = false; }

#///////////////////////////////////////////////////////////////////////////////

Component* Component::SetObject(Object* object) {
	if (this->object_ == object) { return this; }
	if (this->object_) { this->object_->SubCmpt_(this); }
	if (this->object_ = object) { this->object_->AddCmpt_(this); }
	return this;
}

void Component::Active(bool active) { this->active_ = active; }

#///////////////////////////////////////////////////////////////////////////////

bool operator<(const Component& x, const Component& y) { return &x < &y; }

}