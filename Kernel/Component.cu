#include"define.cuh"
#include"Kernel.cuh"

namespace rho {

bool Component::PriorityCmp::operator()(
	const Component* x, const Component* y) {

	return x->active_ && (!y->active_ || x->priority_ > y->priority_);
}

#////////////////////////////////////////////////

code_t Component::id()const { return this->id_; }

bool Component::active()const { return this->active_; }
bool Component::latest()const { return this->latest_; }

priority_t Component::priority()const { return this->priority_; }

Manager* Component::manager()const { return this->manager_; }
Space* Component::root()const { return this->root_; }
Object* Component::object()const { return this->object_; }

size_t Component::dim_r()const { return this->dim_r_; }

#////////////////////////////////////////////////

Component::Component(Type type, Object* object) :
	id_(Manager::get_code()),
	type(type),

	active_(true),
	latest_(false),

	manager_(object->manager_),
	root_(object->root_),
	object_(object),

	dim_r_(object->dim_r_) {

	this->manager_->AddComponent_(this);
	this->object_->AddComponent_(this);
}

Component::~Component() {}

#////////////////////////////////////////////////

void Component::SetLatestFalse_() { this->latest_ = false; }

#////////////////////////////////////////////////

void Component::Active(bool active) {
	if (active == this->active_)
		return;

	if (active) {
		this->object_->ActiveSelfAndAncestor();
		this->manager_->ActiveComponentTrue_(this);
		this->active_ = true;
	} else {
		this->manager_->ActiveComponentFalse_(this);
		this->active_ = false;
	}
}

void Component::Delete() {
	this->manager_->DeleteComponent_(this);
	this->object_->DeleteComponent_(this);
	this->~Component();
	Free(this);
}

#////////////////////////////////////////////////

bool operator<(const Component& x, const Component& y) {
	return x.id() < y.id();
}

}