#include"define.cuh"
#include"Kernel.cuh"

namespace rho {

bool Texture::Data::Check()const {
	return
		this->color[0].ge<0>() &&
		this->color[1].ge<0>() &&
		this->color[2].ge<0>() &&

		this->transmittance[0].ge<0>() &&
		this->transmittance[1].ge<0>() &&
		this->transmittance[2].ge<0>() &&

		this->reflectance[0].ge<0>() &&
		this->reflectance[1].ge<0>() &&
		this->reflectance[2].ge<0>() &&

		this->shininess[0].ge<0>() &&
		this->shininess[1].ge<0>() &&
		this->shininess[2].ge<0>();
}

}