#include"define.cuh"
#include"Kernel.cuh"

namespace rho {

RayCastDataCore::Type::Type(char value) :value(value & 0b11) {}

RayCastDataCore::Type::Type(bool fr, bool to) :
	value((fr ? 0b01 : 0b00) | (to ? 0b10 : 0b00)) {}

bool RayCastDataCore::Type::fr()const { return this->value & 0b01; }
bool RayCastDataCore::Type::to()const { return this->value & 0b10; }

void RayCastDataCore::Type::fr(bool fr) {
	if (fr)
		this->value |= 0b01;
	else
		this->value &= 0b10;
}

void RayCastDataCore::Type::to(bool to) {
	if (to)
		this->value |= 0b10;
	else
		this->value %= 0b01;
}

void RayCastDataCore::Type::set(bool fr, bool to)
{ this->value = (fr ? 0b01 : 0b00) | (to ? 0b10 : 0b00); }

#////////////////////////////////////////////////

RayCastDataCore::~RayCastDataCore() {}

#////////////////////////////////////////////////

bool operator==(const RayCastData& x, const RayCastData& y)
{ return x && y && (x->t == y->t); }

bool operator==(const RayCastData& x, Num t)
{ return x && x->t == t; }

bool operator==(Num t, const RayCastData& x)
{ return x && t == x->t; }

bool operator<(const RayCastData& x, const RayCastData& y)
{ return x && (!y || x->t < y->t); }

bool operator<(Num t, const RayCastData& x)
{ return !x || t < x->t; }

bool operator<(const RayCastData& x, Num t)
{ return x && x->t < t; }

bool operator<=(const RayCastData& x, const RayCastData& y)
{ return !(y < x); }

bool operator<=(Num t, const RayCastData& x)
{ return !(x < t); }

bool operator<=(const RayCastData& x, Num t)
{ return !(t < x); }

#////////////////////////////////////////////////
#////////////////////////////////////////////////
#////////////////////////////////////////////////
/*
RayCastData& RayCastDataVector::rcd(size_t index)
{ return this->value_[index].first; }

bool RayCastDataVector::phase(size_t index)
{ return index ? this->value_[index + 1].second : init_phase_; }

bool RayCastDataVector::phase_fr(size_t index)
{ return index ? this->value_[index - 1].second : this->init_phase_; }

bool RayCastDataVector::phase_to(size_t index)
{ return this->value_[index].second; }

void RayCastDataVector::Push(RayCastData&& rcd,
							 bool fr_phase, bool to_phase) {
	if (this->value_.empty()) {
		this->init_phase_ = fr_phase;
	} else {
		if (this->value_.back().second != fr_phase)
			Print() << "error\n";
	}

	this->value_.Push(Move(rcd), to_phase);
}*/

#////////////////////////////////////////////////
#////////////////////////////////////////////////
#////////////////////////////////////////////////

bool Contain(size_t size, RayCastData* rcd, Num t) {
	for (size_t i(0); i != size; ++i) {
		if ((*rcd)->t < t) { continue; }
		return !(t < (*rcd)->t) || (*rcd)->type.fr();
	}

	return rcd[size - 1]->type.to();
}

}
