#include "define.cuh"
#include "Kernel.cuh"

namespace rho {

const cntr::BidirectionalNode* Manager::instance() { return instance_(); }

cntr::BidirectionalNode* Manager::instance_() {
	static cntr::BidirectionalNode* r(nullptr);
	return r ? r : (r = New<cntr::BidirectionalNode>());
}

size_t Manager::get_code() {
	static size_t code(0);
	return code += 1;
}

Map_t<code_t, void*>& Manager::id_ptr_() {
	static Map_t<code_t, void*>* r(nullptr);
	return *(r ? r : (r = New<Map_t<code_t, void*>>()));
}

#///////////////////////////////////////////////////////////////////////////////

const RBT<Space*>& Manager::space() const { return this->space_; }

const RBT<Object*>& Manager::object() const { return this->object_; }

const ComponentContainer& Manager::cmpt() const { return this->cmpt_; }

const RBT<Object*>& Manager::active_object() const {
	return this->active_object_;
}

const ComponentContainer& Manager::active_cmpt() const {
	return this->active_cmpt_;
}

#///////////////////////////////////////////////////////////////////////////////

Texture* Manager::default_texture() const { return this->default_texture_; }

Material* Manager::default_material() const { return this->default_material_; }

Material* Manager::void_material() const { return this->void_material_; }

#///////////////////////////////////////////////////////////////////////////////

const cntr::Vector<Component*>& Manager::priority_vector() const {
	return this->priority_vector_;
}

bool Manager::priority_vector(const cntr::Vector<Component*>& priority_vector) {
	size_t size(this->priority_vector_.size());

	if (size != priority_vector.size()) { return false; }

	for (size_t i(0); i != priority_vector.size(); ++i) {
		if (BinarySearch(this->priority_vector_, priority_vector.size(),
						 priority_vector[i]) == priority_vector.size()) {
			return false;
		}
	}

	for (size_t i(0); i != size; ++i) {
		(this->priority_vector_[i] = priority_vector[i])->priority_ = i;
	}

	return true;
}

#///////////////////////////////////////////////////////////////////////////////

Manager::Manager(Space* root):
	root_(root), void_material_(New<Material>()),
	default_material_(New<Material>()) {
	//	instance_()->PushFront(this);

	this->void_material_->refraction_index = 1;
	this->void_material_->transmittance[0] = 0;
	this->void_material_->transmittance[1] = 0;
	this->void_material_->transmittance[2] = 0;

	this->default_material_->refraction_index = 1;
	this->default_material_->transmittance[0] = 0;
	this->default_material_->transmittance[1] = 0;
	this->default_material_->transmittance[2] = 0;

	TextureSolid* a(New<TextureSolid>());
	a->data.color[0] = 0;
	a->data.color[1] = 0;
	a->data.color[2] = 0;
	a->data.transmittance[0] = 0;
	a->data.transmittance[1] = 0;
	a->data.transmittance[2] = 0;
	a->data.reflectance[0] = 0;
	a->data.reflectance[1] = 0;
	a->data.reflectance[2] = 0;
	a->data.shininess[0] = 0;
	a->data.shininess[1] = 0;
	a->data.shininess[2] = 0;

	this->default_texture_ = a;

	//this->space_.Insert(root);
}

#///////////////////////////////////////////////////////////////////////////////

void Manager::Refresh() const {}

#///////////////////////////////////////////////////////////////////////////////

void Manager::AddSpace_(Space* space) { this->space_.Insert(space); }

void Manager::AddObject_(Object* object) {
	this->object_.Insert(object);
	this->active_object_.Insert(object);
}

void Manager::AddComponent_(Component* cmpt) {
	this->cmpt_.Insert(cmpt);
	this->active_cmpt_.Insert(cmpt);

	cmpt->priority_ = this->priority_vector_.size();
	this->priority_vector_.Push(cmpt);

	this->ActiveComponentTrue_(cmpt);
}

void Manager::RegisterCamera_(Camera* camera) { this->camera_.Push(camera); }

#///////////////////////////////////////////////////////////////////////////////

void Manager::ActiveObjectTrue_(Object* object) {
	this->active_object_.Insert(object);
}

void Manager::ActiveComponentTrue_(Component* cmpt) {
	this->active_cmpt_.Insert(cmpt);

	if (cmpt->type == Component::Type::collider) {
		this->active_sorted_cmpt_collider_.Push(
			static_cast<ComponentCollider*>(cmpt));

		Sort(this->active_sorted_cmpt_collider_.begin(),
			 this->active_sorted_cmpt_collider_.end(),
			 Component::PriorityCmp());
	}
}

void Manager::ActiveObjectFalse_(Object* object) {
	this->active_object_.FindErase(object);
}

void Manager::ActiveComponentFalse_(Component* cmpt) {
	this->active_cmpt_.FindErase(cmpt);

	if (cmpt->type == Component::Type::collider) {
		this->active_sorted_cmpt_collider_.Erase(
			LinearSearch(this->active_sorted_cmpt_collider_.begin(),
						 this->active_sorted_cmpt_collider_.end(),
						 static_cast<ComponentCollider*>(cmpt)));
	}
}

#///////////////////////////////////////////////////////////////////////////////

void Manager::DeleteSpace_(Space* space) {
	this->space_.Erase(this->space_.Find(space));
}

void Manager::DeleteObject_(Object* object) {
	this->object_.Erase(this->object_.Find(object));
	this->active_object_.Erase(this->active_object_.Find(object));
}

void Manager::DeleteComponent_(Component* cmpt) {
	this->cmpt_.FindErase(cmpt);
	this->active_cmpt_.FindErase(cmpt);

	if (cmpt->type == Component::Type::collider && cmpt->active()) {
		this->active_sorted_cmpt_collider_.Erase(
			LinearSearch(this->active_sorted_cmpt_collider_.begin(),
						 this->active_sorted_cmpt_collider_.end(),
						 static_cast<ComponentCollider*>(cmpt)));
	}
}

void Manager::DeleteCamera_(Camera* camera) {
	this->camera_.Erase(
		LinearSearch(this->camera_.begin(), this->camera_.end(), camera));
}

#///////////////////////////////////////////////////////////////////////////////

ComponentCollider* Manager::GetComponentCollider(const Num* point) const {
	return this->GetComponentCollider(point,
									  this->active_sorted_cmpt_collider_);
}

ComponentCollider* Manager::GetComponentCollider(
	const Num* point,
	const cntr::Vector<ComponentCollider*>& cmpt_collider) const {
	auto iter(this->active_sorted_cmpt_collider_.begin());

	for (auto end(this->active_sorted_cmpt_collider_.end()); iter != end;
		 ++iter) {
		if ((*iter)->domain()->Contain(point)) return *iter;
	}

	return nullptr;
}

cntr::Vector<ComponentCollider*>
Manager::GetComponentCollider_Full(const Num* point) const {
	return this->GetComponentCollider_Full(point,
										   this->active_sorted_cmpt_collider_);
}

cntr::Vector<ComponentCollider*> Manager::GetComponentCollider_Full(
	const Num* point,
	const cntr::Vector<ComponentCollider*>& cmpt_collider) const {
	cntr::Vector<ComponentCollider*> r;
	r.Reserve(this->active_sorted_cmpt_collider_.size());

	for (size_t i(0); i != this->active_sorted_cmpt_collider_.size(); ++i) {
		if (this->active_sorted_cmpt_collider_[i]->domain()->Contain(point))
			r.Push(this->active_sorted_cmpt_collider_[i]);
	}

	return r;
}

}
