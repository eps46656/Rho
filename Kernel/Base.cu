#include "define.cuh"
#include "Kernel.cuh"

namespace rho {

RHO__cuda bool C_next(size_t x, size_t y, size_t* data) {
	if (data[0] == x - y) { return false; }

	size_t i(y);
	for (--i; data[i] == x - (y - i); --i)
		;
	++data[i];
	for (++i; i != y; ++i) { data[i] = data[i - 1] + 1; }

	return true;
}

bool Complement(dim_t col_dim, dim_t row_dim, Num* data) {
	if (row_dim < col_dim) { return false; }
	if (col_dim == row_dim) { return true; }

	dim_t co(row_dim - col_dim - 1);

	if (co) {
		Num m[(RHO__max_dim - 1) * (RHO__max_dim - 1)];
		Num* v(m + (RHO__max_dim - 2) * (RHO__max_dim - 1));
		size_t c[RHO__max_dim - 1];

		for (dim_t i(0); i != co; ++i) { c[i] = i; }

		for (dim_t k(0); k != co; ++k) {
			c[co - k] = RHO__max_dim;

			for (;;) {
				for (size_t i(0); i != col_dim; ++i) {
					Num* m_i(m + RHO__max_dim * i);
					size_t* c_i(c);

					for (size_t j(0); j != row_dim; ++j) {
						if (j == *c_i) {
							++c_i;
						} else {
							*m_i = data[RHO__max_dim * i + j];
							++m_i;
						}
					}
				}

				cross(v, m, col_dim + 1);

				if (!is_zero(row_dim, v)) { break; }

				if (c[0] == col_dim + 1) { return false; }

				size_t i(co - 1);
				while (c[i] == col_dim + 1 + i) { --i; }

				++c[i];
				for (++i; i != co; ++i) { c[i] = c[i - 1] + 1; }
			}

			Num* v_i(v);
			size_t* c_i(c);

			for (size_t i(0); i != row_dim; ++i) {
				if (i == *c_i) {
					data[RHO__max_dim * col_dim + i] = 0;
					++c_i;
				} else {
					data[RHO__max_dim * col_dim + i] = *v_i;
					++v_i;
				}
			}

			++col_dim;
		}
	}

	cross(data + RHO__max_dim * col_dim, data, row_dim);

	return !is_zero(row_dim, data + RHO__max_dim * col_dim);
}

bool Complement(Matrix& matrix) {
	if (Complement(matrix.col_dim(), matrix.row_dim(), matrix)) {
		matrix.set_col_dim(matrix.row_dim());
		return true;
	}

	return false;
}

}