#include "hip/hip_runtime.h"
#include "define.cuh"
#include "Kernel.cuh"

#define RHO__throw__local(desc) RHO__throw(Space, __func__, desc)

#define RHO_ParentCheck                                                        \
	RHO__debug_if(!this->parent_) RHO__throw__local("parent error");

#define RHO__BranchCheck(branch)                                               \
	RHO__debug_if(this->root_ != branch->root_)                                \
		RHO__throw__local("branch error");

#define RHO__SetOriginAxisCheck                                                \
	RHO__debug_if(!this->parent_) RHO__throw__local(                           \
		("Space a root space, can not be set origin or axis."));

#define RHO__dim_check(x, y)                                                   \
	RHO__debug_if((x) != (y)) RHO__throw__local("dim error");

#define RHO__dim_check2(x1, y1, x2, y2)                                        \
	RHO__debug_if((x1) != (y1) || (x2) != (y2)) RHO__throw__local("dim "       \
																  "error");

// #define dim_s_ dim_

namespace rho {

bool Space::latest() const {
	for (const Space* s(this); s; s = s->parent_) {
		if (!s->latest_) { return false; }
	}

	return true;
}

bool Space::is_root() const { return !this->parent_; }

#///////////////////////////////////////////////////////////////////////////////

Space* Space::root() { return this->root_; }
const Space* Space::root() const { return this->root_; }

Space* Space::parent() { return this->parent_; }
const Space* Space::parent() const { return this->parent_; }

const cntr::Vector<Space*>& Space::child() { return this->child_; }
const cntr::Vector<const Space*>& Space::child() const {
	return this->const_child_;
}

size_t Space::depth() const { return this->depth_; }

#///////////////////////////////////////////////////////////////////////////////

RHO__cuda const Num* Space::origin() const { return this->origin_; }
RHO__cuda const Num* Space::axis() const { return this->axis_; }

RHO__cuda const Num* Space::root_origin() const { return this->root_origin_; }
RHO__cuda const Num* Space::root_axis() const { return this->root_axis_; }

RHO__cuda const Num* Space::i_origin() const { return this->i_origin_; }
RHO__cuda const Num* Space::i_axis() const { return this->i_axis_; }

RHO__cuda const Num* Space::i_root_origin() const {
	return this->i_root_origin_;
}
RHO__cuda const Num* Space::i_root_axis() const { return this->i_root_axis_; }

#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////

dim_t Space::dim_s() const { return this->dim_s_; }
dim_t Space::dim_p() const { return this->dim_p_; }
dim_t Space::dim_r() const { return this->dim_r_; }
dim_t Space::dim_cp() const { return this->dim_cp_; }
dim_t Space::dim_cr() const { return this->dim_cr_; }

#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////

Space::Space(dim_t dim):
	latest_(false), parent_(nullptr), root_(this), depth_(0), dim_s_(dim),
	dim_p_(dim), dim_r_(dim), dim_cp_(0), dim_cr_(0) {
	RHO__debug_if(RHO__max_dim < dim) RHO__throw__local("dim error");
}

Space::Space(dim_t dim, Space* parent):
	latest_(false), parent_(parent), root_(parent->root_),
	depth_(parent->depth_ + 1), dim_s_(dim), dim_p_(parent->dim_s_),
	dim_r_(parent->dim_r_), dim_cp_(this->dim_p_ - this->dim_s_),
	dim_cr_(this->dim_r_ - this->dim_s_) {
	RHO__debug_if(RHO__max_dim < dim) { RHO__throw__local("dim error"); }
	this->parent_->AddChild_(this);
}

#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////

#define RHO__args Num *dst, const Num *src

#define RHO__F(x, y)                                                           \
	RHO_ParentCheck;                                                           \
	dot(this->dim_##x##_, this->dim_##y##_, dst, src, this->axis_);            \
	for (dim_t i(0); i != this->dim_##y##_; ++i) { dst[i] += this->origin_[i]; }

void Space::MapPointToParent_sp(RHO__args) const { RHO__F(s, p) }
void Space::MapPointToParent_sr(RHO__args) const { RHO__F(s, r) }
void Space::MapPointToParent_rp(RHO__args) const { RHO__F(r, p) }
void Space::MapPointToParent_rr(RHO__args) const { RHO__F(r, r) }

#undef RHO__F

#define RHO__F(x, y)                                                           \
	RHO_ParentCheck;                                                           \
	dot(this->dim_##x##_, dim_##y##_, dst, src, this->axis_);

void Space::MapVectorToParent_sp(RHO__args) const { RHO__F(s, p) }
void Space::MapVectorToParent_sr(RHO__args) const { RHO__F(s, r) }
void Space::MapVectorToParent_rp(RHO__args) const { RHO__F(r, p) }
void Space::MapVectorToParent_rr(RHO__args) const { RHO__F(r, r) }

#undef RHO__F

#///////////////////////////////////////////////////////////////////////////////

#define RHO__F(x, y)                                                           \
	RHO_ParentCheck;                                                           \
	dot(this->dim_##x##_, this->dim_##y##_, dst, src, this->i_axis_);          \
	for (dim_t i(0); i != this->dim_##y##_; ++i) {                             \
		dst[i] -= this->i_origin_[i];                                          \
	}

void Space::MapPointFromParent_ps(RHO__args) const { RHO__F(p, s) }
void Space::MapPointFromParent_rs(RHO__args) const { RHO__F(r, s) }
void Space::MapPointFromParent_pr(RHO__args) const { RHO__F(p, r) }
void Space::MapPointFromParent_rr(RHO__args) const { RHO__F(r, r) }

#undef RHO__F

#define RHO__F(x, y)                                                           \
	RHO_ParentCheck;                                                           \
	dot(this->dim_##x##_, this->dim_##y##_, dst, src, this->i_axis_);

void Space::MapVectorFromParent_ps(RHO__args) const { RHO__F(p, s) }
void Space::MapVectorFromParent_rs(RHO__args) const { RHO__F(r, s) }
void Space::MapVectorFromParent_pr(RHO__args) const { RHO__F(p, r) }
void Space::MapVectorFromParent_rr(RHO__args) const { RHO__F(r, r) }

#undef RHO__F

#///////////////////////////////////////////////////////////////////////////////

#define RHO__F(x, y)                                                           \
	dot(this->dim_##x##_, dim_##y##_, dst, src, this->root_axis_);             \
	for (size_t i(0); i != this->dim_##y##_; ++i)                              \
		dst[i] += this->root_origin_[i];

void Space::MapPointToRoot_sr(RHO__args) const { RHO__F(s, r) }
void Space::MapPointToRoot_rr(RHO__args) const { RHO__F(r, r) }

#undef RHO__F

#define RHO__F(x, y)                                                           \
	dot(this->dim_##x##_, this->dim_##y##_, dst, src, this->root_axis_);

void Space::MapVectorToRoot_sr(RHO__args) const { RHO__F(s, r) }
void Space::MapVectorToRoot_rr(RHO__args) const { RHO__F(r, r) }

#undef RHO__F

#///////////////////////////////////////////////////////////////////////////////

#define RHO__F(x, y)                                                           \
	dot(this->dim_##x##_, this->dim_##y##_, dst, src, this->i_root_axis_);     \
	for (size_t i(0); i != this->dim_##y##_; ++i)                              \
		dst[i] -= this->i_root_origin_[i];

void Space::MapPointFromRoot_rs(RHO__args) const { RHO__F(r, s) }
void Space::MapPointFromRoot_rr(RHO__args) const { RHO__F(r, r) }

#undef RHO__F

#define RHO__F(x, y)                                                           \
	dot(this->dim_##x##_, this->dim_##y##_, dst, src, this->i_root_axis_);

void Space::MapVectorFromRoot_rs(RHO__args) const {
	dot(this->dim_r_, this->dim_s_, dst, src, this->i_root_axis_);
}
void Space::MapVectorFromRoot_rr(RHO__args) const {
	dot(this->dim_r_, this->dim_r_, dst, src, this->i_root_axis_);
}

#undef RHO__F
#undef RHO__args

#///////////////////////////////////////////////////////////////////////////////

#define RHO__args Num *dst, const Num *src, const Space *branch

#define RHO__F(x, y)                                                           \
	RHO__debug_if(this->root_ != branch->root_)                                \
		RHO__throw__local("root error");                                       \
	Vec temp;                                                                  \
	this->MapPointToRoot_##x##r(temp, src);                                    \
	branch->MapPointFromRoot_r##y(dst, temp);

void Space::MapPointToBranch_sb(RHO__args) const { RHO__F(s, s) }
void Space::MapPointToBranch_sr(RHO__args) const { RHO__F(s, r) }
void Space::MapPointToBranch_rb(RHO__args) const { RHO__F(r, s) }
void Space::MapPointToBranch_rr(RHO__args) const { RHO__F(r, r) }

#undef RHO__F

#define RHO__F(x, y)                                                           \
	RHO__debug_if(this->root_ != branch->root_)                                \
		RHO__throw__local("root error");                                       \
	Vec temp;                                                                  \
	this->MapVectorToRoot_##x##r(temp, src);                                   \
	branch->MapVectorFromRoot_r##y(dst, temp);

void Space::MapVectorToBranch_sb(RHO__args) const { RHO__F(s, s) }
void Space::MapVectorToBranch_sr(RHO__args) const { RHO__F(s, r) }
void Space::MapVectorToBranch_rb(RHO__args) const { RHO__F(r, s) }
void Space::MapVectorToBranch_rr(RHO__args) const { RHO__F(r, r) }

#undef RHO__F

#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////

bool Space::IncludePointFromRoot_r(const Num* src) const {
	if (!this->dim_cr_) { return true; }

	Vec temp;

	dot(this->dim_r_, this->dim_cr_, temp, src,
		this->i_root_axis_ + this->dim_s_);

	return Equal(this->dim_cr_, temp, this->i_root_origin_ + this->dim_s_);
}

bool Space::IncludeVectorFromRoot_r(const Num* src) const {
	if (!this->dim_cr_) { return true; }

	Vec temp;

	dot(this->dim_r_, this->dim_cr_, temp, src,
		this->i_root_axis_ + this->dim_s_);

	for (size_t i(0); i != this->dim_cr_; ++i) {
		if (temp[i].ne<0>()) { return false; }
	}

	return true;
}

#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////

bool Space::Overlap(const Space* branch) const {
	RHO__debug_if(this->root_ != branch->root_) RHO__throw__local("root error");
	return this->dim_s_ == branch->dim_s_ && this->Include_(branch);
}

bool Space::Include(const Space* branch) const {
	RHO__debug_if(this->root_ != branch->root_) RHO__throw__local("root error");
	return branch->dim_s_ <= this->dim_s_ && this->Include_(branch);
}

bool Space::Include_(const Space* branch) const {
	if (!this->IncludePointFromRoot_r(branch->root_origin_)) { return false; }

	for (dim_t i(0); i != branch->dim_s_; ++i) {
		if (!this->IncludeVectorFromRoot_r(branch->root_axis_ +
										   RHO__max_dim * i)) {
			return false;
		}
	}

	return true;
}

#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////

Space* Space::SetParent(Space* parent) {
	if (this->parent_ == parent) { return this; }

	RHO__debug_if(this->dim_p_ != parent->dim_s_)
		RHO__throw__local("dim error");

	if (parent->depth_ <= this->depth_) {
		for (Space* s(parent); s; s = s->parent_) {
			if (this == s) { RHO__throw__local("rank error"); }
		}
	}

	cntr::List<Space*> l;
	l.PushBack(this);

	this->parent_->SubChild_(this);
	this->parent_ = parent;

	while (l.size()) {
		Space* s(l.back());
		l.PopBack();

		s->root_ = this->root_;
		s->dim_r_ = this->dim_r_;
		s->dim_cr_ = s->dim_s_ - this->dim_r_;

		for (size_t i(0); i != s->child_.size(); ++i) {
			l.PushBack(s->child_[i]);
		}
	}

	return this;
}

#///////////////////////////////////////////////////////////////////////////////

Space* Space::SetOrigin(const Num* origin) {
	Vector::Copy(this->origin_, origin);
	this->latest_ = false;
	return this;
}

Space* Space::SetAxis(const Num* axis) {
	Matrix::Copy(this->axis_, axis);
	this->latest_ = false;
	return this;
}

Space* Space::SetOrigin(const Vector& origin) {
	RHO__dim_check(this->dim_p_, origin.dim());
	return this->SetOrigin(&origin[0]);
}

Space* Space::SetAxis(const Matrix& axis) {
	RHO__dim_check(this->dim_s_, axis.col_dim());
	RHO__dim_check(this->dim_p_, axis.row_dim());
	return this->SetAxis(&axis[0]);
}

Space* Space::EnumSetOrigin(const cntr::EnumerateVector<Num>& origin) {
	RHO__debug_if(origin.size() != this->dim_p_) {
		RHO__throw__local("dim error");
	}

	return this->SetOrigin(origin.data());
}

Space* Space::EnumSetAxis(const cntr::EnumerateVector<Num>& axis) {
	RHO__debug_if(axis.size() != this->dim_s_ * this->dim_p_)
		RHO__throw__local("dim error");

	for (size_t i(0); i != axis.size(); ++i) {
		size_t a(i / this->dim_p_);
		this->axis_[RHO__max_dim * a + i - a * this->dim_p_] = axis[i];
	}

	return this;
}

#///////////////////////////////////////////////////////////////////////////////

void Space::Refresh() const {
	RHO__throw__local("do not call this func\n");

	/*cntr::Vector<const Space*> stack;

	if (!this->latest_) {
		stack.Reserve(this->const_child_.size() < this->depth_ ?
					  this->depth_ : this->const_child_.size());
		stack.Push(this);

		for (auto i(this->parent_); i && !i->latest_; i = i->parent_)
			stack.Push(i);

		for (size_t i(stack.size() - 1); i; --i)
			stack[i]->RefreshMain_();

		this->RefreshMain_();
	}

	stack = this->const_child_;

	while (stack.size()) {
		const Space* space(stack.back());
		stack.Pop();
		stack.Insert(stack.end(),
					 space->const_child_.begin(),
					 space->const_child_.end());
		space->RefreshMain_();
	}*/

	this->RefreshSelf();
	this->RefreshDescendant_();
}

bool Space::RefreshSelf() const {
	cntr::Vector<const Space*> s(this->depth_ + 1);

	const Space* a(this);
	size_t j(s.size());

	for (size_t i(s.size()); i; --i, a = a->parent_) {
		if (!((s[i - 1] = a)->latest_)) { j = i - 1; }
	}

	for (; j != s.size(); ++j) {
		if (!s[j]->RefreshMain_()) { return false; }
	}

	return true;
}

void Space::RefreshDescendant_() const {
	auto iter(this->child_.begin());

	for (auto end(this->child_.end()); iter != end; ++iter) {
		(*iter)->RefreshMain_();
		(*iter)->RefreshDescendant_();
	}
}

bool Space::RefreshMain_() const {
	this->latest_ = true;

	if (this->parent_) {
		for (dim_t i(this->dim_p_); i != this->dim_r_; ++i) {
			this->origin_[i] = 0;
		}

		for (dim_t i(0); i != this->dim_s_; ++i) {
			for (dim_t j(this->dim_p_); j != this->dim_r_; ++j) {
				this->axis_[RHO__max_dim * i + j] = 0;
			}
		}

		dot(this->dim_p_, this->dim_r_, this->root_origin_, this->origin_,
			this->parent_->root_axis_);

#pragma unroll
		for (dim_t i(0); i != RHO__max_dim; ++i)
			this->root_origin_[i] += this->parent_->root_origin_[i];

		dot(this->dim_s_, this->dim_p_, this->dim_r_, this->root_axis_,
			this->axis_, this->parent_->root_axis_);

		if (!Complement(this->dim_s_, this->dim_r_, this->root_axis_))
			return false;

		inverse(this->dim_r_, this->i_root_axis_, this->root_axis_);

		dot(this->dim_r_, this->dim_r_, this->i_root_origin_,
			this->root_origin_, this->i_root_axis_);

		dot(this->dim_cr_, this->dim_r_, this->dim_r_,
			this->axis_ + RHO__max_dim * this->dim_s_,
			this->root_axis_ + RHO__max_dim * this->dim_s_,
			this->parent_->i_root_axis_);
	} else {
		Fill<RHO__max_dim>(this->origin_, 0);
		Matrix::identity(this->axis_);

		Fill<RHO__max_dim>(this->root_origin_, 0);
		Matrix::identity(this->root_axis_);

		Fill<RHO__max_dim>(this->i_origin_, 0);
		Matrix::identity(this->i_axis_);

		Fill<RHO__max_dim>(this->i_root_origin_, 0);
		Matrix::identity(this->i_root_axis_);
	}

	return true;
}

#///////////////////////////////////////////////////////////////////////////////

void Space::SetLatestFalse_() {
	this->latest_ = false;

	auto iter(this->child_.begin());

	for (auto end(this->child_.end()); iter != end; ++iter)
		(*iter)->SetLatestFalse_();
}

#///////////////////////////////////////////////////////////////////////////////

void Space::Delete() {
	this->parent_->child_.FindDelete(this);

	cntr::Vector<Space*> stack(this->child_);

	while (stack.size()) {
		Space* space(stack.back());
		stack.Pop();
		stack.Insert(stack.end(), space->child_.begin(), space->child_.end());

		rho::Delete(space);
	}
}
/*
void Space::Delete_() {
	instance_().Erase(instance_().Find(this));

	auto iter(this->child_.begin());

	for (auto end(this->child_.end()); iter != end; ++iter)
		(*iter)->Delete();

	this->manager_->DeleteSpace_(this);
	rho::Delete(this);
}*/

#///////////////////////////////////////////////////////////////////////////////

void Space::AddChild_(Space* space) {
	this->child_.Push(space);
	this->const_child_.Push(space);
}

void Space::SubChild_(Space* space) {
	this->child_.FindDelete(space);
	this->const_child_.FindDelete(space);
}

}
