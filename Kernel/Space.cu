#include "hip/hip_runtime.h"
#include "define.cuh"
#include "Kernel.cuh"

#define RHO__throw__local(desc) RHO__throw(Space, __func__, desc)

#define RHO_ParentCheck                                                        \
	RHO__debug_if(!this->parent_) { RHO__throw__local("parent error"); }

#define RHO__BranchCheck(branch)                                               \
	RHO__debug_if(this->root_ != branch->root_) {                              \
		RHO__throw__local("branch error");                                     \
	}

#define RHO__SetOriginAxisCheck                                                \
	RHO__debug_if(!this->parent_) {                                            \
		RHO__throw__local(("Space a root space, can not "                      \
						   "be set origin or axis."));                         \
	}

#define RHO__dim_check(x, y)                                                   \
	RHO__debug_if((x) != (y)) { RHO__throw__local("dim error"); }

#define RHO__dim_check2(x1, y1, x2, y2)                                        \
	RHO__debug_if((x1) != (y1) || (x2) != (y2)) {                              \
		RHO__throw__local("dim error");                                        \
	}

namespace rho {

dim_t Space::dim() const { return this->dim_; }
dim_t Space::parent_dim() const { return this->parent_dim_; }
dim_t Space::root_dim() const { return this->root_dim_; }
dim_t Space::parent_codim() const { return this->parent_codim_; }
dim_t Space::root_codim() const { return this->root_codim_; }

#///////////////////////////////////////////////////////////////////////////////

const Space* Space::root() const { return this->root_; }
const Space* Space::parent() const { return this->parent_; }

size_t Space::depth() const { return this->depth_; }

bool Space::is_root() const { return !this->parent_; }

#///////////////////////////////////////////////////////////////////////////////

const Num* Space::origin() const { return this->origin_; }
const Num* Space::axis() const { return this->axis_; }

const Num* Space::root_origin() const { return this->root_origin_; }
const Num* Space::root_axis() const { return this->root_axis_; }

const Num* Space::i_origin() const { return this->i_origin_; }
const Num* Space::i_axis() const { return this->i_axis_; }

const Num* Space::i_root_origin() const { return this->i_root_origin_; }
const Num* Space::i_root_axis() const { return this->i_root_axis_; }

#///////////////////////////////////////////////////////////////////////////////

bool Space::latest_arch() const {
	for (const Space* s(this); s; s = s->parent_) {
		if (!s->latest_arch_) {
			return this->latest_arch_ = this->latest_ = false;
		}
	}

	return true;
}

bool Space::latest() const {
	for (const Space* s(this); s; s = s->parent_) {
		if (!s->latest_) { return this->latest_ = false; }
	}

	return true;
}

#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////

Space::Space(dim_t dim, const Space* parent):
	dim_(dim), parent_(parent), parent_dim_(parent ? parent->dim_ : dim),
	parent_codim_(this->parent_dim_ - this->dim_), latest_arch_(false),
	latest_(false) {}

#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////
#/////////////////////////////////////////////////////////////////////

#define RHO__args Num *dst, const Num *src

#define RHO__F(x, y)                                                           \
	RHO_ParentCheck;                                                           \
	dot(this->x##dim_, this->y##dim_, dst, src, this->axis_);                  \
	for (dim_t i(0); i != this->y##dim_; ++i) { dst[i] += this->origin_[i]; }

void Space::MapPointToParent_sp(RHO__args) const { RHO__F(, parent_); }
void Space::MapPointToParent_sr(RHO__args) const { RHO__F(, root_); }
void Space::MapPointToParent_rp(RHO__args) const { RHO__F(root_, parent_); }
void Space::MapPointToParent_rr(RHO__args) const { RHO__F(root_, root_); }

#undef RHO__F

#define RHO__F(x, y)                                                           \
	RHO_ParentCheck;                                                           \
	dot(this->x##dim_, this->y##dim_, dst, src, this->axis_);

void Space::MapVectorToParent_sp(RHO__args) const { RHO__F(, parent_); }
void Space::MapVectorToParent_sr(RHO__args) const { RHO__F(, root_); }
void Space::MapVectorToParent_rp(RHO__args) const { RHO__F(root_, parent_); }
void Space::MapVectorToParent_rr(RHO__args) const { RHO__F(root_, root_); }

#undef RHO__F

#///////////////////////////////////////////////////////////////////////////////

#define RHO__F(x, y)                                                           \
	RHO_ParentCheck;                                                           \
	dot(this->x##dim_, this->y##dim_, dst, src, this->i_axis_);                \
	for (dim_t i(0); i != this->y##dim_; ++i) { dst[i] -= this->i_origin_[i]; }

void Space::MapPointFromParent_ps(RHO__args) const { RHO__F(parent_, ); }
void Space::MapPointFromParent_rs(RHO__args) const { RHO__F(root_, ); }
void Space::MapPointFromParent_pr(RHO__args) const { RHO__F(parent_, root_); }
void Space::MapPointFromParent_rr(RHO__args) const { RHO__F(root_, root_); }

#undef RHO__F

#define RHO__F(x, y)                                                           \
	RHO_ParentCheck;                                                           \
	dot(this->x##dim_, this->y##dim_, dst, src, this->i_axis_);

void Space::MapVectorFromParent_ps(RHO__args) const { RHO__F(parent_, ); }
void Space::MapVectorFromParent_rs(RHO__args) const { RHO__F(root_, ); }
void Space::MapVectorFromParent_pr(RHO__args) const { RHO__F(parent_, root_); }
void Space::MapVectorFromParent_rr(RHO__args) const { RHO__F(root_, root_); }

#undef RHO__F

#///////////////////////////////////////////////////////////////////////////////

#define RHO__F(x)                                                              \
	dot(this->x##dim_, this->root_dim_, dst, src, this->root_axis_);           \
	for (dim_t i(0); i != this->root_dim_; ++i) {                              \
		dst[i] += this->root_origin_[i];                                       \
	}

void Space::MapPointToRoot_sr(RHO__args) const { RHO__F(); }
void Space::MapPointToRoot_rr(RHO__args) const { RHO__F(root_); }

#undef RHO__F

#define RHO__F(x)                                                              \
	dot(this->x##dim_, this->root_dim_, dst, src, this->root_axis_);

void Space::MapVectorToRoot_sr(RHO__args) const { RHO__F(); }
void Space::MapVectorToRoot_rr(RHO__args) const { RHO__F(root_); }

#undef RHO__F

#///////////////////////////////////////////////////////////////////////////////

#define RHO__F(x)                                                              \
	dot(this->root_dim_, this->x##dim_, dst, src, this->i_root_axis_);         \
	for (dim_t i(0); i != this->x##dim_; ++i) {                                \
		dst[i] -= this->i_root_origin_[i];                                     \
	}

void Space::MapPointFromRoot_rs(RHO__args) const { RHO__F(); }
void Space::MapPointFromRoot_rr(RHO__args) const { RHO__F(root_); }

#undef RHO__F

#define RHO__F(x)                                                              \
	dot(this->root_dim_, this->x##dim_, dst, src, this->i_root_axis_);

void Space::MapVectorFromRoot_rs(RHO__args) const { RHO__F(); }
void Space::MapVectorFromRoot_rr(RHO__args) const { RHO__F(root_); }

#undef RHO__F
#undef RHO__args

#///////////////////////////////////////////////////////////////////////////////

#define RHO__args Num *dst, const Num *src, const Space *branch

#define RHO__F(x, y)                                                           \
	RHO__debug_if(this->root_ != branch->root_) {                              \
		RHO__throw__local("root error");                                       \
	}                                                                          \
	Vec temp;                                                                  \
	this->MapPointToRoot_##x##r(temp, src);                                    \
	branch->MapPointFromRoot_r##y(dst, temp);

void Space::MapPointToBranch_sb(RHO__args) const { RHO__F(s, s) }
void Space::MapPointToBranch_sr(RHO__args) const { RHO__F(s, r) }
void Space::MapPointToBranch_rb(RHO__args) const { RHO__F(r, s) }
void Space::MapPointToBranch_rr(RHO__args) const { RHO__F(r, r) }

#undef RHO__F

#define RHO__F(x, y)                                                           \
	RHO__debug_if(this->root_ != branch->root_) {                              \
		RHO__throw__local("root error");                                       \
	}                                                                          \
	Vec temp;                                                                  \
	this->MapVectorToRoot_##x##r(temp, src);                                   \
	branch->MapVectorFromRoot_r##y(dst, temp);

void Space::MapVectorToBranch_sb(RHO__args) const { RHO__F(s, s); }
void Space::MapVectorToBranch_sr(RHO__args) const { RHO__F(s, r); }
void Space::MapVectorToBranch_rb(RHO__args) const { RHO__F(r, s); }
void Space::MapVectorToBranch_rr(RHO__args) const { RHO__F(r, r); }

#undef RHO__F

#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////

bool Space::IncludePointFromRoot_r(const Num* src) const {
	if (!this->root_codim_) { return true; }

	Vec temp;

	dot(this->root_dim_, this->root_codim_, temp, src,
		this->i_root_axis_ + this->dim_);

	return Equal(this->root_codim_, temp, this->i_root_origin_ + this->dim_);
}

bool Space::IncludeVectorFromRoot_r(const Num* src) const {
	if (!this->root_codim_) { return true; }

	Vec temp;

	dot(this->root_dim_, this->root_codim_, temp, src,
		this->i_root_axis_ + this->dim_);

	for (dim_t i(0); i != this->root_codim_; ++i) {
		if (temp[i].ne<0>()) { return false; }
	}

	return true;
}

#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////

bool Space::Overlap(const Space* branch) const {
	RHO__debug_if(this->root_ != branch->root_) {
		RHO__throw__local("root error");
	}

	return this->dim_ == branch->dim_ && this->Include_(branch);
}

bool Space::Include(const Space* branch) const {
	RHO__debug_if(this->root_ != branch->root_) {
		RHO__throw__local("root error");
	}

	return branch->dim_ <= this->dim_ && this->Include_(branch);
}

bool Space::Include_(const Space* branch) const {
	if (!this->IncludePointFromRoot_r(branch->root_origin_)) { return false; }

	for (dim_t i(0); i != branch->dim_; ++i) {
		if (!this->IncludeVectorFromRoot_r(branch->root_axis_ +
										   RHO__max_dim * i)) {
			return false;
		}
	}

	return true;
}

#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////

void Space::SetLatestFalse() const { this->latest_ = false; }

Space* Space::SetDim(dim_t dim) {
	if (this->dim_ != dim) {
		RHO__debug_if(RHO__max_dim < dim) { RHO__throw__local("dim error"); }

		this->dim_ = dim;
		this->latest_arch_ = this->latest_ = false;
	}

	return this;
}

Space* Space::SetParent(const Space* parent) {
	if (this->parent_ == parent) { return this; }

#if RHO__debug_flag

	if (parent && this->root_ == parent->root_ &&
		this->depth_ < parent->depth_) {
		for (const Space* s(parent); s; s = s->parent_) {
			if (s == this) { RHO__throw__local("arch error"); }
		}
	}

#endif

	this->parent_ = parent;
	this->latest_arch_ = this->latest_ = false;

	return this;
}

#///////////////////////////////////////////////////////////////////////////////

Space* Space::SetOrigin(const Num* origin) {
	Vector::Copy(this->origin_, origin);
	this->latest_ = false;
	return this;
}

Space* Space::SetAxis(const Num* axis) {
	Matrix::Copy(this->axis_, axis);
	this->latest_ = false;
	return this;
}

Space* Space::SetOrigin(const Vector& origin) {
	RHO__dim_check(this->parent_dim_, origin.dim());
	return this->SetOrigin(&origin[0]);
}

Space* Space::SetAxis(const Matrix& axis) {
	RHO__dim_check(this->dim_, axis.col_dim());
	RHO__dim_check(this->parent_dim_, axis.row_dim());
	return this->SetAxis(&axis[0]);
}

#///////////////////////////////////////////////////////////////////////////////

Space* Space::RefreshArch() {
	const Space* s(nullptr);
	const Space* i(this);

	for (const Space* j; j = i->parent_; i = j) {
		j->temp_ = i;
		if (!j->latest_arch_) { s = j; }
	}

	if (s) {
		for (; s != this; s = s->temp_) { s->RefreshArch_(); }
		this->RefreshArch_();
	}

	if (this->latest_arch_) { this->RefreshArch_(); }

	return this;
}

const Space* Space::RefreshArch() const {
	return const_cast<Space*>(this)->RefreshArch();
}

void Space::RefreshArch_() const {
	if (this->parent_) {
		this->root_ = this->parent_->root_;
		this->root_dim_ = this->parent_->root_dim_;
		this->root_codim_ = this->root_dim_ - this->dim_;
	} else {
		this->root_ = this;
		this->root_dim_ = this->dim_;
		this->root_codim_ = 0;
	}

	this->latest_arch_ = true;
}

Space* Space::Refresh() {
	const Space* s(nullptr);
	const Space* t(nullptr);
	const Space* i(this);

	for (const Space* j; j = i->parent_; i = j) {
		j->temp_ = i;
		if (!j->latest_arch_) {
			s = t = j;
		} else if (!j->latest_) {
			s = j;
		}
	}

	if (s) {
		for (; s != t; s = s->temp_) { s->Refresh_(); }

		for (; s != this; s = s->temp_) {
			s->RefreshArch_();
			s->Refresh_();
		}

		this->RefreshArch_();
		this->Refresh_();
		return this;
	} else if (this->latest_arch_) {
		if (!this->latest_) { this->Refresh_(); }
	} else {
		this->RefreshArch_();
		this->Refresh_();
	}

	return this;
}

const Space* Space::Refresh() const {
	return const_cast<Space*>(this)->Refresh();
}

void Space::Refresh_() const {
	if (this->parent_) {
		for (dim_t i(this->parent_dim_); i != this->root_dim_; ++i) {
			this->origin_[i] = 0;
		}

		for (dim_t i(0); i != this->dim_; ++i) {
			for (dim_t j(this->parent_dim_); j != this->root_dim_; ++j) {
				this->axis_[RHO__max_dim * i + j] = 0;
			}
		}

		dot(this->parent_dim_, this->root_dim_, this->root_origin_,
			this->origin_, this->parent_->root_axis_);

#pragma unroll
		for (dim_t i(0); i != RHO__max_dim; ++i) {
			this->root_origin_[i] += this->parent_->root_origin_[i];
		}

		dot(this->dim_, this->parent_dim_, this->root_dim_, this->root_axis_,
			this->axis_, this->parent_->root_axis_);

#if RHO__debug_flag
		if (!Complement(this->dim_, this->root_dim_, this->root_axis_)) {
			RHO__throw__local("linear dependent error");
		}
#else
		Complement(this->dim_, this->root_dim_, this->root_axis_);
#endif

		inverse(this->root_dim_, this->i_root_axis_, this->root_axis_);

		dot(this->root_dim_, this->root_dim_, this->i_root_origin_,
			this->root_origin_, this->i_root_axis_);

		dot(this->root_codim_, this->root_dim_, this->root_dim_,
			this->axis_ + RHO__max_dim * this->dim_,
			this->root_axis_ + RHO__max_dim * this->dim_,
			this->parent_->i_root_axis_);
	} else {
		Fill<RHO__max_dim>(this->origin_, 0);
		Matrix::identity(this->axis_);

		Fill<RHO__max_dim>(this->root_origin_, 0);
		Matrix::identity(this->root_axis_);

		Fill<RHO__max_dim>(this->i_origin_, 0);
		Matrix::identity(this->i_axis_);

		Fill<RHO__max_dim>(this->i_root_origin_, 0);
		Matrix::identity(this->i_root_axis_);
	}

	this->latest_ = true;
}

}