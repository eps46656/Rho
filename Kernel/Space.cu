#include "hip/hip_runtime.h"
#include "define.cuh"
#include "Kernel.cuh"

#define RHO__throw__local(desc) RHO__throw(Space, __func__, desc)

#define RHO_ParentCheck                                                        \
	RHO__debug_if(!this->parent_)                                              \
		RHO__throw__local("Space " << this->id_ << " parent space error");

#define RHO__BranchCheck(branch)                                               \
	RHO__debug_if(this->root_ != branch->root_)                                \
		RHO__throw__local("Space " << branch->id_ << " is not a branch of "    \
								   << this->root_->id << ".");

#define RHO__SetOriginAxisCheck                                                \
	RHO__debug_if(!this->parent_) RHO__throw__local(                           \
		("Space " << this->id_                                                 \
				  << ", a root space, can not be set origin or axis."));

#define RHO__dim_check(x, y)                                                   \
	RHO__debug_if((x) != (y)) RHO__throw__local("dim error");

#define RHO__dim_check2(x1, y1, x2, y2)                                        \
	RHO__debug_if((x1) != (y1) || (x2) != (y2)) RHO__throw__local("dim "       \
																  "error");

namespace rho {

bool Space::latest() const {
	for (const Space* s(this); s; s = s->parent_) {
		if (!s->latest_) { return false; }
	}

	return true;
}

bool Space::is_root() const { return !this->parent_; }

#////////////////////////////////////////////////

Manager* Space::manager() { return this->manager_; }
const Manager* Space::manager() const { return this->manager_; }

Space* Space::root() { return this->root_; }
const Space* Space::root() const { return this->root_; }

Space* Space::parent() { return this->parent_; }
const Space* Space::parent() const { return this->parent_; }

const cntr::Vector<Space*>& Space::child() { return this->child_; }
const cntr::Vector<const Space*>& Space::child() const {
	return this->child_const_;
}

size_t Space::depth() const { return this->depth_; }

#////////////////////////////////////////////////

RHO__cuda const Num* Space::origin() const { return this->origin_; }
RHO__cuda const Num* Space::axis() const { return this->axis_; }

RHO__cuda const Num* Space::root_origin() const { return this->root_origin_; }
RHO__cuda const Num* Space::root_axis() const { return this->root_axis_; }

RHO__cuda const Num* Space::i_origin() const { return this->i_origin_; }
RHO__cuda const Num* Space::i_axis() const { return this->i_axis_; }

RHO__cuda const Num* Space::i_root_origin() const {
	return this->i_root_origin_;
}
RHO__cuda const Num* Space::i_root_axis() const { return this->i_root_axis_; }

#////////////////////////////////////////////////
#////////////////////////////////////////////////
#////////////////////////////////////////////////

dim_t Space::dim_s() const { return this->dim_s_; }
dim_t Space::dim_p() const { return this->dim_p_; }
dim_t Space::dim_r() const { return this->dim_r_; }
dim_t Space::dim_cp() const { return this->dim_cp_; }
dim_t Space::dim_cr() const { return this->dim_cr_; }

#////////////////////////////////////////////////

Space::Space(dim_t dim):
	latest_(false),

	manager_(new Manager(this)), root_(this), parent_(nullptr),

	depth_(0),

	dim_s_(dim), dim_p_(dim), dim_r_(dim), dim_cp_(0), dim_cr_(0) {
	RHO__debug_if(RHO__max_dim < dim) RHO__throw__local("dim error");
}

Space::Space(dim_t dim, Space* parent):
	latest_(false),

	manager_(parent->manager_), root_(parent->root_), parent_(parent),

	depth_(parent->depth_ + 1),

	dim_s_(dim), dim_p_(parent->dim_s_), dim_r_(parent->dim_r_),
	dim_cp_(this->dim_p_ - this->dim_s_), dim_cr_(this->dim_r_ - this->dim_s_) {
	RHO__debug_if(RHO__max_dim < dim) { RHO__throw__local("dim error"); }

	this->manager_->AddSpace_(this);

	this->manager_->space_.Insert(this);

	this->parent_->child_.Push(this);
}

#////////////////////////////////////////////////

void Space::Check() const {}

void Space::Refresh() const {
	RHO__throw__local("do not call this func\n");

	/*cntr::Vector<const Space*> stack;

	if (!this->latest_) {
		stack.Reserve(this->child_const_.size() < this->depth_ ?
					  this->depth_ : this->child_const_.size());
		stack.Push(this);

		for (auto i(this->parent_); i && !i->latest_; i = i->parent_)
			stack.Push(i);

		for (size_t i(stack.size() - 1); i; --i)
			stack[i]->RefreshMain_();

		this->RefreshMain_();
	}

	stack = this->child_const_;

	while (stack.size()) {
		const Space* space(stack.back());
		stack.Pop();
		stack.Insert(stack.end(),
					 space->child_const_.begin(),
					 space->child_const_.end());
		space->RefreshMain_();
	}*/

	this->RefreshSelf();
	this->RefreshDescendant_();
}

bool Space::RefreshSelf() const {
	/*cntr::Vector<const Space*> s(this->depth_ + 1);

	Space* a(this->parent_);

	for (size_t i(s.size() - 1); i; --i, a = a->parent_)
		s[i - 1] = a;

	s.back() = this;

	for (size_t i(0); i != s.size(); ++i) {
		if (s[i]->latest_) { continue; }

		for (; i != s.size(); ++i) {
			if (!s[i]->RefreshMain_()) { return false; }
		}

		return this->RefreshMain_();
	}

	return true;*/

	cntr::Vector<const Space*> s(this->depth_ + 1);

	const Space* a(this);
	size_t j(s.size());

	for (size_t i(s.size()); i; --i, a = a->parent_) {
		if (!((s[i - 1] = a)->latest_)) { j = i - 1; }
	}

	for (; j != s.size(); ++j) {
		if (!s[j]->RefreshMain_()) { return false; }
	}

	return true;
}

void Space::RefreshDescendant_() const {
	auto iter(this->child_.begin());

	for (auto end(this->child_.end()); iter != end; ++iter) {
		(*iter)->RefreshMain_();
		(*iter)->RefreshDescendant_();
	}
}

bool Space::RefreshMain_() const {
	this->Check();

	this->latest_ = true;

	if (this->parent_) {
		for (dim_t i(this->dim_p_); i != this->dim_r_; ++i) {
			this->origin_[i] = 0;
		}

		for (dim_t i(0); i != this->dim_s_; ++i) {
			for (dim_t j(this->dim_p_); j != this->dim_r_; ++j) {
				this->axis_[RHO__max_dim * i + j] = 0;
			}
		}

		dot(this->dim_p_, this->dim_r_, this->root_origin_, this->origin_,
			this->parent_->root_axis_);

#pragma unroll
		for (dim_t i(0); i != RHO__max_dim; ++i)
			this->root_origin_[i] += this->parent_->root_origin_[i];

		dot(this->dim_s_, this->dim_p_, this->dim_r_, this->root_axis_,
			this->axis_, this->parent_->root_axis_);

		if (!Complement(this->dim_s_, this->dim_r_, this->root_axis_))
			return false;

		inverse(this->dim_r_, this->i_root_axis_, this->root_axis_);

		dot(this->dim_r_, this->dim_r_, this->i_root_origin_,
			this->root_origin_, this->i_root_axis_);

		dot(this->dim_cr_, this->dim_r_, this->dim_r_,
			this->axis_ + RHO__max_dim * this->dim_s_,
			this->root_axis_ + RHO__max_dim * this->dim_s_,
			this->parent_->i_root_axis_);
	} else {
		Fill<RHO__max_dim>(this->origin_, 0);
		Matrix::identity(this->axis_);

		Fill<RHO__max_dim>(this->root_origin_, 0);
		Matrix::identity(this->root_axis_);

		Fill<RHO__max_dim>(this->i_origin_, 0);
		Matrix::identity(this->i_axis_);

		Fill<RHO__max_dim>(this->i_root_origin_, 0);
		Matrix::identity(this->i_root_axis_);
	}

	return true;
}

#///////////////////////////////////////////////////////////////////////////////

void Space::SetLatestFalse_() {
	this->latest_ = false;

	auto iter(this->child_.begin());

	for (auto end(this->child_.end()); iter != end; ++iter)
		(*iter)->SetLatestFalse_();
}

#///////////////////////////////////////////////////////////////////////////////

void Space::Delete() {
	this->parent_->child_.Erase(LinearSearch(
		this->parent_->child_.begin(), this->parent_->child_.end(), this));

	cntr::Vector<Space*> stack(this->child_);

	while (stack.size()) {
		Space* space(stack.back());
		stack.Pop();
		stack.Insert(stack.end(), space->child_.begin(), space->child_.end());

		this->manager_->DeleteSpace_(this);
		rho::Delete(space);
	}
}
/*
void Space::Delete_() {
	instance_().Erase(instance_().Find(this));

	auto iter(this->child_.begin());

	for (auto end(this->child_.end()); iter != end; ++iter)
		(*iter)->Delete();

	this->manager_->DeleteSpace_(this);
	rho::Delete(this);
}*/

#////////////////////////////////////////////////

Space* Space::set_origin(const Num* origin) {
	Copy<RHO__max_dim>(this->origin_, origin);
	this->latest_ = false;
	return this;
}

Space* Space::set_axis(const Num* axis) {
	Copy<RHO__max_dim_sq>(this->axis_, axis);
	this->latest_ = false;
	return this;
}

Space* Space::set_origin(const Vector& origin) {
	RHO__dim_check(this->dim_p_, origin.dim());
	return this->set_origin(&origin[0]);
}

Space* Space::set_axis(const Matrix& axis) {
	RHO__dim_check(this->dim_s_, axis.col_dim());
	RHO__dim_check(this->dim_p_, axis.row_dim());
	return this->set_axis(&axis[0]);
}

#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////

#define RHO__args Num *dst, const Num *src

#define RHO__F(x, y)                                                           \
	RHO_ParentCheck;                                                           \
	dot(this->dim_##x##_, this->dim_##y##_, dst, src, this->axis_);            \
	for (dim_t i(0); i != this->dim_##y##_; ++i) { dst[i] += this->origin_[i]; }

void Space::MapPointToParent_sp(RHO__args) const { RHO__F(s, p) }
void Space::MapPointToParent_sr(RHO__args) const { RHO__F(s, r) }
void Space::MapPointToParent_rp(RHO__args) const { RHO__F(r, p) }
void Space::MapPointToParent_rr(RHO__args) const { RHO__F(r, r) }

#undef RHO__F

#define RHO__F(x, y)                                                           \
	RHO_ParentCheck;                                                           \
	dot(this->dim_##x##_, dim_##y##_, dst, src, this->axis_);

void Space::MapVectorToParent_sp(RHO__args) const { RHO__F(s, p) }
void Space::MapVectorToParent_sr(RHO__args) const { RHO__F(s, r) }
void Space::MapVectorToParent_rp(RHO__args) const { RHO__F(r, p) }
void Space::MapVectorToParent_rr(RHO__args) const { RHO__F(r, r) }

#undef RHO__F

#///////////////////////////////////////////////////////////////////////////////

#define RHO__F(x, y)                                                           \
	RHO_ParentCheck;                                                           \
	dot(this->dim_##x##_, this->dim_##y##_, dst, src, this->i_axis_);          \
	for (dim_t i(0); i != this->dim_##y##_; ++i) {                             \
		dst[i] -= this->i_origin_[i];                                          \
	}

void Space::MapPointFromParent_ps(RHO__args) const { RHO__F(p, s) }
void Space::MapPointFromParent_rs(RHO__args) const { RHO__F(r, s) }
void Space::MapPointFromParent_pr(RHO__args) const { RHO__F(p, r) }
void Space::MapPointFromParent_rr(RHO__args) const { RHO__F(r, r) }

#undef RHO__F

#define RHO__F(x, y)                                                           \
	RHO_ParentCheck;                                                           \
	dot(this->dim_##x##_, this->dim_##y##_, dst, src, this->i_axis_);

void Space::MapVectorFromParent_ps(RHO__args) const { RHO__F(p, s) }
void Space::MapVectorFromParent_rs(RHO__args) const { RHO__F(r, s) }
void Space::MapVectorFromParent_pr(RHO__args) const { RHO__F(p, r) }
void Space::MapVectorFromParent_rr(RHO__args) const { RHO__F(r, r) }

#undef RHO__F

#///////////////////////////////////////////////////////////////////////////////

#define RHO__F(x, y)                                                           \
	dot(this->dim_##x##_, dim_##y##_, dst, src, this->root_axis_);             \
	for (size_t i(0); i != this->dim_##y##_; ++i)                              \
		dst[i] += this->root_origin_[i];

void Space::MapPointToRoot_sr(RHO__args) const { RHO__F(s, r) }
void Space::MapPointToRoot_rr(RHO__args) const { RHO__F(r, r) }

#undef RHO__F

#define RHO__F(x, y)                                                           \
	dot(this->dim_##x##_, this->dim_##y##_, dst, src, this->root_axis_);

void Space::MapVectorToRoot_sr(RHO__args) const { RHO__F(s, r) }
void Space::MapVectorToRoot_rr(RHO__args) const { RHO__F(r, r) }

#undef RHO__F

#///////////////////////////////////////////////////////////////////////////////

#define RHO__F(x, y)                                                           \
	dot(this->dim_##x##_, this->dim_##y##_, dst, src, this->i_root_axis_);     \
	for (size_t i(0); i != this->dim_##y##_; ++i)                              \
		dst[i] -= this->i_root_origin_[i];

void Space::MapPointFromRoot_rs(RHO__args) const { RHO__F(r, s) }
void Space::MapPointFromRoot_rr(RHO__args) const { RHO__F(r, r) }

#undef RHO__F

#define RHO__F(x, y)                                                           \
	dot(this->dim_##x##_, this->dim_##y##_, dst, src, this->i_root_axis_);

void Space::MapVectorFromRoot_rs(RHO__args) const {
	dot(this->dim_r_, this->dim_s_, dst, src, this->i_root_axis_);
}
void Space::MapVectorFromRoot_rr(RHO__args) const {
	dot(this->dim_r_, this->dim_r_, dst, src, this->i_root_axis_);
}

#undef RHO__F

#///////////////////////////////////////////////////////////////////////////////

#define RHO__F(x, y)                                                           \
	RHO__debug_if(this->root_ != branch->root_)                                \
		RHO__throw__local("root error");                                       \
	Vec temp;                                                            \
	this->MapPointToRoot_##x##r(temp, src);                                    \
	branch->MapPointFromRoot_r##y(dst, temp);

void Space::MapPointToBranch_sb(RHO__args, const Space* branch) const {
	RHO__F(s, s)
}
void Space::MapPointToBranch_sr(RHO__args, const Space* branch) const {
	RHO__F(s, r)
}
void Space::MapPointToBranch_rb(RHO__args, const Space* branch) const {
	RHO__F(r, s)
}
void Space::MapPointToBranch_rr(RHO__args, const Space* branch) const {
	RHO__F(r, r)
}

#undef RHO__F

#define RHO__F(x, y)                                                           \
	RHO__debug_if(this->root_ != branch->root_)                                \
		RHO__throw__local("root error");                                       \
	Vec temp;                                                            \
	this->MapVectorToRoot_##x##r(temp, src);                                   \
	branch->MapVectorFromRoot_r##y(dst, temp);

void Space::MapVectorToBranch_sb(RHO__args, const Space* branch) const {
	RHO__F(s, s)
}
void Space::MapVectorToBranch_sr(RHO__args, const Space* branch) const {
	RHO__F(s, r)
}
void Space::MapVectorToBranch_rb(RHO__args, const Space* branch) const {
	RHO__F(r, s)
}
void Space::MapVectorToBranch_rr(RHO__args, const Space* branch) const {
	RHO__F(r, r)
}

#undef RHO__F

#////////////////////////////////////////////////
#////////////////////////////////////////////////
#////////////////////////////////////////////////

bool Space::IncludePointFromRoot_r(const Num* src) const {
	if (!this->dim_cr_) { return true; }

	Vec temp;

	// dot(this->dim_r_, this->dim_cr_, temp, src, this->i_root_axis_rcr_);

	dot(this->dim_r_, this->dim_cr_, temp, src,
		this->i_root_axis_ + this->dim_s_);

	return Equal(this->dim_cr_, this->i_root_origin_ + this->dim_s_, temp);
}

bool Space::IncludeVectorFromRoot_r(const Num* src) const {
	if (!this->dim_cr_) { return true; }

	Vec temp;

	// dot(this->dim_r_, this->dim_cr_, temp, src, this->i_root_axis_rcr_);

	dot(this->dim_r_, this->dim_cr_, temp, src,
		this->i_root_axis_ + this->dim_s_);

	for (size_t i(0); i != this->dim_cr_; ++i) {
		if (temp[i].ne<0>()) { return false; }
	}

	return true;
}

#////////////////////////////////////////////////
#////////////////////////////////////////////////
#////////////////////////////////////////////////

bool Space::Overlap(const Space* branch) const {
	RHO__debug_if(this->root_ != branch->root_)
		RHO__throw__local("root space error");

	return this->dim_s_ == branch->dim_s_ && this->Include_(branch);
}

bool Space::Include(const Space* branch) const {
	RHO__debug_if(this->root_ != branch->root_)
		RHO__throw__local("root space error");

	return branch->dim_s_ <= this->dim_s_ && this->Include_(branch);
}

bool Space::Include_(const Space* branch) const {
	if (!this->IncludePointFromRoot_r(branch->root_origin_)) return false;

	for (size_t i(0); i != branch->dim_s_; ++i) {
		if (!this->IncludeVectorFromRoot_r(branch->root_axis_ +
										   this->dim_r_ * i)) {
			return false;
		}
	}

	return true;
}

#////////////////////////////////////////////////

void Space::AddChild_(Space* space) {
	this->child_.Push(space);
	this->child_const_.Push(space);
}

void Space::SubChild_(Space* space) {
	this->child_.Erase(
		LinearSearch(this->child_.begin(), this->child_.end(), space));
	this->child_const_.Erase(LinearSearch(this->child_const_.begin(),
										  this->child_const_.end(), space));
}

}
