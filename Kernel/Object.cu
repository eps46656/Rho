#include "define.cuh"
#include "Kernel.cuh"

namespace rho {

bool Object::active() const { return this->active_; }
bool Object::latest() const { return this->latest_; }

Manager* Object::manager() const { return this->manager_; }
Space* Object::root() const { return this->root_; }

size_t Object::dim_r() const { return this->dim_r_; }

const ComponentContainer& Object::cmpt() const { return this->cmpt_; }

const ComponentContainer& Object::active_cmpt() const {
	return this->active_cmpt_;
}

ComponentCollider* Object::cmpt_collider() const {
	return this->cmpt_collider_;
}

Material* Object::material() const { return this->material_; }

#///////////////////////////////////////////////////////////////////////////////

Object::Object(Space* root, Material* material):
	active_(true), latest_(false),

	manager_(root->manager()), root_(root),

	dim_r_(root->dim_s()),

	material_(material) {
	RHO__debug_if(root->parent()) RHO__throw(Object, __func__, "root error");

	this->manager_->AddObject_(this);
}

Object::~Object() {
	auto iter(this->cmpt_.begin());

	for (auto end(this->cmpt_.end()); iter != end; ++iter) (*iter)->Delete();

	this->manager_->DeleteObject_(this);
}

#///////////////////////////////////////////////////////////////////////////////

bool Object::Refresh() const {
	if (this->latest_) { return true; }

	this->latest_ = true;

	if (!this->material_) {
		this->material_ = this->manager_->default_material();
	}

	auto iter(this->cmpt_.begin());

	for (auto end(this->cmpt_.end()); iter != end; ++iter) {
		if (!(*iter)->Refresh()) { return false; }
	}

	return true;
}

#///////////////////////////////////////////////////////////////////////////////

void Object::Active(bool active) {
	if (active) { this->ActiveSelfAndAncestor(); }
	this->ActiveDescendant_(active);
}

void Object::ActiveSelfAndAncestor() {
	this->manager_->ActiveObjectTrue_(this);
	this->active_ = true;
}

void Object::ActiveDescendant_(bool active) {
	auto iter(this->cmpt_.begin());

	for (auto end(this->cmpt_.end()); iter != end; ++iter)
		(*iter)->Active(active);
}

#///////////////////////////////////////////////////////////////////////////////

void Object::Delete() {
	auto iter(this->cmpt_.begin());

	for (auto end(this->cmpt_.end()); iter != end; ++iter) (*iter)->Delete();
}

#///////////////////////////////////////////////////////////////////////////////

void Object::SetLatestFalse_() {
	auto iter(this->cmpt_.begin());

	for (auto end(this->cmpt_.end()); iter != end; ++iter)
		(*iter)->SetLatestFalse_();

	this->latest_ = false;
}

#///////////////////////////////////////////////////////////////////////////////

void Object::AddComponent_(Component* cmpt) {
	this->cmpt_.Insert(cmpt);
	this->latest_ = false;
}

void Object::DeleteComponent_(Component* cmpt) { this->cmpt_.FindErase(cmpt); }

}