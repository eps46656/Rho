
#include <hip/hip_runtime.h>
#if false

#include "define.cuh"
#include "Kernel.cuh"

namespace rho {

Space* DomainUniverse::root() const { return this->ref_->root(); }

#///////////////////////////////////////////////////////////////////////////////

DomainUniverse::DomainUniverse(Space* ref): Domain(Type::universe), ref_(ref) {}

#///////////////////////////////////////////////////////////////////////////////

bool DomainUniverse::Contain(const Num* root_point) const { return true; }

#///////////////////////////////////////////////////////////////////////////////

bool DomainUniverse::RayCastB(const Ray& ray) const { return false; }

RayCastData DomainUniverse::RayCast(const Ray& ray) const {
	return RayCastData();
}

void DomainUniverse::RayCastForRender(RayCastDataPair& rcdp,
									  ComponentCollider* cmpt_collider,
									  const Ray& ray) const {
	return;
}

bool DomainUniverse::RayCastFull(RayCastDataVector& dst, const Ray& ray) const {
	return true;
}

#///////////////////////////////////////////////////////////////////////////////

void DomainUniverse::GetTodTan(Num* dst, const RayCastData& rcd,
							   const Num* root_direct) const {
	Vector::Copy(dst, root_direct);
}

#///////////////////////////////////////////////////////////////////////////////

size_t DomainUniverse::Complexity() const { return 0; }

}

#endif