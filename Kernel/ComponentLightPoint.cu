#include "hip/hip_runtime.h"
#include "define.cuh"
#include "ComponentLightPoint.cuh"

#define B 0.1

namespace rho {

Space* ComponentLightPoint::ref() const { return this->ref_; }

Num3& ComponentLightPoint::intensity() { return this->intensity_; }
const Num3& ComponentLightPoint::intensity() const { return this->intensity_; }

#///////////////////////////////////////////////////////////////////////////////

ComponentLightPoint::ComponentLightPoint(Object* object, Space* ref,
										 const Num3& intensity):
	ComponentLight(object),
	intensity_(intensity), ref_(ref) {}

#///////////////////////////////////////////////////////////////////////////////

bool ComponentLightPoint::Refresh() const {
	return this->ref_->RefreshSelf() && this->intensity_[0].ge<0>() &&
		   this->intensity_[1].ge<0>() && this->intensity_[2].ge<0>();
}

#///////////////////////////////////////////////////////////////////////////////

Num3 ComponentLightPoint::intensity(
	const Num* root_point, const Tod& tod,
	const cntr::Vector<ComponentCollider*>& cmpt_collider,
	const Num* reflection_vector, const Texture::Data& texture_data, Ray& ray,
	Num pre_dist) const {
	// from light point to hit point

	Vec direct;
	Vector::sub(direct, root_point, this->ref_->root_origin());

	Num face_angle_cos(angle_cos(this->dim_r(), direct, tod.orth));
	bool indirect(face_angle_cos.lt<0>());

	// check if light is blocked by other colliders

	if (!indirect) {
		Copy<RHO__max_dim>(ray.origin, this->ref_->root_origin());
		Copy<RHO__max_dim>(ray.direct, direct);

		RayCastData a;

		for (size_t i(0); i != cmpt_collider.size(); ++i) {
			if (cmpt_collider[i]->domain()->RayCastB(ray)) {
				indirect = true;
				break;
			}
		}
	}

	Num3 r;
	Num length_sq(sq(pre_dist + abs(this->dim_r(), direct)));
	Num half_cos_sq(
		(Num(1) + angle_cos(this->dim_r(), direct, reflection_vector)) / 2);

	r[0] = (this->intensity_[0] / length_sq) *
		   pow(half_cos_sq, texture_data.shininess[0] / 2);
	r[1] = (this->intensity_[1] / length_sq) *
		   pow(half_cos_sq, texture_data.shininess[1] / 2);
	r[2] = (this->intensity_[2] / length_sq) *
		   pow(half_cos_sq, texture_data.shininess[2] / 2);

	if (indirect) {
		Num n(1 - abs(face_angle_cos));

		r[0] *= n;
		r[1] *= n;
		r[2] *= n;
	}

	return r;
}

}

#undef B