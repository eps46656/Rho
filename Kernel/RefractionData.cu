#include "hip/hip_runtime.h"
#include "define.cuh"
#include "Kernel.cuh"

namespace rho {

RefractionData::RefractionData(const RayCastData& rcd, const Tod& tod,
							   const ComponentCollider::Material* x,
							   const ComponentCollider::Material* y) {
	Num n(y->refraction_index / x->refraction_index);

	if (n.eq<1>()) {
		this->transmittance = 1;
		this->parallel_ratio = 1;
		return;
	}

	Num p_length_sq(sq(rcd->domain->dim_r(), tod.tan));
	Num n_length_sq(sq(rcd->domain->dim_r(), tod.orth));

#///////////////////////////////////////////////////////////////////////////////

	Num k(sq(n) * (p_length_sq + n_length_sq) - p_length_sq);

	if (k.lt<0>()) {
		this->transmittance = 0;
		return;
	}

	this->parallel_ratio = sqrt(n_length_sq / k);

#///////////////////////////////////////////////////////////////////////////////

	Num cos(sqrt(n_length_sq / (p_length_sq + n_length_sq)));
	Num sin(sqrt(p_length_sq / (p_length_sq + n_length_sq)));

	k = sqrt(sq(n) - sq(sin));

	this->transmittance = (sq(cos - k) / sq(cos + k) +
						   sq(sq(n) * cos - k) / sq(sq(n) * cos + k)) /
						  2;
}

}