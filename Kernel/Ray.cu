#include "define.cuh"
#include "Kernel.cuh"

namespace rho {

void Ray::point(Num* dst, Num t) const {
	line<RHO__max_dim>(dst, t, this->direct, this->origin);
}

bool Ray::RayCastFull(RayCastDataVector& rcdv,
					  const cntr::Vector<ComponentCollider*>& cmpt) const {
	for (size_t i(0); i != cmpt.size(); ++i) cmpt[i]->RayCastFull(rcdv, *this);

	return false;
}

void Ray::RayCastForRender(RayCastDataPair& dst,
						   const cntr::Vector<ComponentCollider*>& cmpt) const {
	dst[0] = nullptr;
	dst[1] = nullptr;

	for (size_t i(0); i != cmpt.size(); ++i)
		cmpt[i]->RayCastForRender(dst, *this);
}

}