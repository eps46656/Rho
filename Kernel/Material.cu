#include"define.cuh"
#include"Kernel.cuh"

namespace rho {

bool Material::Check()const {
	return
		this->refraction_index.ge<1>() &&

		this->transmittance[0].ge<0>() &&
		this->transmittance[1].ge<0>() &&
		this->transmittance[2].ge<0>() &&

		this->transmittance[0].le<1>() &&
		this->transmittance[1].le<1>() &&
		this->transmittance[2].le<1>();
}

}