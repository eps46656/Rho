#include "hip/hip_runtime.h"
#include "define.cuh"
#include "Kernel.cuh"

#define RHO__throw__local(description) RHO__throw(Camera, __func__, description)

#define RHO__task_stack_second (task_stack.second + task_stack.first)

namespace rho {

void Camera::RenderData::Clear() const {
	this->intensity[0] = this->intensity[1] = this->intensity[2] = this->dist =
		0;
}

#///////////////////////////////////////////////////////////////////////////////

Manager* Camera::manager() const { return this->manager_; }

Space* Camera::root() const { return this->root_; }
Space* Camera::ref() const { return this->ref_; }

size_t Camera::dim_r() const { return this->dim_r_; }

size_t Camera::image_height() const { return this->image_height_; }
size_t Camera::image_width() const { return this->image_width_; }
size_t Camera::image_size() const { return this->image_size_; }

void Camera::resize(size_t image_height, size_t image_width) {
	if (!image_height) { this->image_height_ = image_height; }
	if (!image_width) { this->image_width_ = image_width; }
}

size_t Camera::max_depth() const { return this->max_depth_; }

void Camera::set_max_depth(size_t max_depth) { this->max_depth_ = max_depth; }

pair<size_t, Camera::RenderData*>& Camera::render_data() const {
	return this->render_data_;
}

#///////////////////////////////////////////////////////////////////////////////

Camera::Camera(Space* ref, size_t image_height, size_t image_width,
			   size_t max_depth):
	manager_(ref->manager()),
	root_(ref->root()), ref_(ref), dim_r_(ref->dim_r()),
	image_height_(image_height), image_width_(image_width),
	image_size_(image_height * image_width), max_depth_(max_depth),
	render_data_(0, nullptr) {
	this->manager_->RegisterCamera_(this);
}

Camera::~Camera() {}

#///////////////////////////////////////////////////////////////////////////////

void Camera::RenderReady(size_t size) const {
	// printf("render ready begin\n");

	Camera_Render_pre_<<<1, 1>>>(this, size);

	// printf("render ready end\n");
}

void Camera::Render(size_t block_pos_h, size_t block_pos_w, size_t block_size_h,
					size_t block_size_w) const {
	/*printf("render begin\n");

	Camera_Render_main_ << <
		dim3(1, 1, 480),
		dim3(1, 1, 270) >> > (this);

	hipDeviceSynchronize();

	printf("render end\n");*/

	/*Camera_Render_ << <1, 1 >> > (this);*/

	Camera_Render_main_<<<3, 1024>>>(this, block_pos_h, block_pos_w,
									 block_size_h, block_size_w);
}

RHO__glb void Camera_Render_(const Camera* camera) {
	for (size_t i(0); i != camera->render_data_.first; ++i)
		camera->render_data_.second[i].rendered = false;
}

RHO__glb void Camera_Render_pre_(const Camera* camera, size_t size) {
	camera->ref_->RefreshSelf();

	{
		const RBT<Object*>& object(camera->manager_->active_object());

		auto iter(object.begin());

		if (RHO__debug_flag) {
			for (auto end(object.end()); iter != end; ++iter)
				if (!(*iter)->Refresh())
					RHO__throw__local("ReadyForRendering error");
		} else {
			for (auto end(object.end()); iter != end; ++iter)
				(*iter)->Refresh();
		}
	}

	{
		const ComponentContainer& cmpt_cntr(camera->manager_->active_cmpt());

		camera->cmpt_collider_.Clear();
		camera->cmpt_collider_.Reserve(cmpt_cntr.size());

		camera->cmpt_light_.Clear();
		camera->cmpt_light_.Reserve(cmpt_cntr.size());

		auto iter(cmpt_cntr.begin());

		for (auto end(cmpt_cntr.end()); iter != end; ++iter) {
			if (!(*iter)->Refresh()) RHO__throw__local("Refresh error");

			switch ((*iter)->type) {
				case Component::Type::collider: {
					camera->cmpt_collider_.Push(
						static_cast<ComponentCollider*>(*iter));

					break;
				}
				case Component::Type::light: {
					camera->cmpt_light_.Push(
						static_cast<ComponentLight*>(*iter));

					break;
				}
			}
		}

		Sort(camera->cmpt_collider_.begin(), camera->cmpt_collider_.end(),
			 [](const ComponentCollider* x, const ComponentCollider* y) {
				 return x->domain()->Complexity() < y->domain()->Complexity();
			 });
	}

#///////////////////////////////////////////////////////////////////////////////

	camera->direct_f_.set_dim(3);
	camera->direct_h_.set_dim(3);
	camera->direct_w_.set_dim(3);

	Copy(camera->dim_r_, camera->direct_f_, camera->ref_->root_axis());
	Copy(camera->dim_r_, camera->direct_h_,
		 camera->ref_->root_axis() + RHO__max_dim);
	Copy(camera->dim_r_, camera->direct_w_,
		 camera->ref_->root_axis() + RHO__max_dim * 2);

	camera->direct_f_ -= camera->direct_h_;
	camera->direct_f_ -= camera->direct_w_;

	camera->direct_h_ *= Num(2) / camera->image_height_;
	camera->direct_w_ *= Num(2) / camera->image_width_;

#///////////////////////////////////////////////////////////////////////////////

	if (camera->render_data_.first < size) {
		camera->render_data_.first = size;
		Delete(camera->render_data_.second);
		camera->render_data_.second = Malloc<Camera::RenderData>(size);
	}

	// camera->RenderDataRefresh_(render_data);
}

#///////////////////////////////////////////////////////////////////////////////

RHO__glb void Camera_Render_main_(const Camera* camera,
								  const size_t block_pos_h,
								  const size_t block_pos_w,
								  const size_t block_size_h,
								  const size_t block_size_w) {
	const size_t thread_num(RHO__thread_num);
	const size_t thread_id(RHO__thread_index);
	const size_t block_size(block_size_h * block_size_w);

	// variable to store current trace data

	size_t render_index(thread_id);
	Camera::RenderData* render_data;

	Ray ray;

	Num dist;
	size_t depth;
	Num3 decay;

	Num dist_sq;
	Num d_dist;

	RayCastDataPair rcdp;
	Vector point[2];

	ComponentCollider* collider_a;
	ComponentCollider* collider_b;

	Material* material_a;
	Material* material_b;

	Texture::Data texture_data;

	Num3 transmittance;
	Num3 reflectance;
	Num3 difuss_reflectance;

	Tod tod;

	Vector reflection_vector;

	pair<size_t, Camera::Task*> task_stack{ 0, Malloc<Camera::Task>(
												   camera->max_depth_) };

	NumVector temp;

#///////////////////////////////////////////////////////////////////////////////

	for (;;) {
		if (task_stack.first) {
			// the current have not been done
			// we pop the task from pre-tracing

			--task_stack.first;

			Vector::Copy(ray.origin, RHO__task_stack_second->origin);
			Vector::Copy(ray.direct, RHO__task_stack_second->direct);

			dist = RHO__task_stack_second->dist;
			depth = RHO__task_stack_second->depth;

			decay = RHO__task_stack_second->decay;

		} else {
			// if then current pixel have been done
			// task_stack will be vacant
			// then we can process the next

			if (block_size <= render_index) { return; }

			render_data = camera->render_data_.second + render_index;
			render_data->dist = 0;
			render_data->intensity[0] = 0;
			render_data->intensity[1] = 0;
			render_data->intensity[2] = 0;

			size_t i(render_index / block_size_w);
			size_t j(render_index - i * block_size_w);

			Vector::Copy(ray.origin, camera->ref_->root_origin());

#pragma unroll
			for (dim_t k(0); k != RHO__max_dim; ++k) {
				ray.direct[k] = camera->direct_f_[k] +
								camera->direct_h_[k] * (block_pos_h + i) +
								camera->direct_w_[k] * (block_pos_w + j);
			}

			dist = 0;
			depth = 0;

			decay[0] = 1;
			decay[1] = 1;
			decay[2] = 1;

			render_index += thread_num;
		}

#///////////////////////////////////////////////////////////////////////////////

		// every point between ray.origin and the first hit point is
		// in the material a
		// every point between the first and second hit points is
		// in the material b

		ray.RayCastForRender(rcdp, camera->cmpt_collider_);

		if (!rcdp[0]) {
			/*goto function_head; */
			continue;
		}

		line<RHO__max_dim>(point[0], rcdp[0]->t, ray.direct, ray.origin);

		if (rcdp[1])
			line<RHO__max_dim>(point[1], rcdp[1]->t, ray.direct, ray.origin);

#///////////////////////////////////////////////////////////////////////////////

			// 在另一面計算在材質a中的路徑長
			// 計算在材質a中的穿透率

#pragma unroll
		for (dim_t i(0); i != RHO__max_dim; ++i)
			temp[i] = (ray.origin[i] + point[0][i]) / 2;

		material_a = (collider_a = camera->manager_->GetComponentCollider(temp))
						 ? collider_a->object()->material()
						 : camera->manager_->void_material();

		// get collider_a

		// if collider_a is exist
		// get material from its object

		// if not
		// get void material from manager

		// object's material are initialized to default material

		d_dist = abs(camera->dim_r(), ray.direct) * rcdp[0]->t;

		if (render_data->dist.eq<0>()) { render_data->dist = d_dist; }

		dist_sq = sq(dist += d_dist);

		decay[0] *= pow(material_a->transmittance[0], d_dist);
		decay[1] *= pow(material_a->transmittance[1], d_dist);
		decay[2] *= pow(material_a->transmittance[2], d_dist);

#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////

		rcdp[0]->domain->GetTodTan(tod.tan, rcdp[0], ray.direct);

#pragma unroll
		for (dim_t i(0); i != RHO__max_dim; ++i)
			tod.orth[i] = ray.direct[i] - tod.tan[i];

		texture_data =
			rcdp[0]->cmpt_collider->texture()->GetData(point[0], tod.tan);

#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////

		transmittance = texture_data.transmittance;
		reflectance = texture_data.reflectance;

		// transmission

		if (transmittance[0].ne<0>() || transmittance[1].ne<0>() ||
			transmittance[2].ne<0>()) {
			/*collider_b = camera->manager_->GetComponentCollider(
				rcdp[1] ? (point[0] + point[1]) / 2 : point[0] + ray.direct);*/

			if (rcdp[1]) {
#pragma unroll
				for (dim_t i(0); i != RHO__max_dim; ++i)
					temp[i] = (point[0][i] + point[1][i]) / 2;
			} else {
#pragma unroll
				for (dim_t i(0); i != RHO__max_dim; ++i)
					temp[i] = point[0][i] + ray.direct[i];
			}

			material_b =
				(collider_b = camera->manager_->GetComponentCollider(temp))
					? collider_b->object()->material()
					: camera->manager_->void_material();

			if (material_b->transmittance[0].ne<0>() ||
				material_b->transmittance[1].ne<0>() ||
				material_b->transmittance[2].ne<0>()) {
				RefractionData refraction(rcdp[0], tod, material_a, material_b);

				if (refraction.transmittance.eq<0>()) {
					reflectance[0] += transmittance[0];
					reflectance[1] += transmittance[1];
					reflectance[2] += transmittance[2];

					transmittance[0] = 0;
					transmittance[1] = 0;
					transmittance[2] = 0;
				} else {
					Num n(1 - refraction.transmittance);
					Num3 next_decay;

					reflectance[0] += transmittance[0] * n;
					reflectance[1] += transmittance[1] * n;
					reflectance[2] += transmittance[2] * n;

					transmittance[0] *= refraction.transmittance;
					transmittance[1] *= refraction.transmittance;
					transmittance[2] *= refraction.transmittance;

					next_decay[0] = decay[0] * transmittance[0];
					next_decay[1] = decay[1] * transmittance[1];
					next_decay[2] = decay[2] * transmittance[2];

					// after (long long) judge
					// we push a task to task_stack

					// printf("reflection task add\n");

					if (task_stack.first < camera->max_depth_) {
						Vector::Copy(RHO__task_stack_second->origin, point[0]);
						line<RHO__max_dim>(RHO__task_stack_second->direct,
										   refraction.parallel_ratio, tod.tan,
										   tod.orth);
						RHO__task_stack_second->dist = dist;
						RHO__task_stack_second->depth = depth + 1;
						RHO__task_stack_second->decay = Move(next_decay);

						++task_stack.first;
					}
				}
			}
		}

#pragma unroll
		for (dim_t i(0); i != RHO__max_dim; ++i)
			reflection_vector[i] = tod.tan[i] - tod.orth[i];

		// reflection

		if (reflectance[0].ne<0>() || reflectance[1].ne<0>() ||
			reflectance[2].ne<0>()) {
			Num3 next_decay;
			next_decay[0] = decay[0] * reflectance[0];
			next_decay[1] = decay[1] * reflectance[1];
			next_decay[2] = decay[2] * reflectance[2];

			/*
			camera->min_recv_intensity_[0] < camera->intensity_sum_[0]
				* next_intensity_decay[0] / dist_sq ||
				camera->min_recv_intensity_[1] < camera->intensity_sum_[1]
				* next_intensity_decay[1] / dist_sq ||
				camera->min_recv_intensity_[2] < camera->intensity_sum_[2]
				* next_intensity_decay[2] / dist_sq
			*/

			if (task_stack.first < camera->max_depth_) {
				// after (long long) judge
				// we push a task to task_stack

				Vector::Copy(RHO__task_stack_second->origin, point[0]);
				Vector::Copy(RHO__task_stack_second->direct, reflection_vector);
				RHO__task_stack_second->dist = dist;
				RHO__task_stack_second->depth = depth + 1;
				RHO__task_stack_second->decay = Move(next_decay);

				++task_stack.first;
			}
		}

#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////

		// Bling Phong specular
		// check blocked

		{
			bool b(true);

			for (size_t i(0); i != 3; ++i) {
				difuss_reflectance[i] = 1 - transmittance[i] - reflectance[i];
				if ((difuss_reflectance[i]).le<0>()) {
					difuss_reflectance[i] = 0;
				} else {
					b = false;
				}
			}

			if (b) {
				// goto function_head;
				continue;
			}
		}

		for (size_t i(0); i != camera->cmpt_light_.size(); ++i) {
			// influence caused by position is processed in
			// ComponentLight::intensity point tod reflection_vector ray
			// pre_length

			// influence caused by texture or material is processed in Camera
			// material transmittence
			// texture reflectance
			// texture transmittance
			// refraction transmittance

			Num3 intensity(camera->cmpt_light_[i]->intensity(
				point[0], tod, camera->cmpt_collider_, reflection_vector,
				texture_data, ray, dist));

			render_data->intensity[0] += texture_data.color[0] / 255 *
										 intensity[0] * difuss_reflectance[0] *
										 decay[0];
			render_data->intensity[1] += texture_data.color[1] / 255 *
										 intensity[1] * difuss_reflectance[1] *
										 decay[1];
			render_data->intensity[2] += texture_data.color[2] / 255 *
										 intensity[2] * difuss_reflectance[2] *
										 decay[2];
		}
	}

	// goto function_head;
}

void Camera::RenderDataRefresh_(RenderData* render_data) const {
	/*if (this->render_data_.first < this->size_) {
		Free(this->render_data_.second);
		this->render_data_.second =
			Malloc<RenderData>(this->size_);
	}*/

	this->render_data_.second = render_data;

	printf("render data alloc end\n");
}
}
