#include "hip/hip_runtime.h"
#include "define.cuh"
#include "Kernel.cuh"

#define RHO__throw__local(desc) RHO__throw(Camera, __func__, desc)

#define RHO__task_stack_second (task_stack.second + task_stack.first)

namespace rho {

void Camera::RenderData::Clear() const {
	this->intensity[0] = this->intensity[1] = this->intensity[2] = this->dist =
		0;
}

#///////////////////////////////////////////////////////////////////////////////

Space* Camera::root() const { return this->ref_->root(); }
Space* Camera::ref() const { return this->ref_; }

size_t Camera::dim_r() const { return this->ref_->dim_r(); }

size_t Camera::image_height() const { return this->image_height_; }
size_t Camera::image_width() const { return this->image_width_; }
size_t Camera::image_size() const { return this->image_size_; }

void Camera::resize(size_t image_height, size_t image_width) {
	if (!image_height) { this->image_height_ = image_height; }
	if (!image_width) { this->image_width_ = image_width; }
}

size_t Camera::max_depth() const { return this->max_depth_; }
void Camera::set_max_depth(size_t max_depth) { this->max_depth_ = max_depth; }

size_t Camera::render_data_size() const { return this->render_data_size_; }
Camera::RenderData* Camera::render_data() const { return this->render_data_; }

ComponentCollider::Material& Camera::void_cmpt_collider_material() {
	return this->void_cmpt_collider_material_;
}

const ComponentCollider::Material& Camera::void_cmpt_collider_material() const {
	return this->void_cmpt_collider_material_;
}

#///////////////////////////////////////////////////////////////////////////////

void Camera::AddCmptCollider(ComponentCollider* cmpt_collider) {
	this->cmpt_collider__ray_cast_order_.Push(cmpt_collider);
	this->cmpt_collider__detect_order_.Push(cmpt_collider);
}

void Camera::AddCmptLight(ComponentLight* cmpt_light) {
	this->cmpt_light_.Push(cmpt_light);
}

#///////////////////////////////////////////////////////////////////////////////

Camera::Camera(Space* ref, size_t image_height, size_t image_width,
			   size_t max_depth):
	ref_(ref),
	image_height_(image_height), image_width_(image_width),
	image_size_(image_height * image_width), max_depth_(max_depth),
	render_data_size_(0), render_data_(nullptr) {}

Camera::~Camera() {}

#///////////////////////////////////////////////////////////////////////////////

void Camera::RenderReady(size_t size) const {
	CameraRenderReady_<<<1, 1>>>(this, size);
}

void Camera::Render(size_t block_pos_h, size_t block_pos_w, size_t block_size_h,
					size_t block_size_w) const {
	CameraRenderMain_<<<32, 1024>>>(this, block_pos_h, block_pos_w,
									block_size_h, block_size_w);
}

RHO__glb void CameraRenderReady_(const Camera* camera, size_t size) {
	RHO__debug_if(!camera->void_cmpt_collider_material_.Check()) {
		RHO__throw__local("material error");
	}

	camera->ref_->RefreshSelf();

	for (size_t i(0); i != camera->cmpt_light_.size(); ++i) {
#if RHO__debug_flag
		for (size_t j(i + 1); j != camera->cmpt_light_.size(); ++j) {
			if (camera->cmpt_light_[i] == camera->cmpt_light_[j]) {
				RHO__throw__local("repeat cmpt light");
			}
		}

		if (!camera->cmpt_light_[i]->Refresh()) {
			RHO__throw__local("cmpt light refresh error");
		}
#else
		camera->cmpt_light_[i]->Refresh();
#endif
	}

	{
		// compare cmpt_collider__ray_cast_order_ and
		// cmpt_collider__detect_order_
		// to ensure their elements are all same
		// and refresh them

		size_t size(camera->cmpt_collider__ray_cast_order_.size());

		RHO__debug_if(size != camera->cmpt_collider__detect_order_.size()) {
			RHO__throw__local("size error");
		}

		cntr::Vector<ComponentCollider*> temp(
			camera->cmpt_collider__ray_cast_order_);
		Sort(temp.begin(), temp.end());

		for (size_t i(0); i != size; ++i) {
			if (BinarySearch(temp, size,
							 camera->cmpt_collider__detect_order_[i]) == size) {
				RHO__throw__local("not found error");
			}

			if (!camera->cmpt_collider__detect_order_[i]->Refresh()) {
				RHO__throw__local("refresh error");
			}
		}

		Sort(camera->cmpt_collider__ray_cast_order_.begin(),
			 camera->cmpt_collider__ray_cast_order_.end(),
			 [](const ComponentCollider* x, const ComponentCollider* y) {
				 return x->domain()->RayCastComplexity() <
						y->domain()->RayCastComplexity();
			 });
	}

#///////////////////////////////////////////////////////////////////////////////

	{
		const Num* a[]{ camera->ref_->root_axis(),
						camera->ref_->root_axis() + RHO__max_dim,
						camera->ref_->root_axis() + RHO__max_dim * 2 };

#pragma unroll
		for (dim_t i(0); i != RHO__max_dim; ++i)
			camera->direct_f_[i] = a[0][i] - a[1][i] - a[2][i];

#pragma unroll
		for (dim_t i(0); i != RHO__max_dim; ++i)
			camera->direct_h_[i] = a[1][i] * 2 / camera->image_height_;

#pragma unroll
		for (dim_t i(0); i != RHO__max_dim; ++i)
			camera->direct_w_[i] = a[2][i] * 2 / camera->image_width_;
	}

#///////////////////////////////////////////////////////////////////////////////

	if (camera->render_data_size_ < size) {
		camera->render_data_size_ = size;
		Delete(camera->render_data_);
		camera->render_data_ = Malloc<Camera::RenderData>(size);
	}
}

#///////////////////////////////////////////////////////////////////////////////

RHO__glb void CameraRenderMain_(const Camera* camera, const size_t block_pos_h,
								const size_t block_pos_w,
								const size_t block_size_h,
								const size_t block_size_w) {
	const size_t thread_num(RHO__thread_num);
	const size_t thread_id(RHO__thread_index);
	const size_t block_size(block_size_h * block_size_w);

	size_t render_index(thread_id);
	Camera::RenderData* render_data;

	Num dist_sq;
	Num d_dist;

	RayCastDataPair rcdp;
	Vec point[2];

	ComponentCollider* collider[3];

	const ComponentCollider::Material* material[2];

	Texture::Data texture_data;

	Num3 transmittance;
	Num3 reflectance;
	Num3 difuss_reflectance;

	Tod tod;

	Vector reflection_vector;

	size_t task_size(0);
	cntr::BidirectionalNode task_node;

#define RHO__static_task_size 10

	Camera::Task static_task[RHO__static_task_size];
	// this task is to avoid using New<Camera::Task>()

	for (size_t i(0); i != RHO__static_task_size; ++i)
		task_node.PushNext(static_task + i);

	Camera::Task* task;
	Camera::Task* next_task;

	Vec temp;

#///////////////////////////////////////////////////////////////////////////////

	for (;; --task_size) {
		if (task_size) {
			// the current have not been done
			// we pop the task from pre-tracing
			task = static_cast<Camera::Task*>(task->prev);
		} else {
			// if then current pixel have been done
			// task_stack will be vacant
			// then we can process the next

			if (block_size <= render_index) { return; }

			++task_size;
			task = static_cast<Camera::Task*>(task_node.next);

			render_data = camera->render_data_ + render_index;
			render_data->dist = 0;
			render_data->intensity[0] = 0;
			render_data->intensity[1] = 0;
			render_data->intensity[2] = 0;

			size_t i(render_index / block_size_w);
			size_t j(render_index - i * block_size_w);

			Vector::Copy(task->ray.origin, camera->ref_->root_origin());

			i += block_pos_h;
			j += block_pos_w;

#pragma unroll
			for (dim_t k(0); k != RHO__max_dim; ++k) {
				task->ray.direct[k] = camera->direct_f_[k] +
									  camera->direct_h_[k] * i +
									  camera->direct_w_[k] * j;
			}

			task->dist = 0;
			task->depth = 0;

			task->decay[0] = 1;
			task->decay[1] = 1;
			task->decay[2] = 1;

			render_index += thread_num;
		}

#///////////////////////////////////////////////////////////////////////////////

		// every point between ray.origin and the first hit point is
		// in the material a
		// every point between the first and second hit points is
		// in the material b

		rcdp[0] = nullptr;
		rcdp[1] = nullptr;
		collider[2] = nullptr;

		{
			Num pre_t(-1);

			for (size_t i(0);
				 i != camera->cmpt_collider__ray_cast_order_.size(); ++i) {
				camera->cmpt_collider__ray_cast_order_[i]
					->domain()
					->RayCastPair(rcdp, task->ray);

				if (rcdp[0]) {
					if (pre_t != rcdp[0]->t) {
						pre_t = rcdp[0]->t;
						collider[2] = camera->cmpt_collider__ray_cast_order_[i];
					}
				}
			}
		}

		if (!rcdp[0]) {
			/*goto function_head; */
			continue;
		}

		task->ray.point(point[0], rcdp[0]->t);

		if (rcdp[1]) { task->ray.point(point[1], rcdp[1]->t); }

#///////////////////////////////////////////////////////////////////////////////

		// calculate the dist fromt origin to point[0]
		// to get the transmittance through material a

		task->ray.point(temp, rcdp[0]->t / 2);

		collider[0] = nullptr;

		for (size_t i(0); i != camera->cmpt_collider__detect_order_.size();
			 ++i) {
			if (camera->cmpt_collider__detect_order_[i]->Contain(temp)) {
				collider[0] = camera->cmpt_collider__detect_order_[i];
				break;
			}
		}

		material[0] = collider[0] ? &collider[0]->material()
								  : &camera->void_cmpt_collider_material();

		// get collider[0]

		// if collider[0] is exist
		// get material from its object

		// if not
		// get void material from manager

		// object's material are initialized to default material

		d_dist = abs(camera->dim_r(), task->ray.direct) * rcdp[0]->t;

		if (render_data->dist.eq<0>()) { render_data->dist = d_dist; }

		dist_sq = sq(task->dist += d_dist);

		task->decay[0] *= pow(material[0]->transmittance[0], d_dist);
		task->decay[1] *= pow(material[0]->transmittance[1], d_dist);
		task->decay[2] *= pow(material[0]->transmittance[2], d_dist);

#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////

		rcdp[0]->domain->GetTodTan(tod.tan, rcdp[0], task->ray.direct);

#pragma unroll
		for (dim_t i(0); i != RHO__max_dim; ++i)
			tod.orth[i] = task->ray.direct[i] - tod.tan[i];

		texture_data = collider[2]->texture()->GetData(point[0], tod.tan);

#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////

		transmittance = texture_data.transmittance;
		reflectance = texture_data.reflectance;

		// transmission

		if (transmittance[0].ne<0>() || transmittance[1].ne<0>() ||
			transmittance[2].ne<0>()) {
			task->ray.point(temp, rcdp[1] ? ((rcdp[0]->t + rcdp[1]->t) / 2)
										  : (rcdp[0]->t + 1));

			collider[1] = nullptr;

			for (size_t i(0); i != camera->cmpt_collider__detect_order_.size();
				 ++i) {
				if (camera->cmpt_collider__detect_order_[i]->Contain(temp)) {
					collider[1] = camera->cmpt_collider__detect_order_[i];
					break;
				}
			}

			material[1] = collider[1] ? &collider[1]->material()
									  : &camera->void_cmpt_collider_material();

			if (material[1]->transmittance[0].ne<0>() ||
				material[1]->transmittance[1].ne<0>() ||
				material[1]->transmittance[2].ne<0>()) {
				RefractionData refraction(rcdp[0], tod, material[0],
										  material[1]);

				if (refraction.transmittance.eq<0>()) {
					reflectance[0] += transmittance[0];
					reflectance[1] += transmittance[1];
					reflectance[2] += transmittance[2];

					transmittance[0] = 0;
					transmittance[1] = 0;
					transmittance[2] = 0;
				} else {
					Num n(1 - refraction.transmittance);
					Num3 next_decay;

					reflectance[0] += transmittance[0] * n;
					reflectance[1] += transmittance[1] * n;
					reflectance[2] += transmittance[2] * n;

					transmittance[0] *= refraction.transmittance;
					transmittance[1] *= refraction.transmittance;
					transmittance[2] *= refraction.transmittance;

					next_decay[0] = task->decay[0] * transmittance[0];
					next_decay[1] = task->decay[1] * transmittance[1];
					next_decay[2] = task->decay[2] * transmittance[2];

					// after (long long) judge
					// we push a task to task_stack

					// printf("reflection task add\n");

					if (task->depth < camera->max_depth_) {
						if (task->next == &task_node) {
							task->PushPrev(next_task = New<Camera::Task>());
						} else {
							cntr::BidirectionalNode::Swap(
								*task, *(next_task = static_cast<Camera::Task*>(
											 task->next)));
						}

						Vector::Copy(next_task->ray.origin, point[0]);
						line<RHO__max_dim>(next_task->ray.direct,
										   refraction.parallel_ratio, tod.tan,
										   tod.orth);
						next_task->dist = task->dist;
						next_task->depth = task->depth + 1;
						next_task->decay = Move(next_decay);

						++task_size;
					}
				}
			}
		}

#pragma unroll
		for (dim_t i(0); i != RHO__max_dim; ++i)
			reflection_vector[i] = tod.tan[i] - tod.orth[i];

		// reflection

		if (reflectance[0].ne<0>() || reflectance[1].ne<0>() ||
			reflectance[2].ne<0>()) {
			Num3 next_decay;
			next_decay[0] = task->decay[0] * reflectance[0];
			next_decay[1] = task->decay[1] * reflectance[1];
			next_decay[2] = task->decay[2] * reflectance[2];

			if (task->depth < camera->max_depth_) {
				// after (long long) judge
				// we push a task to task_stack

				if (task->next == &task_node) {
					task->PushPrev(next_task = New<Camera::Task>());
				} else {
					cntr::BidirectionalNode::Swap(
						*task,
						*(next_task = static_cast<Camera::Task*>(task->next)));
				}

				Vector::Copy(next_task->ray.origin, point[0]);
				Vector::Copy(next_task->ray.direct, reflection_vector);
				next_task->dist = task->dist;
				next_task->depth = task->depth + 1;
				next_task->decay = next_decay;

				++task_size;
			}
		}

#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////

		// Bling Phong specular
		// check blocked

		{
			bool b(true);

			for (size_t i(0); i != 3; ++i) {
				difuss_reflectance[i] = 1 - transmittance[i] - reflectance[i];
				if ((difuss_reflectance[i]).le<0>()) {
					difuss_reflectance[i] = 0;
				} else {
					b = false;
				}
			}

			if (b) {
				// goto function_head;
				continue;
			}
		}

		for (size_t i(0); i != camera->cmpt_light_.size(); ++i) {
			// influence caused by position is processed in
			// ComponentLight::intensity point tod reflection_vector ray
			// pre_length

			// influence caused by texture or material is processed in Camera
			// material transmittence
			// texture reflectance
			// texture transmittance
			// refraction transmittance

			Num3 intensity(camera->cmpt_light_[i]->intensity(
				point[0], tod, camera->cmpt_collider__ray_cast_order_,
				reflection_vector, texture_data, task->ray, task->dist));

			render_data->intensity[0] += texture_data.color[0] / 255 *
										 intensity[0] * difuss_reflectance[0] *
										 task->decay[0];
			render_data->intensity[1] += texture_data.color[1] / 255 *
										 intensity[1] * difuss_reflectance[1] *
										 task->decay[1];
			render_data->intensity[2] += texture_data.color[2] / 255 *
										 intensity[2] * difuss_reflectance[2] *
										 task->decay[2];
		}
	}

	Camera::Task* n(static_cast<Camera::Task*>(task_node.next));
	Camera::Task* m;

	while (n != &task_node) {
		m = static_cast<Camera::Task*>(n->next);
		int k(n - static_task);
		if (!(0 < k && k < RHO__static_task_size)) { Delete(n); }
		n = m;
	}
}
}
