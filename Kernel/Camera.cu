#include "hip/hip_runtime.h"
#include "define.cuh"
#include "Kernel.cuh"

#define RHO__throw__local(desc) RHO__throw(Camera, __func__, desc)

#define RHO__task_stack_second (task_stack.second + task_stack.first)

namespace rho {

void Camera::RenderData::Clear() const {
	this->intensity[0] = this->intensity[1] = this->intensity[2] = this->dist =
		0;
}

#///////////////////////////////////////////////////////////////////////////////

Manager* Camera::manager() const { return this->manager_; }

Space* Camera::root() const { return this->root_; }
Space* Camera::ref() const { return this->ref_; }

size_t Camera::dim_r() const { return this->dim_r_; }

size_t Camera::image_height() const { return this->image_height_; }
size_t Camera::image_width() const { return this->image_width_; }
size_t Camera::image_size() const { return this->image_size_; }

void Camera::resize(size_t image_height, size_t image_width) {
	if (!image_height) { this->image_height_ = image_height; }
	if (!image_width) { this->image_width_ = image_width; }
}

size_t Camera::max_depth() const { return this->max_depth_; }

void Camera::set_max_depth(size_t max_depth) { this->max_depth_ = max_depth; }

pair<size_t, Camera::RenderData*>& Camera::render_data() const {
	return this->render_data_;
}

#///////////////////////////////////////////////////////////////////////////////

Camera::Camera(Space* ref, size_t image_height, size_t image_width,
			   size_t max_depth):
	manager_(ref->manager()),
	root_(ref->root()), ref_(ref), dim_r_(ref->dim_r()),
	image_height_(image_height), image_width_(image_width),
	image_size_(image_height * image_width), max_depth_(max_depth),
	render_data_(0, nullptr) {
	this->manager_->RegisterCamera_(this);
}

Camera::~Camera() {}

#///////////////////////////////////////////////////////////////////////////////

void Camera::RenderReady(size_t size) const {
	Camera_Render_pre_<<<1, 1>>>(this, size);
}

void Camera::Render(size_t block_pos_h, size_t block_pos_w, size_t block_size_h,
					size_t block_size_w) const {
	Camera_Render_main_<<<32, 1024>>>(this, block_pos_h, block_pos_w,
									  block_size_h, block_size_w);
}

RHO__glb void Camera_Render_(const Camera* camera) {
	for (size_t i(0); i != camera->render_data_.first; ++i)
		camera->render_data_.second[i].rendered = false;
}

RHO__glb void Camera_Render_pre_(const Camera* camera, size_t size) {
	camera->ref_->RefreshSelf();

	{
		const RBT<Object*>& object(camera->manager_->active_object());

		auto iter(object.begin());

		if (RHO__debug_flag) {
			for (auto end(object.end()); iter != end; ++iter)
				if (!(*iter)->Refresh())
					RHO__throw__local("ReadyForRendering error");
		} else {
			for (auto end(object.end()); iter != end; ++iter)
				(*iter)->Refresh();
		}
	}

	{
		const ComponentContainer& cmpt_cntr(camera->manager_->active_cmpt());

		camera->cmpt_collider_.Clear();
		camera->cmpt_collider_.Reserve(cmpt_cntr.size());

		camera->cmpt_light_.Clear();
		camera->cmpt_light_.Reserve(cmpt_cntr.size());

		auto iter(cmpt_cntr.begin());

		for (auto end(cmpt_cntr.end()); iter != end; ++iter) {
			if (!(*iter)->Refresh()) RHO__throw__local("Refresh error");

			switch ((*iter)->type) {
				case Component::Type::collider: {
					camera->cmpt_collider_.Push(
						static_cast<ComponentCollider*>(*iter));

					break;
				}
				case Component::Type::light: {
					camera->cmpt_light_.Push(
						static_cast<ComponentLight*>(*iter));

					break;
				}
			}
		}

		Sort(camera->cmpt_collider_.begin(), camera->cmpt_collider_.end(),
			 [](const ComponentCollider* x, const ComponentCollider* y) {
				 return x->domain()->Complexity() < y->domain()->Complexity();
			 });
	}

#///////////////////////////////////////////////////////////////////////////////

	camera->direct_f_.set_dim(3);
	camera->direct_h_.set_dim(3);
	camera->direct_w_.set_dim(3);

	Copy(camera->dim_r_, camera->direct_f_, camera->ref_->root_axis());
	Copy(camera->dim_r_, camera->direct_h_,
		 camera->ref_->root_axis() + RHO__max_dim);
	Copy(camera->dim_r_, camera->direct_w_,
		 camera->ref_->root_axis() + RHO__max_dim * 2);

	camera->direct_f_ -= camera->direct_h_;
	camera->direct_f_ -= camera->direct_w_;

	camera->direct_h_ *= Num(2) / camera->image_height_;
	camera->direct_w_ *= Num(2) / camera->image_width_;

#///////////////////////////////////////////////////////////////////////////////

	if (camera->render_data_.first < size) {
		camera->render_data_.first = size;
		Delete(camera->render_data_.second);
		camera->render_data_.second = Malloc<Camera::RenderData>(size);
	}

	// camera->RenderDataRefresh_(render_data);
}

#///////////////////////////////////////////////////////////////////////////////

RHO__glb void Camera_Render_main_(const Camera* camera,
								  const size_t block_pos_h,
								  const size_t block_pos_w,
								  const size_t block_size_h,
								  const size_t block_size_w) {
	const size_t thread_num(RHO__thread_num);
	const size_t thread_id(RHO__thread_index);
	const size_t block_size(block_size_h * block_size_w);

	// variable to store current trace data

	size_t render_index(thread_id);
	Camera::RenderData* render_data;

	Num dist_sq;
	Num d_dist;

	RayCastDataPair rcdp;
	Vector point[2];

	ComponentCollider* collider_a;
	ComponentCollider* collider_b;

	Material* material_a;
	Material* material_b;

	Texture::Data texture_data;

	Num3 transmittance;
	Num3 reflectance;
	Num3 difuss_reflectance;

	Tod tod;

	Vector reflection_vector;

	size_t task_size(0);
	cntr::BidirectionalNode task_node;

#define RHO__static_task_size 5

	Camera::Task static_task[RHO__static_task_size];
	// this task is to avoid using New<Camera::Task>()

	for (size_t i(0); i != RHO__static_task_size; ++i)
		task_node.PushNext(static_task + i);

	Camera::Task* task;
	Camera::Task* next_task;

	Vec temp;

#///////////////////////////////////////////////////////////////////////////////

	for (;; --task_size) {
		if (task_size) {
			// the current have not been done
			// we pop the task from pre-tracing
			task = static_cast<Camera::Task*>(task->prev);
		} else {
			// if then current pixel have been done
			// task_stack will be vacant
			// then we can process the next

			if (block_size <= render_index) { return; }

			++task_size;
			task = static_cast<Camera::Task*>(task_node.next);

			render_data = camera->render_data_.second + render_index;
			render_data->dist = 0;
			render_data->intensity[0] = 0;
			render_data->intensity[1] = 0;
			render_data->intensity[2] = 0;

			size_t i(render_index / block_size_w);
			size_t j(render_index - i * block_size_w);

			Vector::Copy(task->ray.origin, camera->ref_->root_origin());

			i += block_pos_h;
			j += block_pos_w;

#pragma unroll
			for (dim_t k(0); k != RHO__max_dim; ++k) {
				task->ray.direct[k] = camera->direct_f_[k] +
									  camera->direct_h_[k] * i +
									  camera->direct_w_[k] * j;
			}

			task->dist = 0;
			task->depth = 0;

			task->decay[0] = 1;
			task->decay[1] = 1;
			task->decay[2] = 1;

			render_index += thread_num;
		}

#///////////////////////////////////////////////////////////////////////////////

		// every point between ray.origin and the first hit point is
		// in the material a
		// every point between the first and second hit points is
		// in the material b

		task->ray.RayCastForRender(rcdp, camera->cmpt_collider_);

		if (!rcdp[0]) {
			/*goto function_head; */
			continue;
		}

		task->ray.point(point[0], rcdp[0]->t);

		if (rcdp[1]) { task->ray.point(point[1], rcdp[1]->t); }

#///////////////////////////////////////////////////////////////////////////////

		// calculate the dist fromt origin to point[0]
		// to get the transmittance through material a

		task->ray.point(temp, rcdp[0]->t / 2);

		material_a = (collider_a = camera->manager_->GetComponentCollider(temp))
						 ? collider_a->object()->material()
						 : camera->manager_->void_material();

		// get collider_a

		// if collider_a is exist
		// get material from its object

		// if not
		// get void material from manager

		// object's material are initialized to default material

		d_dist = abs(camera->dim_r(), task->ray.direct) * rcdp[0]->t;

		if (render_data->dist.eq<0>()) { render_data->dist = d_dist; }

		dist_sq = sq(task->dist += d_dist);

		task->decay[0] *= pow(material_a->transmittance[0], d_dist);
		task->decay[1] *= pow(material_a->transmittance[1], d_dist);
		task->decay[2] *= pow(material_a->transmittance[2], d_dist);

#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////

		rcdp[0]->domain->GetTodTan(tod.tan, rcdp[0], task->ray.direct);

#pragma unroll
		for (dim_t i(0); i != RHO__max_dim; ++i)
			tod.orth[i] = task->ray.direct[i] - tod.tan[i];

		texture_data =
			rcdp[0]->cmpt_collider->texture()->GetData(point[0], tod.tan);

#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////

		transmittance = texture_data.transmittance;
		reflectance = texture_data.reflectance;

		// transmission

		if (transmittance[0].ne<0>() || transmittance[1].ne<0>() ||
			transmittance[2].ne<0>()) {
			task->ray.point(temp, rcdp[1] ? ((rcdp[0]->t + rcdp[1]->t) / 2)
										  : (rcdp[0]->t + 1));

			material_b =
				(collider_b = camera->manager_->GetComponentCollider(temp))
					? collider_b->object()->material()
					: camera->manager_->void_material();

			if (material_b->transmittance[0].ne<0>() ||
				material_b->transmittance[1].ne<0>() ||
				material_b->transmittance[2].ne<0>()) {
				RefractionData refraction(rcdp[0], tod, material_a, material_b);

				if (refraction.transmittance.eq<0>()) {
					reflectance[0] += transmittance[0];
					reflectance[1] += transmittance[1];
					reflectance[2] += transmittance[2];

					transmittance[0] = 0;
					transmittance[1] = 0;
					transmittance[2] = 0;
				} else {
					Num n(1 - refraction.transmittance);
					Num3 next_decay;

					reflectance[0] += transmittance[0] * n;
					reflectance[1] += transmittance[1] * n;
					reflectance[2] += transmittance[2] * n;

					transmittance[0] *= refraction.transmittance;
					transmittance[1] *= refraction.transmittance;
					transmittance[2] *= refraction.transmittance;

					next_decay[0] = task->decay[0] * transmittance[0];
					next_decay[1] = task->decay[1] * transmittance[1];
					next_decay[2] = task->decay[2] * transmittance[2];

					// after (long long) judge
					// we push a task to task_stack

					// printf("reflection task add\n");

					if (task->depth < camera->max_depth_) {
						if (task->next == &task_node) {
							task->PushPrev(next_task = New<Camera::Task>());
						} else {
							cntr::BidirectionalNode::Swap(
								*task, *(next_task = static_cast<Camera::Task*>(
											 task->next)));
						}

						Vector::Copy(next_task->ray.origin, point[0]);
						line<RHO__max_dim>(next_task->ray.direct,
										   refraction.parallel_ratio, tod.tan,
										   tod.orth);
						next_task->dist = task->dist;
						next_task->depth = task->depth + 1;
						next_task->decay = Move(next_decay);

						++task_size;
					}
				}
			}
		}

#pragma unroll
		for (dim_t i(0); i != RHO__max_dim; ++i)
			reflection_vector[i] = tod.tan[i] - tod.orth[i];

		// reflection

		if (reflectance[0].ne<0>() || reflectance[1].ne<0>() ||
			reflectance[2].ne<0>()) {
			Num3 next_decay;
			next_decay[0] = task->decay[0] * reflectance[0];
			next_decay[1] = task->decay[1] * reflectance[1];
			next_decay[2] = task->decay[2] * reflectance[2];

			if (task->depth < camera->max_depth_) {
				// after (long long) judge
				// we push a task to task_stack

				if (task->next == &task_node) {
					task->PushPrev(next_task = New<Camera::Task>());
				} else {
					cntr::BidirectionalNode::Swap(
						*task,
						*(next_task = static_cast<Camera::Task*>(task->next)));
				}

				Vector::Copy(next_task->ray.origin, point[0]);
				Vector::Copy(next_task->ray.direct, reflection_vector);
				next_task->dist = task->dist;
				next_task->depth = task->depth + 1;
				next_task->decay = Move(next_decay);

				++task_size;
			}
		}

#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////

		// Bling Phong specular
		// check blocked

		{
			bool b(true);

			for (size_t i(0); i != 3; ++i) {
				difuss_reflectance[i] = 1 - transmittance[i] - reflectance[i];
				if ((difuss_reflectance[i]).le<0>()) {
					difuss_reflectance[i] = 0;
				} else {
					b = false;
				}
			}

			if (b) {
				// goto function_head;
				continue;
			}
		}

		for (size_t i(0); i != camera->cmpt_light_.size(); ++i) {
			// influence caused by position is processed in
			// ComponentLight::intensity point tod reflection_vector ray
			// pre_length

			// influence caused by texture or material is processed in Camera
			// material transmittence
			// texture reflectance
			// texture transmittance
			// refraction transmittance

			Num3 intensity(camera->cmpt_light_[i]->intensity(
				point[0], tod, camera->cmpt_collider_, reflection_vector,
				texture_data, task->ray, task->dist));

			render_data->intensity[0] += texture_data.color[0] / 255 *
										 intensity[0] * difuss_reflectance[0] *
										 task->decay[0];
			render_data->intensity[1] += texture_data.color[1] / 255 *
										 intensity[1] * difuss_reflectance[1] *
										 task->decay[1];
			render_data->intensity[2] += texture_data.color[2] / 255 *
										 intensity[2] * difuss_reflectance[2] *
										 task->decay[2];
		}
	}

	Camera::Task* n(static_cast<Camera::Task*>(task_node.next));
	Camera::Task* m;

	while (n != &task_node) {
		m = static_cast<Camera::Task*>(n->next);
		int k(n - static_task);
		if (!(0 < k && k < RHO__static_task_size)) { Delete(n); }
		n = m;
	}
}

void Camera::RenderDataRefresh_(RenderData* render_data) const {
	/*if (this->render_data_.first < this->size_) {
		Free(this->render_data_.second);
		this->render_data_.second =
			Malloc<RenderData>(this->size_);
	}*/

	this->render_data_.second = render_data;

	printf("render data alloc end\n");
}
}
