#include "define.cuh"
#include "Kernel.cuh"

namespace rho {

bool ComponentCollider::Material::Check() const {
	return this->refraction_index.ge<1>() &&

		   this->transmittance[0].ge<0>() && this->transmittance[1].ge<0>() &&
		   this->transmittance[2].ge<0>() &&

		   this->transmittance[0].le<1>() && this->transmittance[1].le<1>() &&
		   this->transmittance[2].le<1>();
}

void ComponentCollider::Material::SetDefault() {
	this->refraction_index = 1;
	this->transmittance[0] = 0;
	this->transmittance[1] = 0;
	this->transmittance[2] = 0;
}

void ComponentCollider::Material::Set(Num refration_index, Num transmittance_0,
									  Num transmittance_1,
									  Num transmittance_2) {
	this->refraction_index = refraction_index;
	this->transmittance[0] = transmittance_0;
	this->transmittance[1] = transmittance_1;
	this->transmittance[2] = transmittance_2;
}

#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////

const Domain* ComponentCollider::domain() const { return this->domain_; }
const Texture* ComponentCollider::texture() const { return this->texture_; }

ComponentCollider::Material& ComponentCollider::material() {
	return this->material_;
}
const ComponentCollider::Material& ComponentCollider::material() const {
	return this->material_;
}

ComponentCollider* ComponentCollider::set_domain(const Domain* domain) {
	this->domain_ = domain;
	return this;
}

ComponentCollider* ComponentCollider::set_texture(const Texture* texture) {
	this->texture_ = texture;
	return this;
}

#///////////////////////////////////////////////////////////////////////////////

ComponentCollider::ComponentCollider(Object* object, const Domain* domain,
									 Texture* texture):
	Component(Type::collider, object),
	domain_(domain), texture_(texture) {}

#///////////////////////////////////////////////////////////////////////////////

bool ComponentCollider::Refresh() const {
	return this->domain_->root() && this->domain_->Refresh() &&
		   this->texture_->Refresh();
}

#///////////////////////////////////////////////////////////////////////////////

bool ComponentCollider::Contain(const Num* point) const {
	return this->domain_->Contain(point);
}

#///////////////////////////////////////////////////////////////////////////////

RayCastData ComponentCollider::RayCast(const Ray& ray) const {
	RayCastData r(this->domain_->RayCast(ray));
	if (r) { r->cmpt_collider = const_cast<ComponentCollider*>(this); }
	return r;
}

bool ComponentCollider::RayCastFull(RayCastDataVector& rcdv,
									const Ray& ray) const {
	bool phase(this->domain_->RayCastFull(rcdv, ray));

	for (size_t i(0); i != rcdv.size(); ++i) {
		rcdv[i]->cmpt_collider = const_cast<ComponentCollider*>(this);
	}

	return phase;
}

void ComponentCollider::RayCastForRender(RayCastDataPair& rcdp,
										 const Ray& ray) const {
	this->domain_->RayCastForRender(rcdp, const_cast<ComponentCollider*>(this),
									ray);
}

}