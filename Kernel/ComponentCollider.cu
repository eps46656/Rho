#include"define.cuh"
#include"Kernel.cuh"

namespace rho {

const Domain* ComponentCollider::domain()const
{ return this->domain_; }

Texture* ComponentCollider::texture()const
{ return this->texture_; }

ComponentCollider* ComponentCollider::
set_domain(const Domain* domain) {
	this->domain_ = domain;
	return this;
}

ComponentCollider* ComponentCollider::
set_texture(Texture* texture) {
	this->texture_ = texture;
	return this;
}

#////////////////////////////////////////////////

ComponentCollider::ComponentCollider(
	Object* object, const Domain* domain, Texture* texture) :

	Component(Type::collider, object),
	domain_(domain),
	texture_(texture ? texture :
			 this->manager()->default_texture()) {}

#////////////////////////////////////////////////

bool ComponentCollider::Refresh()const {
	return this->root_ == this->domain_->root() &&
		this->domain_->Refresh() && this->texture_->Refresh();
}

#////////////////////////////////////////////////

bool ComponentCollider::Contain(const Vector& point)const
{ return this->domain_->Contain(point); }

#////////////////////////////////////////////////

RayCastData ComponentCollider::
RayCast(const Ray& ray)const {
	RayCastData r(this->domain_->RayCast(ray));

	if (r) {
		r->cmpt_collider =
			const_cast<ComponentCollider*>(this);
	}

	return r;
}

bool ComponentCollider::
RayCastFull(RayCastDataVector& rcdv, const Ray& ray)const {
	bool phase(this->domain_->RayCastFull(rcdv, ray));

	for (size_t i(0); i != rcdv.size(); ++i) {
		rcdv[i]->cmpt_collider =
			const_cast<ComponentCollider*>(this);
	}

	return phase;
}

void ComponentCollider::RayCastForRender(
	RayCastDataPair& rcdp, const Ray& ray)const {

	this->domain_->RayCastForRender(
		rcdp,
		const_cast<ComponentCollider*>(this),
		ray);
}

}