#include "define.cuh"
#include "Kernel.cuh"

#define RHO__throw__local(desc) RHO__throw(Tod, __func__, desc)

namespace rho {

void Tod::TanMatrix(dim_t dim_s, dim_t dim_r, Num* dst, const Num* src) {
	if (dim_s) {
		NumMatrix temp;

		Copy<RHO__max_dim_sq>(temp, src);

		if (!Complement(dim_s, dim_r, temp))
			RHO__throw__local("linear dependence error");

		NumMatrix temp_i;
		inverse(dim_r, temp_i, temp);
		dot(dim_r, dim_s, dim_r, dst, temp_i, temp);
	} else {
		Matrix::identity(dst, dim_r);
	}
}

void Tod::OrthMatrix(dim_t dim_s, dim_t dim_r, Num* dst, const Num* src) {
	if (dim_s) {
		NumMatrix temp;

		Copy<RHO__max_dim_sq>(temp, src);

		if (!Complement(dim_s, dim_r, temp))
			RHO__throw__local("linear dependence error");

		NumMatrix temp_i;
		inverse(dim_r, temp_i, temp);
		dot(dim_r, dim_r - dim_s, dim_r, dst, temp_i + dim_s,
			temp + RHO__max_dim * dim_s);
	} else {
		for (dim_t i(0); i != dim_r; ++i) {
			for (dim_t j(0); j != RHO__max_dim; ++j)
				dst[RHO__max_dim * i + j] = 0;
		}
	}
}

void Tod::TanMatrix(Matrix& axis) {
	TanMatrix(axis.col_dim(), axis.row_dim(), axis, axis);
	axis.set_col_dim(axis.row_dim());
}

void Tod::OrthMatrix(Matrix& axis) {
	OrthMatrix(axis.col_dim(), axis.row_dim(), axis, axis);
	axis.set_col_dim(axis.row_dim());
}

Vector Tod::Tan(const Vector& vector, const Vector& axis) {
	RHO__debug_if(vector.dim() != axis.dim()) RHO__throw__local("dim error");

	Vector r(vector.dim());
	Tan(vector.dim(), r, vector, axis);
	return r;
}

Vector Tod::Orth(const Vector& vector, const Vector& axis) {
	RHO__debug_if(vector.dim() != axis.dim()) RHO__throw__local("dim error");

	Vector r(vector.dim());
	Orth(vector.dim(), r, vector, axis);
	return r;
}

void Tod::Tan(dim_t dim_r, Num* dst, const Num* vector, const Num* axis) {
	RHO__debug_if(!dim_r) RHO__throw__local("dim error");

	Num dot(0);
	Num axis_sq(0);

	for (dim_t i(0); i != dim_r; ++i) {
		dot += vector[i] * axis[i];
		axis_sq += sq(axis[i]);
	}

	if (axis_sq.ne<0>()) { dot /= axis_sq; }

	for (dim_t i(0); i != dim_r; ++i) { dst[i] = axis[i] * dot; }
}

void Tod::Orth(dim_t dim_r, Num* dst, const Num* vector, const Num* axis) {
	RHO__debug_if(!dim_r) RHO__throw__local("dim error");

	Num dot(0);
	Num axis_sq(0);

	for (dim_t i(0); i != dim_r; ++i) {
		dot += vector[i] * axis[i];
		axis_sq += sq(axis[i]);
	}

	if (axis_sq.ne<0>()) { dot /= axis_sq; }

	for (dim_t i(0); i != dim_r; ++i) { dst[i] = vector[i] - axis[i] * dot; }
}

}