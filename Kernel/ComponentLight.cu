#include "define.cuh"
#include "Kernel.cuh"

namespace rho {

const Num3& ComponentLight::intensity() const { return this->intensity_; }

#///////////////////////////////////////////////////////////////////////////////

ComponentLight::ComponentLight(Object* object):
	Component(Type::light, object) {}

ComponentLight::ComponentLight(Object* object, const Num3& intensity):
	Component(Type::light, object), intensity_(intensity) {}

}