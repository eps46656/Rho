#include "define.cuh"
#include "Kernel.cuh"

namespace rho {

Manager* Domain::manager() const { return this->manager_; }
Space* Domain::root() const { return this->root_; }

dim_t Domain::dim_r() const { return this->root_->dim_r(); }

#///////////////////////////////////////////////////////////////////////////////

Domain::Domain(Space* root): manager_(root->manager()), root_(root) {
	RHO__debug_if(root->parent()) RHO__throw(Domain, __func__, "root error");
}

Domain::~Domain() {}

#///////////////////////////////////////////////////////////////////////////////

bool Domain::RayCastB(const Ray& ray) const {
	auto rcd(this->RayCast(ray));
	return rcd && rcd->t.lt<1>();
}

RayCastData Domain::RayCast(const Ray& ray) const {
	RayCastDataVector rcdv;
	this->RayCastFull(rcdv, ray);
	return rcdv.empty() ? RayCastData() : Move(rcdv[0]);
}

void Domain::RayCastForRender(RayCastDataPair& rcdp,
							  ComponentCollider* cmpt_collider,
							  const Ray& ray) const {
	RayCastDataVector rcdv;
	this->RayCastFull(rcdv, ray);

	if (rcdv.empty()) { return; }

	if (rcdv.size() == 1) {
		if (rcdv[0] < rcdp[0]) {
			rcdp[1] = Move(rcdp[0]);
			rcdp[0] = Move(rcdv[0]);

			rcdp[0]->cmpt_collider = cmpt_collider;
		} else if (rcdv[0] < rcdp[1]) {
			rcdp[1] = Move(rcdv[0]);

			rcdp[1]->cmpt_collider = cmpt_collider;
		}
	} else {
		if (rcdv[1] < rcdp[0]) {
			rcdp[0] = Move(rcdv[0]);
			rcdp[1] = Move(rcdv[1]);

			rcdp[0]->cmpt_collider = cmpt_collider;
			rcdp[1]->cmpt_collider = cmpt_collider;
		} else if (rcdv[0] < rcdp[0]) {
			rcdp[1] = Move(rcdp[0]);
			rcdp[0] = Move(rcdv[0]);

			rcdp[0]->cmpt_collider = cmpt_collider;
		} else if (rcdv[0] < rcdp[1]) {
			rcdp[1] = Move(rcdv[0]);

			rcdp[1]->cmpt_collider = cmpt_collider;
		}
	}
}

}