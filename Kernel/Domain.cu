#include "define.cuh"
#include "Kernel.cuh"

namespace rho {

dim_t Domain::dim_r() const { return this->root()->dim_r(); }

#///////////////////////////////////////////////////////////////////////////////

Domain::Domain(Type type_): type(type_) {}

Domain::~Domain() {}

#///////////////////////////////////////////////////////////////////////////////

bool Domain::RayCastB(const Ray& ray) const {
	auto rcd(this->RayCast(ray));
	return rcd && rcd->t.lt<1>();
}

RayCastData Domain::RayCast(const Ray& ray) const {
	RayCastDataVector rcdv;
	this->RayCastFull(rcdv, ray);
	return rcdv.empty() ? RayCastData() : Move(rcdv[0]);
}

void Domain::RayCastForRender(RayCastDataPair& rcdp,
							  ComponentCollider* cmpt_collider,
							  const Ray& ray) const {
	RayCastDataVector rcdv;
	this->RayCastFull(rcdv, ray);

	if (rcdv.empty()) { return; }

	if (rcdv.size() == 1) {
		if (rcdv[0] < rcdp[0]) {
			rcdp[1] = Move(rcdp[0]);
			rcdp[0] = Move(rcdv[0]);

			rcdp[0]->cmpt_collider = cmpt_collider;
		} else if (rcdv[0] < rcdp[1]) {
			rcdp[1] = Move(rcdv[0]);

			rcdp[1]->cmpt_collider = cmpt_collider;
		}
	} else {
		if (rcdv[1] < rcdp[0]) {
			rcdp[0] = Move(rcdv[0]);
			rcdp[1] = Move(rcdv[1]);

			rcdp[0]->cmpt_collider = cmpt_collider;
			rcdp[1]->cmpt_collider = cmpt_collider;
		} else if (rcdv[0] < rcdp[0]) {
			rcdp[1] = Move(rcdp[0]);
			rcdp[0] = Move(rcdv[0]);

			rcdp[0]->cmpt_collider = cmpt_collider;
		} else if (rcdv[0] < rcdp[1]) {
			rcdp[1] = Move(rcdv[0]);

			rcdp[1]->cmpt_collider = cmpt_collider;
		}
	}
}

}