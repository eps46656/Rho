#include "hip/hip_runtime.h"
#include "define.cuh"
#include "Kernel.cuh"

namespace rho {

const Space* ComponentLightAmbience::ref() const { return this->ref_; }
const Space* ComponentLightAmbience::root() const { return this->ref_->root(); }

dim_t ComponentLightAmbience::root_dim() const {
	return this->ref_->root_dim();
}

Num3& ComponentLightAmbience::intensity() { return this->intensity_; }
const Num3& ComponentLightAmbience::intensity() const {
	return this->intensity_;
}

ComponentLightAmbience* ComponentLightAmbience::set_ref(const Space* ref) {
	this->ref_ = ref;
	return this;
}

#///////////////////////////////////////////////////////////////////////////////

ComponentLightAmbience::ComponentLightAmbience(const Space* ref,
											   const Num3& intensity):
	ref_(ref),
	intensity_(intensity) {}

#///////////////////////////////////////////////////////////////////////////////

bool ComponentLightAmbience::Refresh() const {
	return this->intensity_[0].ge<0>() && this->intensity_[1].ge<0>() &&
		   this->intensity_[2].ge<0>();
}

#///////////////////////////////////////////////////////////////////////////////

Num3 ComponentLightAmbience::intensity(
	const Num* root_point, const Tod& tod,
	const cntr::Vector<const ComponentCollider*>& cmpt_collider,
	const Num* reflection_vector, const Texture::Data& texture_data, Ray& ray,
	Num pre_dist) const {
	Num a(pow(pre_dist, 0.1));
	Num3 r;

	r[0] = this->intensity_[0] / a;
	r[1] = this->intensity_[1] / a;
	r[2] = this->intensity_[2] / a;

	return r;
}

}