#include "hip/hip_runtime.h"
#include "define.cuh"
#include "ComponentLightAmbience.cuh"

namespace rho {

Num3& ComponentLightAmbience::intensity() { return this->intensity_; }
const Num3& ComponentLightAmbience::intensity() const {
	return this->intensity_;
}

#///////////////////////////////////////////////////////////////////////////////

ComponentLightAmbience::ComponentLightAmbience(Object* object,
											   const Num3& intensity):
	ComponentLight(object),
	intensity_(intensity) {}

#///////////////////////////////////////////////////////////////////////////////

bool ComponentLightAmbience::Refresh() const {
	return this->intensity_[0].ge<0>() && this->intensity_[1].ge<0>() &&
		   this->intensity_[2].ge<0>();
}

#///////////////////////////////////////////////////////////////////////////////

Num3 ComponentLightAmbience::intensity(
	const Num* root_point, const Tod& tod,
	const cntr::Vector<ComponentCollider*>& cmpt_collider,
	const Num* reflection_vector, const Texture::Data& texture_data, Ray& ray,
	Num pre_distance) const {
	Num a(pow(pre_distance, 0.1));
	Num3 r;

	r[0] = this->intensity_[0] / a;
	r[1] = this->intensity_[1] / a;
	r[2] = this->intensity_[2] / a;

	return r;
}

}