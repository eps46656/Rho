#include "hip/hip_runtime.h"
#include "define.cuh"
#include "ComponentLightAmbience.cuh"

namespace rho {

ComponentLightAmbience::ComponentLightAmbience(Object* object,
											   const Num3& intensity):

	ComponentLight(object, intensity) {}

#///////////////////////////////////////////////////////////////////////////////

bool ComponentLightAmbience::Refresh() const {
	return this->intensity_[0].ge<0>() && this->intensity_[1].ge<0>() &&
		   this->intensity_[2].ge<0>();
}

#///////////////////////////////////////////////////////////////////////////////

Num3 ComponentLightAmbience::intensity(
	const Vector& root_point, const Tod& tod,
	const cntr::Vector<ComponentCollider*>& cmpt_collider,
	const Vector& reflection_vector, const Texture::Data& texture_data,
	Ray& ray, Num pre_distance) const {
	Num a(pow(pre_distance, 0.1));
	Num3 r;

	r[0] = this->intensity_[0] / a;
	r[1] = this->intensity_[1] / a;
	r[2] = this->intensity_[2] / a;

	return r;
}

}