#include "define.cuh"
#include "Kernel.cuh"

namespace rho {

Space* DomainSole::ref() const { return this->ref_; }
void DomainSole::set_ref(Space* ref) { this->ref_ = ref; }

dim_t DomainSole::dim_s() const { return this->ref_->dim_s(); }
dim_t DomainSole::dim_cr() const { return this->ref_->dim_cr(); }

#///////////////////////////////////////////////////////////////////////////////

DomainSole::DomainSole(Space* ref): Domain(ref->root()), ref_(ref) {}

#///////////////////////////////////////////////////////////////////////////////

bool DomainSole::Contain(const Num* root_point) const {
	if (this->root() == this->ref_) { return this->Contain_s(root_point); }

	NumVector point;
	this->ref_->MapPointFromRoot_rr(point, root_point);

	for (dim_t i(this->dim_s()); i != this->dim_r(); ++i) {
		if (point[i].ne<0>()) { return false; }
	}

	return this->Contain_s(point);
}

}