
#include <hip/hip_runtime.h>
#if false

#include "../define.cuh"
#include "Scene.cuh"

namespace rho {

Scene::Scene() {}

#///////////////////////////////////////////////////////////////////////////////

Space* Scene::root() const { return this->root_; }

RBT<Space*> Scene::space() const { return this->space_; }
RBT<Object*> Scene::object() const { return this->object_; }

#///////////////////////////////////////////////////////////////////////////////

void Scene::AddSpace_(Space* space) { this->space_.Insert(space); }

void Scene::AddObject_(Object* object) {
	this->object_.Insert(object);
	if (object.axtive()) { this->active_object_.Insert(object); }
}

void Manager::AddCmpt_(Component* cmpt) {
	this->cmpt_.Insert(cmpt);
	this->active_cmpt_.Insert(cmpt);

	cmpt->priority_ = this->priority_vector_.size();
	this->priority_vector_.Push(cmpt);
}

#///////////////////////////////////////////////////////////////////////////////

void Scene::SubSpace_(Space* space) { this->space_.FindErase(space); }

void Scene::SubObject_(Object* object) {
	this->object_.Erase(this->object_.Find(object));
	this->active_object_.FindDeleteErase(object);
}

}

#endif