#include "hip/hip_runtime.h"
#include "Calculus.cuh"
#include "define.cuh"

#define RHO__throw__local(desc) RHO__throw(Vector, __func__, desc)

#define RHO__full_loop for (dim_t i(0); i != RHO__max_dim; ++i)

namespace rho {

dim_t Vector::dim() const { return this->dim_; }

void Vector::set_dim(dim_t dim) {
	RHO__debug_if(RHO__max_dim < dim) RHO__throw__local("dim error");
	this->dim_ = dim;
}

#///////////////////////////////////////////////////////////////////////////////

Vector::Vector(): dim_(0) {}
Vector::Vector(dim_t dim): dim_(dim) {}
Vector::Vector(const Vector& vector): dim_(vector.dim_) { Copy(*this, vector); }

#///////////////////////////////////////////////////////////////////////////////

Vector& Vector::operator=(const Vector& vector) {
	if (this == &vector) { return *this; }
	this->dim_ = vector.dim_;
	Copy(*this, vector);
	return *this;
}

#///////////////////////////////////////////////////////////////////////////////

Num& Vector::operator[](dim_t index) {
	RHO__debug_if(RHO__max_dim < index) RHO__throw__local("index error");
	return this->data[index];
}

const Num& Vector::operator[](dim_t index) const {
	RHO__debug_if(RHO__max_dim < index) RHO__throw__local("index error");
	return this->data[index];
}

Vector Vector::slice(dim_t min_index, dim_t max_index) const& {
	RHO__debug_if(this->dim_ < min_index || this->dim_ < max_index ||
				  max_index < min_index) {
		RHO__throw__local("index error");
	}

	Vector r;
	slice(r, *this, min_index, max_index);
	return r;
}

Vector&& Vector::slice(dim_t min_index, dim_t max_index) && {
	slice(*this, *this, min_index, max_index);
	return Move(*this);
}

void Vector::slice(Vector& dst, const Vector& src, dim_t min_index,
				   dim_t max_index) {
	RHO__debug_if(src.dim_ < min_index || src.dim_ < max_index ||
				  min_index < min_index) {
		RHO__throw__local("index error");
	}

	rho::Copy(dst.dim_ = max_index - min_index, dst, src + min_index);
}

#////////////////////////////////////////////////

Vector& Vector::operator+() & { return *this; }
Vector&& Vector::operator+() && { return Move(*this); }
const Vector& Vector::operator+() const& { return *this; }

Vector Vector::operator-() const& {
	Vector r(this->dim_);

#pragma unroll
	RHO__full_loop { r[i] = -(*this)[i]; }

	return r;
}

Vector&& Vector::operator-() && {
#pragma unroll
	RHO__full_loop(*this)[i] = -(*this)[i];

	return Move(*this);
}

#////////////////////////////////////////////////

bool operator==(const Vector& x, const Vector& y) {
	return (x.dim() == y.dim() && Equal(x.dim(), x, y));
}

bool operator!=(const Vector& x, const Vector& y) { return !(x == y); }

#////////////////////////////////////////////////

Vector operator+(const Vector& x, const Vector& y) {
	RHO__debug_if(x.dim() != y.dim()) RHO__throw__local("dim error");

	Vector r(x.dim());

#pragma unroll
	RHO__full_loop { r[i] = x[i] + y[i]; }

	return r;
}

Vector&& operator+(const Vector& x, Vector&& y) { return Move(y += x); }

Vector&& operator+(Vector&& x, const Vector& y) { return Move(x += y); }

Vector&& operator+(Vector&& x, Vector&& y) { return Move(x += y); }

#////////////////////////////////////////////////

Vector operator-(const Vector& x, const Vector& y) {
	RHO__debug_if(x.dim() != y.dim()) RHO__throw__local("dim error");

	Vector r(x.dim());

#pragma unroll
	RHO__full_loop { r[i] = x[i] - y[i]; }

	return r;
}

Vector&& operator-(const Vector& x, Vector&& y) {
	RHO__debug_if(x.dim() != y.dim()) RHO__throw__local("dim error");

#pragma unroll
	RHO__full_loop { y[i] = x[i] - y[i]; }

	return Move(y);
}

Vector&& operator-(Vector&& x, const Vector& y) { return Move(x -= y); }

Vector&& operator-(Vector&& x, Vector&& y) { return Move(x -= y); }

#////////////////////////////////////////////////

Vector operator*(Num num, const Vector& vector) { return vector * num; }

Vector operator*(const Vector& vector, Num num) {
	Vector r(vector.dim());

#pragma unroll
	RHO__full_loop { r[i] = vector[i] * num; }

	return r;
}

Num operator*(const Vector& x, const Vector& y) {
	RHO__debug_if(x.dim() != y.dim()) RHO__throw__local("dim error");

	Num r(0);
	dot(x.dim(), r, x, y);

	return r;
}

Vector&& operator*(Num num, Vector&& vector) { return Move(vector *= num); }

Vector&& operator*(Vector&& vector, Num num) { return Move(vector *= num); }

#///////////////////////////////////////////////////////////////////////////////

Vector operator/(const Vector& vector, Num num) {
	Vector r(vector.dim());

#pragma unroll
	RHO__full_loop { r[i] = vector[i] / num; }

	return r;
}

Vector&& operator/(Vector&& vector, Num num) { return Move(vector /= num); }

#///////////////////////////////////////////////////////////////////////////////

Vector& Vector::operator+=(const Vector& vector) & {
	RHO__debug_if(this->dim_ != vector.dim_) { RHO__throw__local("dim error"); }
	iadd(*this, vector);
	return *this;
}

Vector& Vector::operator-=(const Vector& vector) & {
	RHO__debug_if(this->dim_ != vector.dim_) { RHO__throw__local("dim error"); }
	isub(*this, vector);
	return *this;
}

Vector& Vector::operator*=(Num num) & {
	imul(*this, num);
	return *this;
}

Vector& Vector::operator/=(Num num) & {
	idiv(*this, num);
	return *this;
}

#///////////////////////////////////////////////////////////////////////////////

Print operator<<(Print p, const Vector& vector) {
	if (!vector.dim()) { Print() << "( void vector )\n"; }

	Print() << "( ", vector[0];

	for (dim_t i(1); i != vector.dim(); ++i) { Print() << ", ", vector[i]; }

	return Print() << " )\n";
}

#///////////////////////////////////////////////////////////////////////////////

bool Vector::is_zero() const {
	for (dim_t i(0); i != this->dim_; ++i) {
		if ((*this)[i].ne<0>()) { return false; }
	}

	return true;
}

#///////////////////////////////////////////////////////////////////////////////

bool is_zero(dim_t dim, const Num* src) {
	for (dim_t i(0); i != dim; ++i) {
		if (src[i].ne<0>()) { return false; }
	}

	return true;
}

void Vector::set_length(Num length) {
	Num l(sqrt(sq(this->dim(), *this)));

	RHO__debug_if(l.eq<0>()) RHO__throw__local("zero div");

	Num a(length / l);

#pragma unroll
	RHO__full_loop { (*this)[i] *= a; }
}

void Vector::set_sq(Num sq) { this->set_length(sqrt(sq)); }

#///////////////////////////////////////////////////////////////////////////////

Num sq(const Vector& vector) { return sq(vector.dim(), vector); }

Num abs(const Vector& vector) { return sqrt(sq(vector.dim(), vector)); }

Num dist(const Vector& x, const Vector& y) { return sqrt(dist_sq(x, y)); }

Num dist_sq(const Vector& x, const Vector& y) {
	RHO__debug_if(x.dim() != y.dim()) RHO__throw__local("dim error");

	Num r(0);
	dist_sq(x.dim(), r, x, y);
	return r;
}

#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////

void Vector::add(Num* dst, const Num* x, const Num* y) {
#pragma unroll
	RHO__full_loop { dst[i] = x[i] + y[i]; }
}

void Vector::sub(Num* dst, const Num* x, const Num* y) {
#pragma unroll
	RHO__full_loop { dst[i] = x[i] + y[i]; }
}

void Vector::mul(Num* dst, const Num* x, const Num& t) {
#pragma unroll
	RHO__full_loop { dst[i] = x[i] * t; }
}

void Vector::sub(Num* dst, const Num* x, const Num& t) { mul(dst, x, 1 / t); }

void Vector::iadd(Num* dst, const Num* x) {
#pragma unroll
	RHO__full_loop { dst[i] += x[i]; }
}

void Vector::isub(Num* dst, const Num* x) {
#pragma unroll
	RHO__full_loop { dst[i] -= x[i]; }
}

void Vector::imul(Num* dst, Num t) {
#pragma unroll
	RHO__full_loop { dst[i] *= t; }
}

void Vector::idiv(Num* dst, Num t) { imul(dst, 1 / t); }

#///////////////////////////////////////////////////////////////////////////////

Num angle_sin(const Vector& x, const Vector& y) {
	return sqrt(angle_sin_sq(x, y));
}

Num angle_sin_sq(const Vector& x, const Vector& y) {
	return 1 - angle_cos_sq(x, y);
}

Num angle_cos(const Vector& x, const Vector& y) {
	RHO__debug_if(x.dim() != y.dim()) { RHO__throw__local("dim error"); }
	return angle_cos(x.dim(), x, y);
}

Num angle_cos_sq(const Vector& x, const Vector& y) {
	RHO__debug_if(x.dim() != y.dim()) { RHO__throw__local("dim error"); }
	return angle_cos_sq(x.dim(), x, y);
}

Num angle_sin(dim_t dim, const Num* x, const Num* y) {
	return sqrt(angle_sin_sq(dim, x, y));
}

Num angle_cos(dim_t dim, const Num* x, const Num* y) {
	Num x_sq, dot, y_sq;
	sq__dot__sq(dim, x_sq, dot, y_sq, x, y);

	RHO__debug_if(x_sq.eq<0>() || y_sq.eq<0>()) RHO__throw__local("zero div");

	return dot / sqrt(x_sq * y_sq);
}

Num angle_sin_sq(dim_t dim, const Num* x, const Num* y) {
	return 1 - angle_cos_sq(dim, x, y);
}

Num angle_cos_sq(dim_t dim, const Num* x, const Num* y) {
	Num x_sq, dot, y_sq;
	sq__dot__sq(dim, x_sq, dot, y_sq, x, y);

	RHO__debug_if(x_sq.eq<0>() || y_sq.eq<0>()) RHO__throw__local("zero div");

	return sq(dot) / (x_sq * y_sq);
}

#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////

Num sq(dim_t dim, const Num* src) {
	RHO__debug_if(!dim) RHO__throw(, __func__, "dim error");

	if (dim == 3) { return sq(src[0]) + sq(src[1]) + sq(src[2]); }

	Num r(0);
	for (dim_t i(0); i != dim; ++i) { r += sq(src[i]); }
	return r;
}

Num abs(dim_t dim, const Num* src) { return sqrt(sq(dim, src)); }

Num dot(dim_t dim, const Num* x, const Num* y) {
	Num r(0);
	dot(dim, r, x, y);
	return r;
}

void dot(dim_t dim, Num& dst, const Num* x, const Num* y) {
	RHO__debug_if(!dim) RHO__throw(, __func__, "dim error");

	if (dim == 3) {
		dst += x[0] * y[0] + x[1] * y[1] + x[2] * y[2];
		return;
	}

	for (dim_t i(0); i != dim; ++i) { dst += x[i] * y[i]; }
}

Num dist_sq(dim_t dim, const Num* x, const Num* y) {
	Num r(0);
	dist_sq(dim, r, x, y);
	return r;
}

void dist_sq(dim_t dim, Num& dst, const Num* x, const Num* y) {
	RHO__debug_if(!dim) RHO__throw(, __func__, "dim error");

	for (dim_t i(0); i != dim; ++i) { dst += sq(x[i] - y[i]); }
}

void sq__dot__sq(dim_t dim, Num& dst_x_sq, Num& dst_dot, Num& dst_y_sq,
				 const Num* x, const Num* y) {
	RHO__debug_if(!dim) RHO__throw(, __func__, "dim error");

	dst_x_sq = sq(x[0]);
	dst_dot = x[0] * y[0];
	dst_y_sq = sq(y[0]);

	for (dim_t i(1); i != dim; ++i) {
		dst_x_sq += sq(x[i]);
		dst_dot += x[i] * y[i];
		dst_y_sq += sq(y[i]);
	}
}

}