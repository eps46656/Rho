#include "define.cuh"
#include "Calculus.cuh"

#define RHO__det_2(x0, x1, x2, x3) ((x0) * (x3) - (x1) * (x2))

#define RHO__det_3(x00, x01, x02, x10, x11, x12, x20, x21, x22)                \
	((x00)*RHO__det_2(x11, x12, x21, x22) -                                    \
	 (x01)*RHO__det_2(x10, x12, x20, x22) +                                    \
	 (x02)*RHO__det_2(x10, x11, x20, x21))

#define RHO__det_4(x00, x01, x02, x03, x10, x11, x12, x13, x20, x21, x22, x23, \
				   x30, x31, x32, x33)                                         \
	((x00)*RHO__det_3(x11, x12, x13, x21, x22, x23, x31, x32, x33) -           \
	 (x01)*RHO__det_3(x10, x12, x13, x20, x22, x23, x30, x32, x33) +           \
	 (x02)*RHO__det_3(x10, x11, x13, x20, x21, x23, x30, x31, x33) -           \
	 (x02)*RHO__det_3(x10, x11, x12, x20, x21, x22, x30, x31, x32))

#///////////////////////////////////////////////////////////////////////////////

#define RHO__cross_3_0(x00, x01, x02, x10, x11, x12)                           \
	(RHO__det_2(x01, x02, x11, x12))

#define RHO__cross_3_1(x00, x01, x02, x10, x11, x12)                           \
	(-RHO__det_2(x00, x02, x10, x12))

#define RHO__cross_3_2(x00, x01, x02, x10, x11, x12)                           \
	(RHO__det_2(x00, x01, x10, x11))

#///////////////////////////////////////////////////////////////////////////////

#define RHO__cross_4_0(x00, x01, x02, x03, x10, x11, x12, x13, x20, x21, x22,  \
					   x23)                                                    \
	(RHO__det_3(x01, x02, x03, x11, x12, x13, x21, x22, x23))

#define RHO__cross_4_1(x00, x01, x02, x03, x10, x11, x12, x13, x20, x21, x22,  \
					   x23)                                                    \
	(-RHO__det_3(x00, x02, x03, x10, x12, x13, x20, x22, x23))

#define RHO__cross_4_2(x00, x01, x02, x03, x10, x11, x12, x13, x20, x21, x22,  \
					   x23)                                                    \
	(RHO__det_3(x00, x01, x03, x10, x11, x13, x20, x21, x23))

#define RHO__cross_4_3(x00, x01, x02, x03, x10, x11, x12, x13, x20, x21, x22,  \
					   x23)                                                    \
	(-RHO__det_3(x00, x01, x02, x10, x11, x12, x20, x21, x22))

#///////////////////////////////////////////////////////////////////////////////

#define RHO__throw__local(desc) RHO__throw(Matrix, __func__, desc)
#define RHO__full_loop for (dim_t i(0); i != RHO__max_dim_sq; ++i)

namespace rho {

dim_t Matrix::col_dim() const { return this->col_dim_; }
dim_t Matrix::row_dim() const { return this->row_dim_; }

void Matrix::set_col_dim(dim_t col_dim) { this->col_dim_ = col_dim; }
void Matrix::set_row_dim(dim_t row_dim) { this->row_dim_ = row_dim; }

void Matrix::set_dim(dim_t col_dim, dim_t row_dim) {
	RHO__debug_if(RHO__max_dim < col_dim || RHO__max_dim < row_dim) {
		RHO__throw__local("capacity error");
	}

	this->col_dim_ = col_dim;
	this->row_dim_ = row_dim;
}

#///////////////////////////////////////////////////////////////////////////////

Matrix::Matrix(): col_dim_(0), row_dim_(0) {}

Matrix::Matrix(dim_t col_dim, dim_t row_dim):
	col_dim_(col_dim), row_dim_(row_dim) {}

Matrix::Matrix(const Matrix& matrix):
	col_dim_(matrix.col_dim_), row_dim_(matrix.row_dim_) {
	Copy(*this, matrix);
}

#///////////////////////////////////////////////////////////////////////////////

void Matrix::identity(Num* dst) {
#pragma unroll
	for (dim_t i(1); i != RHO__max_dim; ++i) {
		*dst = 1;
		++dst;

#pragma unroll
		for (dim_t j(0); j != RHO__max_dim; ++j, ++dst) { *dst = 0; }
	}

	*dst = 1;
}

void Matrix::identity(Num* dst, dim_t dim) {
	for (dim_t i(1); i != dim; ++i) {
		*dst = 1;
		++dst;

		for (dim_t j(0); j != RHO__max_dim; ++j, ++dst) { *dst = 0; }
	}

	*dst = 1;
}

void Matrix::identity(Matrix& dst, dim_t dim) {
	RHO__debug_if(RHO__max_dim < dim) { RHO__throw__local("dim error"); }

	dst.col_dim_ = dst.row_dim_ = dim;

	for (dim_t i(0); i != dim; ++i) {
		for (dim_t j(0); j != dim; ++j) {
			dst[RHO__max_dim * i + j] = i == j ? 1 : 0;
		}
	}
}

#///////////////////////////////////////////////////////////////////////////////

Matrix& Matrix::operator=(const Matrix& matrix) {
	if (this == &matrix) { return *this; }
	this->col_dim_ = matrix.col_dim_;
	this->row_dim_ = matrix.row_dim_;
	Copy(*this, matrix);
	return *this;
}

#///////////////////////////////////////////////////////////////////////////////

Num& Matrix::operator[](dim_t index) {
	RHO__debug_if(RHO__max_dim_sq < index) { RHO__throw__local("index error"); }

	return this->data[index];
}

const Num& Matrix::operator[](dim_t index) const {
	RHO__debug_if(RHO__max_dim_sq < index) { RHO__throw__local("index error"); }

	return this->data[index];
}

Num& Matrix::get(dim_t col_index, dim_t row_index) {
	RHO__debug_if(this->col_dim_ <= col_index || this->row_dim_ <= row_index) {
		RHO__throw__local("index error");
	}

	return (*this)[RHO__max_dim * col_index + row_index];
}

const Num& Matrix::get(dim_t col_index, dim_t row_index) const {
	RHO__debug_if(this->col_dim_ <= col_index || this->row_dim_ <= row_index) {
		RHO__throw__local("index error");
	}

	return (*this)[RHO__max_dim * col_index + row_index];
}

Matrix Matrix::slice(dim_t max_col_index, dim_t max_row_index) const& {
	Matrix r;
	slice(r, *this, max_col_index, max_row_index);
	return r;
}

Matrix&& Matrix::slice(dim_t max_col_index, dim_t max_row_index) && {
	slice(*this, *this, max_col_index, max_row_index);
	return Move(*this);
}

Matrix Matrix::slice(dim_t min_col_index, dim_t min_row_index,
					 dim_t max_col_index, dim_t max_row_index) const& {
	Matrix r;
	slice(r, *this, min_col_index, min_row_index, max_col_index, max_row_index);
	return r;
}

Matrix&& Matrix::slice(dim_t min_col_index, dim_t min_row_index,
					   dim_t max_col_index, dim_t max_row_index) && {
	slice(*this, *this, min_col_index, min_row_index, max_col_index,
		  max_row_index);
	return Move(*this);
}

void Matrix::slice(Matrix& dst, const Matrix& src, dim_t max_col_index,
				   dim_t max_row_index) {
	RHO__debug_if(src.col_dim_ < max_col_index ||
				  src.row_dim_ < max_row_index) {
		RHO__throw__local("index error");
	}

	dst.col_dim_ = max_col_index;
	dst.row_dim_ = max_row_index;

	if (dst.col_dim_ * dst.row_dim_) {
		for (dim_t i(0); i != dst.col_dim_; ++i) {
			CopyForward(dst.row_dim_, dst + RHO__max_dim * i,
						src + RHO__max_dim * i);
		}
	}
}

void Matrix::slice(Matrix& dst, const Matrix& src, dim_t min_col_index,
				   dim_t min_row_index, dim_t max_col_index,
				   dim_t max_row_index) {
	RHO__debug_if(
		src.col_dim_ < min_col_index || src.row_dim_ < min_row_index ||
		src.col_dim_ < max_col_index || src.row_dim_ < max_row_index ||
		max_col_index < min_col_index || max_row_index < min_row_index) {
		RHO__throw__local("index error");
	}

	dst.col_dim_ = max_col_index - min_col_index;
	dst.row_dim_ = max_row_index - min_row_index;

	if (dst.col_dim_ * dst.row_dim_) {
		for (dim_t i(0); i != dst.col_dim_; ++i) {
			CopyForward(dst.row_dim_, dst + RHO__max_dim * i,
						src + RHO__max_dim * (min_col_index + i) +
							min_row_index);
		}
	}
}

#///////////////////////////////////////////////////////////////////////////////

bool operator==(const Matrix& x, const Matrix& y) {
	if (x.col_dim() != y.col_dim() || x.row_dim() != y.row_dim()) {
		return false;
	}

	for (dim_t i(0); i != x.col_dim(); ++i) {
		for (dim_t j(0); j != x.row_dim(); ++j) {
			if (x[RHO__max_dim * i + j] != y[RHO__max_dim * i + j])
				return false;
		}
	}

	return true;
}

bool operator!=(const Matrix& x, const Matrix& y) { return !(x == y); }

#///////////////////////////////////////////////////////////////////////////////

Matrix& Matrix::operator+() & { return *this; }
Matrix&& Matrix::operator+() && { return Move(*this); }
const Matrix& Matrix::operator+() const& { return *this; }

Matrix Matrix::Matrix::operator-() const& {
	Matrix r(this->col_dim_, this->row_dim_);

#pragma unroll
	RHO__full_loop { r[i] = -(*this)[i]; }

	return r;
}

Matrix& Matrix::operator-() && {
#pragma unroll
	RHO__full_loop { (*this)[i] = -(*this)[i]; }

	return *this;
}

#///////////////////////////////////////////////////////////////////////////////

Matrix operator+(const Matrix& x, const Matrix& y) {
	RHO__debug_if(x.col_dim() != y.col_dim() || x.row_dim() != y.row_dim()) {
		RHO__throw__local("dim error");
	}

	Matrix r(x.col_dim(), x.row_dim());

#pragma unroll
	RHO__full_loop { r[i] = x[i] + y[i]; }

	return r;
}

Matrix&& operator+(const Matrix& x, Matrix&& y) { return Move(y += x); }

Matrix&& operator+(Matrix&& x, const Matrix& y) { return Move(x += y); }

Matrix&& operator+(Matrix&& x, Matrix&& y) { return Move(x += y); }

#///////////////////////////////////////////////////////////////////////////////

Matrix operator-(const Matrix& x, const Matrix& y) {
	RHO__debug_if(x.col_dim() != y.col_dim() || x.row_dim() != y.row_dim()) {
		RHO__throw__local("dim error");
	}

	Matrix r(x.col_dim(), x.row_dim());

#pragma unroll
	RHO__full_loop { r[i] = x[i] - y[i]; }

	return r;
}

Matrix&& operator-(const Matrix& x, Matrix&& y) {
	RHO__debug_if(x.col_dim() != y.col_dim() || x.row_dim() != y.row_dim()) {
		RHO__throw__local("dim error");
	}

#pragma unroll
	RHO__full_loop { y[i] = x[i] - y[i]; }

	return Move(y);
}

Matrix&& operator-(Matrix&& x, const Matrix& y) { return Move(x -= y); }

Matrix&& operator-(Matrix&& x, Matrix&& y) { return Move(x -= y); }

#///////////////////////////////////////////////////////////////////////////////

Matrix operator*(Num num, const Matrix& matrix) { return matrix * num; }

Matrix operator*(const Matrix& matrix, Num num) {
	Matrix r(matrix.col_dim(), matrix.row_dim());

#pragma unroll
	RHO__full_loop { r[i] = matrix[i] * num; }

	return r;
}

Vector operator*(const Vector& vector, const Matrix& matrix) {
	RHO__debug_if(vector.dim() != matrix.col_dim())
		RHO__throw__local("dim error");

	Vector r(matrix.row_dim());
	dot(matrix.col_dim(), matrix.row_dim(), r, vector, matrix);

	return r;
}

Matrix operator*(const Matrix& x, const Matrix& y) {
	RHO__debug_if(x.row_dim() != y.col_dim()) RHO__throw__local("dim error");

	Matrix r(x.col_dim(), y.row_dim());
	dot(x.col_dim(), y.col_dim(), y.row_dim(), r, x, y);

	return r;
}

Matrix&& operator*(Num num, Matrix&& matrix) { return Move(matrix *= num); }

Matrix&& operator*(Matrix&& matrix, Num num) { return Move(matrix *= num); }

#///////////////////////////////////////////////////////////////////////////////

Matrix operator/(const Matrix& matrix, Num num) { return matrix * (1 / num); }
Matrix&& operator/(Matrix&& matrix, Num num) { return Move(matrix /= num); }

#///////////////////////////////////////////////////////////////////////////////

Matrix& Matrix::operator+=(const Matrix& matrix) & {
	RHO__debug_if(this->col_dim_ != matrix.col_dim_ ||
				  this->row_dim_ != matrix.row_dim_) {
		RHO__throw__local("dim error");
	}

#pragma unroll
	RHO__full_loop { (*this)[i] += matrix[i]; }

	return *this;
}

Matrix& Matrix::operator-=(const Matrix& matrix) & {
	RHO__debug_if(this->col_dim_ != matrix.col_dim_ ||
				  this->row_dim_ != matrix.row_dim_) {
		RHO__throw__local("dim error");
	}

#pragma unroll
	RHO__full_loop { (*this)[i] -= matrix[i]; }

	return *this;
}

Matrix& Matrix::operator*=(Num num) & {
#pragma unroll
	RHO__full_loop { (*this)[i] *= num; }

	return *this;
}

Matrix& Matrix::operator*=(const Matrix& matrix) & {
	return (*this) = (*this) * matrix;
}

Matrix& Matrix::operator/=(Num num) & { return (*this) *= 1 / num; }

#///////////////////////////////////////////////////////////////////////////////

void Matrix::Print(dim_t col_dim, dim_t row_dim, const Num* data) {
	if (!(col_dim * row_dim)) {
		rho::Print() << "[ void matrix ]\n";
		return;
	}

	rho::Print() << "[";

	for (dim_t i(0); i != col_dim; ++i) {
		rho::Print() << "\n" << data[RHO__max_dim * i];

		for (dim_t j(1); j != row_dim; ++j) {
			rho::Print() << ", " << data[RHO__max_dim * i + j];
		}
	}

	rho::Print() << "\n]\n";
}

const Print& operator<<(const Print& p, const Matrix& matrix) {
	Matrix::Print(matrix.col_dim(), matrix.row_dim(), matrix);
	return p;
}

#///////////////////////////////////////////////////////////////////////////////

Matrix Matrix::transpose() const& {
	Matrix r(this->row_dim_, this->col_dim_);

	for (dim_t i(0); i != this->row_dim_; ++i) {
		for (dim_t j(0); j != this->col_dim_; ++j) {
			r.get(i, j) = this->get(j, i);
		}
	}

	return r;
}

Matrix&& Matrix::transpose() && { return Move(this->transpose_self()); }

Matrix& Matrix::transpose_self() {
	Swap(this->col_dim_, this->row_dim_);

	for (dim_t i(0); i != this->col_dim_; ++i) {
		for (dim_t j(i + 1); j != this->row_dim_; ++j) {
			Swap(this->get(i, j), this->get(j, i));
		}
	}

	return *this;
}

#////////////////////////////////////////////////

Num Matrix::det() const {
	RHO__debug_if(this->col_dim_ != this->row_dim_) {
		RHO__throw__local("dim error");
	}

	return rho::det(*this, this->col_dim_);
}

Matrix Matrix::inverse() const {
	RHO__debug_if(this->col_dim_ != this->row_dim_) {
		RHO__throw__local("dim error");
	}

	Matrix r(this->col_dim_, this->row_dim_);
	rho::inverse(this->col_dim_, r, *this);
	return r;
}

Vector Matrix::cross() const {
	RHO__debug_if(this->col_dim_ + 1 != this->row_dim_) {
		RHO__throw__local("dim error");
	}

	Vector r(this->row_dim_);
	rho::cross(r, *this, this->row_dim_);
	return r;
}

#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////
#///////////////////////////////////////////////////////////////////////////////

void identity(Num* dst, dim_t dim) {
	for (dim_t i(1); i != dim; ++i) {
		*dst = 1;
		++dst;
		for (dim_t j(0); j != dim; ++j, ++dst) { *dst = 0; }
	}

	*dst = 1;
}

#///////////////////////////////////////////////////////////////////////////////

#define RHO__index(i, j) (RHO__max_dim * i + j)
#define RHO__F(i, j) (v[i] * m[RHO__max_dim * i + j])

void dot(dim_t col_dim, dim_t row_dim, Num* dst, const Num* v, const Num* m) {
	if (col_dim == 2 && row_dim == 2) {
		dst[0] = RHO__F(0, 0) + RHO__F(1, 0);
		dst[1] = RHO__F(0, 1) + RHO__F(1, 1);
		return;
	}

	if (col_dim == 3 && row_dim == 3) {
		dst[0] = RHO__F(0, 0) + RHO__F(1, 0) + RHO__F(2, 0);
		dst[1] = RHO__F(0, 1) + RHO__F(1, 1) + RHO__F(2, 1);
		dst[2] = RHO__F(0, 2) + RHO__F(1, 2) + RHO__F(2, 2);
		return;
	}

	if (col_dim == 4 && row_dim == 4) {
		dst[0] = RHO__F(0, 0) + RHO__F(1, 0) + RHO__F(2, 0) + RHO__F(3, 0);
		dst[1] = RHO__F(0, 1) + RHO__F(1, 1) + RHO__F(2, 1) + RHO__F(3, 1);
		dst[2] = RHO__F(0, 2) + RHO__F(1, 2) + RHO__F(2, 2) + RHO__F(3, 2);
		dst[3] = RHO__F(0, 3) + RHO__F(1, 3) + RHO__F(2, 3) + RHO__F(3, 3);
		return;
	}

	for (dim_t i(0); i != row_dim; ++i) { dst[i] = RHO__F(0, i); }

	for (dim_t i(0); i != col_dim; ++i) {
		for (dim_t j(1); j != row_dim; ++j) { dst[j] += RHO__F(i, j); }
	}
}

#undef RHO__F

#define RHO__dst(i, j) dst[RHO__max_dim * i + j]
#define RHO__F(i, j, k) x[RHO__max_dim * i + k] * y[RHO__max_dim * k + j]

void dot(dim_t x_col_dim, dim_t y_col_dim, dim_t y_row_dim, Num* dst,
		 const Num* x, const Num* y) {
	if (x_col_dim == 3 && y_col_dim == 3 && y_row_dim == 3) {
		RHO__dst(0, 0) = RHO__F(0, 0, 0) + RHO__F(0, 0, 1) + RHO__F(0, 0, 2);
		RHO__dst(0, 1) = RHO__F(0, 1, 0) + RHO__F(0, 1, 1) + RHO__F(0, 1, 2);
		RHO__dst(0, 2) = RHO__F(0, 2, 0) + RHO__F(0, 2, 1) + RHO__F(0, 2, 2);
		RHO__dst(1, 0) = RHO__F(1, 0, 0) + RHO__F(1, 0, 1) + RHO__F(1, 0, 2);
		RHO__dst(1, 1) = RHO__F(1, 1, 0) + RHO__F(1, 1, 1) + RHO__F(1, 1, 2);
		RHO__dst(1, 2) = RHO__F(1, 2, 0) + RHO__F(1, 2, 1) + RHO__F(1, 2, 2);
		RHO__dst(2, 0) = RHO__F(2, 0, 0) + RHO__F(2, 0, 1) + RHO__F(2, 0, 2);
		RHO__dst(2, 1) = RHO__F(2, 1, 0) + RHO__F(2, 1, 1) + RHO__F(2, 1, 2);
		RHO__dst(2, 2) = RHO__F(2, 2, 0) + RHO__F(2, 2, 1) + RHO__F(2, 2, 2);
		return;
	}

	/*for (dim_t i(0); i != x_col_dim; ++i) {
		dot(y_col_dim, y_row_dim, dst + RHO__max_dim * i,
			x + RHO__max_dim * i, y);
	}*/

	for (dim_t i(0); i != x_col_dim; ++i) {
		for (dim_t j(0); j != y_row_dim; ++j) {
			RHO__dst(i, j) = RHO__F(i, j, 0);

			for (dim_t k(1); k != y_col_dim; ++k)
				RHO__dst(i, j) += RHO__F(i, j, k);
		}
	}
}

#undef RHO__dst
#undef RHO__F

#///////////////////////////////////////////////////////////////////////////////

#define RHO__F(i, j) src[RHO__max_dim * i + j]

Num det(const Num* src, dim_t size) {
	switch (size) {
		case 0: return 0;
		case 1: return *src;
	}

	if (size == 2) {
		return RHO__det_2(RHO__F(0, 0), RHO__F(0, 1), RHO__F(1, 0),
						  RHO__F(1, 1));
	}

	if (size == 3) {
		return RHO__det_3(RHO__F(0, 0), RHO__F(0, 1), RHO__F(0, 2),
						  RHO__F(1, 0), RHO__F(1, 1), RHO__F(1, 2),
						  RHO__F(2, 0), RHO__F(2, 1), RHO__F(2, 2));
	}

	if (size == 4) {
		return RHO__det_4(
			RHO__F(0, 0), RHO__F(0, 1), RHO__F(0, 2), RHO__F(0, 3),
			RHO__F(1, 0), RHO__F(1, 1), RHO__F(1, 2), RHO__F(1, 3),
			RHO__F(2, 0), RHO__F(2, 1), RHO__F(2, 2), RHO__F(2, 3),
			RHO__F(3, 0), RHO__F(3, 1), RHO__F(3, 2), RHO__F(3, 3));
	}

	return 0;
}

#undef RHO__F

#define RHO__src(i, j) src[RHO__max_dim * i + j]
#define RHO__dst(i, j) dst[RHO__max_dim * i + j]

void inverse(dim_t dim, Num* dst, const Num* src) {
	if (dim == 2) {
		Num det(RHO__det_2(src[0], src[1], src[2], src[3]));

		if (det.eq<0>()) {
			dst[0] = 0;
			dst[1] = 0;
			dst[2] = 0;
			dst[3] = 0;
		} else {
			dst[0] = src[3] / det;
			dst[1] = -src[1] / det;
			dst[2] = -src[2] / det;
			dst[3] = src[0] / det;
		}

		return;
	}

	if (dim == 3) {
		Num a[3];

		a[0] = RHO__det_2(RHO__src(1, 1), RHO__src(1, 2), RHO__src(2, 1),
						  RHO__src(2, 2));
		a[1] = RHO__det_2(RHO__src(2, 1), RHO__src(2, 2), RHO__src(0, 1),
						  RHO__src(0, 2));
		a[2] = RHO__det_2(RHO__src(0, 1), RHO__src(0, 2), RHO__src(1, 1),
						  RHO__src(1, 2));

		Num det(RHO__src(0, 0) * a[0] + RHO__src(1, 0) * a[1] +
				RHO__src(2, 0) * a[2]);

		if (det.eq<0>()) {
			RHO__dst(0, 0) = 0;
			RHO__dst(0, 1) = 0;
			RHO__dst(0, 2) = 0;
			RHO__dst(1, 0) = 0;
			RHO__dst(1, 1) = 0;
			RHO__dst(1, 2) = 0;
			RHO__dst(2, 0) = 0;
			RHO__dst(2, 1) = 0;
			RHO__dst(2, 2) = 0;
		} else {
			Num idet(1 / det);

			RHO__dst(0, 0) = a[0] * idet;
			RHO__dst(0, 1) = a[1] * idet;
			RHO__dst(0, 2) = a[2] * idet;
			RHO__dst(1, 0) = RHO__det_2(RHO__src(2, 0), RHO__src(2, 2),
										RHO__src(1, 0), RHO__src(1, 2)) *
							 idet;
			// src[6], src[8], src[3], src[5]) * idet;
			RHO__dst(1, 1) = RHO__det_2(RHO__src(0, 0), RHO__src(0, 2),
										RHO__src(2, 0), RHO__src(2, 2)) *
							 idet;
			// src[0], src[2], src[6], src[8]) * idet;
			RHO__dst(1, 2) = RHO__det_2(RHO__src(0, 2), RHO__src(1, 2),
										RHO__src(0, 0), RHO__src(1, 0)) *
							 idet;
			// src[2], src[5], src[0], src[3]) * idet;
			RHO__dst(2, 0) = RHO__det_2(RHO__src(1, 0), RHO__src(1, 1),
										RHO__src(2, 0), RHO__src(2, 1)) *
							 idet;
			// src[3], src[4], src[6], src[7]) * idet;
			RHO__dst(2, 1) = RHO__det_2(RHO__src(0, 1), RHO__src(2, 1),
										RHO__src(0, 0), RHO__src(2, 0)) *
							 idet;
			// src[1], src[7], src[0], src[6]) * idet;
			RHO__dst(2, 2) = RHO__det_2(RHO__src(0, 0), RHO__src(0, 1),
										RHO__src(1, 0), RHO__src(1, 1)) *
							 idet;
			// src[0], src[1], src[3], src[4]) * idet;
		}

		return;
	}

	::printf("this->col_dim_ : %d\n", int(dim));
}

#undef RHO__src
#undef RHO__dst

#define RHO__src(i, j) src[RHO__max_dim * i + j]

void cross(Num* dst, const Num* src, dim_t size) {
	if (size == 2) {
		dst[0] = src[1];
		dst[1] = -src[0];
		return;
	}

	if (size == 3) {
		dst[0] = RHO__cross_3_0(RHO__src(0, 0), RHO__src(0, 1), RHO__src(0, 2),
								RHO__src(1, 0), RHO__src(1, 1), RHO__src(1, 2));
		dst[1] = RHO__cross_3_1(RHO__src(0, 0), RHO__src(0, 1), RHO__src(0, 2),
								RHO__src(1, 0), RHO__src(1, 1), RHO__src(1, 2));
		dst[2] = RHO__cross_3_2(RHO__src(0, 0), RHO__src(0, 1), RHO__src(0, 2),
								RHO__src(1, 0), RHO__src(1, 1), RHO__src(1, 2));
		return;
	}

	if (size == 4) {
		dst[0] = RHO__cross_4_0(RHO__src(0, 0), RHO__src(0, 1), RHO__src(0, 2),
								RHO__src(0, 3), RHO__src(1, 0), RHO__src(1, 1),
								RHO__src(1, 2), RHO__src(1, 3), RHO__src(2, 0),
								RHO__src(2, 1), RHO__src(2, 2), RHO__src(2, 3));
		dst[1] = RHO__cross_4_1(RHO__src(0, 0), RHO__src(0, 1), RHO__src(0, 2),
								RHO__src(0, 3), RHO__src(1, 0), RHO__src(1, 1),
								RHO__src(1, 2), RHO__src(1, 3), RHO__src(2, 0),
								RHO__src(2, 1), RHO__src(2, 2), RHO__src(2, 3));
		dst[2] = RHO__cross_4_2(RHO__src(0, 0), RHO__src(0, 1), RHO__src(0, 2),
								RHO__src(0, 3), RHO__src(1, 0), RHO__src(1, 1),
								RHO__src(1, 2), RHO__src(1, 3), RHO__src(2, 0),
								RHO__src(2, 1), RHO__src(2, 2), RHO__src(2, 3));
		dst[3] = RHO__cross_4_3(RHO__src(0, 0), RHO__src(0, 1), RHO__src(0, 2),
								RHO__src(0, 3), RHO__src(1, 0), RHO__src(1, 1),
								RHO__src(1, 2), RHO__src(1, 3), RHO__src(2, 0),
								RHO__src(2, 1), RHO__src(2, 2), RHO__src(2, 3));
		return;
	}

	printf("cross error\n");
}

}