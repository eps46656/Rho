#include "define.cuh"
#include "TextureParallelotope.cuh"

#define RHO__throw__local(description)                                         \
	RHO__throw(TextureParallelotope, __func__, description);

namespace rho {

size_t TextureParallelotope::dim() const { return this->dim_; }

size_t TextureParallelotope::size(size_t index) const {
	RHO__debug_if(this->dim_ <= index) RHO__throw__local("index error");

	return this->size_[index].first;
}

size_t TextureParallelotope::total_size() const { return this->total_size_; }

TextureParallelotope::data_t& TextureParallelotope::data() {
	return this->data_;
}

const TextureParallelotope::data_t& TextureParallelotope::data() const {
	return this->data_;
}

#///////////////////////////////////////////////////////////////////////////////

TextureParallelotope& TextureParallelotope::set_size(size_t* size) {
	for (size_t i(0); i != this->dim_; ++i) {
		this->size_[i].second = double_t(this->size_[i].first = size[i]);
	}

	return *this;
}

TextureParallelotope& TextureParallelotope::set_size(size_t index,
													 size_t size) {
	RHO__debug_if(this->dim_ <= index) RHO__throw__local("index error");

	this->size_[index].second = double_t(this->size_[index].first = size);

	return *this;
}

#///////////////////////////////////////////////////////////////////////////////

TextureParallelotope::TextureParallelotope(size_t dim):
	dim_(dim), size_(Malloc<pair<size_t, double_t>>(dim)), data_(dim) {}

TextureParallelotope::~TextureParallelotope() { Free(this->size_); }

#///////////////////////////////////////////////////////////////////////////////

bool TextureParallelotope::Refresh() const { return true; }

#///////////////////////////////////////////////////////////////////////////////

Texture::Data TextureParallelotope::GetData(const Num* root_point,
											const Num* tod_tan) const {
	RHO__debug_if(this->dim_ != root_point.size())
		RHO__throw__local("dim error");

	size_t direct(this->dim_);
	size_t side(0);

	size_t index(0);

	size_t a;

	for (size_t i(0); i != this->dim_; ++i) {
		if (direct == this->dim_) {
			if (root_point[i].eq<-1>()) {
				direct = i;
				side = 0;
				continue;
			} else if (root_point[i].eq<1>()) {
				direct = i;
				side = 1;
				continue;
			}
		}

		if (root_point[i].ge<-1>() && root_point[i].le<1>()) {
			a = size_t(this->size_[i].second * (root_point[i] + 1) / 2);

			(index *= this->size_[i].first) +=
				a < this->size_[i].first ? a : this->size_[i].first - 1;
		} else {
			return this->data_[0][0][0];
		}
	}

	return direct == this->dim_ ? this->data_[0][0][0]
								: this->data_[direct][side][index];
}

}