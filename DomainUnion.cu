#include "define.cuh"
#include "DomainUnion.cuh"

#define RHO__throw__local(description)                                         \
	RHO__throw(DomainUnion, __func__, description);

namespace rho {

cntr::RBT<Domain*>& DomainUnion::domain() { return this->domain_; }
const cntr::RBT<Domain*>& DomainUnion::domain() const { return this->domain_; }

#///////////////////////////////////////////////////////////////////////////////

DomainUnion::DomainUnion(Space* root): DomainComplex(domain[0]->root()) {
	RHO__debug_if(!root->is_root()) RHO__throw_local("root error");
}

#///////////////////////////////////////////////////////////////////////////////

bool DomainUnion::Refresh() const {
	auto iter(this->domain_.begin());

	for (auto end(this->domain_.end()); iter != end; ++i) {
		if (this->root() != (*iter)->root() || !(*iter)->Refresh())
			return false;
	}

	return true;
}

bool DomainUnion::Contain(const Num* root_point) const {
	auto iter(this->domain_.begin());

	for (auto end(this->domain_.end()); iter != end; ++iter) {
		if ((*iter)->Contain(root_point)) { return true; }
	}

	return false;
}

#///////////////////////////////////////////////////////////////////////////////

RayCastData DomainUnion::RayCast(const Ray& ray) const {
	RayCastDataVector rcdv;
	this->RayCastDataFull(rcdv);

	if (rcdv.size()) { return rcdv[0]; }
	return RayCastData();
}

bool DomainUnion::RayCastFull(RayCastDataVector& dst, const Ray& ray) const {
	if (this->domain_.empty()) { return false; }

	if (this->domain_.size() == 1)
		return this->domain_[0].RayCastFull(dst, ray);

	cntr::Vector<RayCastDataVector> rcdvv(this->domain_.size());

	{
		auto iter(this->domain_.begin());
		size_t i(0);

		for (auto end(this->domain_.end()); iter != end; ++i) {
			bool phase((*iter)->RayCastFull(rcdvv[i]));
			if (phase && rcdvv[i].empty()) { return true; }
		}
	}

	if (this->domain_.size() == 2) {
		RayCastData__(dst, rcdvv[0], rcdvv[1]);
	} else {
		RayCastDataVector temp;
		RayCastData__(temp, rcdvv[0], rcdvv[1]);

		for (size_t i(2); i != this->domain_.size() - 1; ++i) {
			rcdvv[0] = Move(temp);
			RayCastData__(temp, rcdvv[0], rcdvv[i]);
		}

		RayCastData__(dst, temp, rcdvv.back());
	}

	return false;
}

void RayCastData__(RayCastDataVector& dst, RayCastDataVector& a,
				   RayCastDataVector& b) {
	if (a.empty()) {
		if (b.size()) { dst = Move(b); }
		return;
	}

	if (b.empty()) {
		dst = Move(a);
		return;
	}

	size_t i(0);
	size_t j(0);

	bool last_a_to(a.back()->type.to());
	bool last_b_to(b.back()->type.to());

	for (;;) {
		if (a[i] < b[j]) {
			if (!b[j]->type.fr()) { dst.Push(Move(a[i])); }
			++i;
		} else if (b[j] < a[i]) {
			if (!a[i]->type.fr()) { dst.Push(Move(b[j])); }
			++j;
		} else {
			a[i]->type.fr(a[i]->type.fr() || b[j]->type.fr());
			a[i]->type.to(a[i]->type.to() || b[j]->type.to());

			dst.Push(Move(a[i]));
			++i;
			++j;
		}

		if (i == a.size()) {
			if (!last_a_to) {
				for (; j != b.size(); ++j) { dst.Push(Move(b[j])); }
			}

			return;
		}

		if (j == b.size()) {
			if (!last_b_to) {
				for (; i != a.size(); ++i) { dst.Push(Move(a[j])); }
			}

			return;
		}
	}
}

void DomainUnion::RayCast_(RayCastTemp& rct, const Ray& ray) const {
	for (size_t i(0); i != this->domain_.size(); ++i)
		rct->rcdvv.Push(this->domain_[i]->RayCastFull(ray));

	/*

	.a..
	++b.
	F

	*/
	/*
	for (size_t a(0); a != this->domain_.size(); ++a) {
		for (size_t b(0); b != this->domain_.size(); ++b) {
			if (a == b) { continue; }

			size_t i(0);
			size_t j(0);

			for (size_t)
		}
	}*/
}

}