#include"define.cuh"
#include"DomainUnion.cuh"

#define RHO__throw__local(description) \
	RHO__throw(DomainUnion, __func__, description);

namespace rho {

cntr::Vector<Domain*>&
DomainUnion::domain() { return this->domain_; }

const cntr::Vector<Domain*>&
DomainUnion::domain()const { return this->domain_; }

#////////////////////////////////////////////////

DomainUnion* DomainUnion::add_domain(Domain* domain) {
	RHO__debug_if(std::find(
		this->domain_.begin(),
		this->domain_.end(),
		domain) != this->domain_.end()) {

		RHO__throw__local("");
	}

	this->domain_.Push(domain);
	return this;
}

DomainUnion* DomainUnion::sub_domain(Domain* domain) {
	auto iter = std::find(
		this->domain_.begin(),
		this->domain_.end(),
		domain);

	if (iter != this->domain_.end())
		this->domain_.Erase(iter);

	return this;
}

#////////////////////////////////////////////////

DomainUnion::DomainUnion(const cntr::Vector<Domain*>& domain) :
	DomainComplex(domain[0]->root()), domain_(domain) {}

DomainUnion::DomainUnion(std::initializer_list<Domain*> domain) :
	DomainComplex((*domain.begin())->root()),
	domain_(domain.begin(), domain.end()) {}

#////////////////////////////////////////////////

void DomainUnion::Refresh()const {
	auto iter(this->domain_.begin());

	for (auto end(this->domain_.end()); iter != end; ++iter)
		(*iter)->Refresh();
}

bool DomainUnion::ReadyForRendering()const {
	auto iter(this->domain_.begin());

	for (auto end(this->domain_.end()); iter != end; ++iter)
		if (!(*iter)->ReadyForRendering()) { return false; }

	return true;
}

bool DomainUnion::Contain(const Vector& root_point)const {
	auto iter(this->domain_.begin());

	for (auto end(this->domain_.end()); iter != end; ++iter)
		if ((*iter)->Contain(root_point)) { return true; }

	return false;
}

#////////////////////////////////////////////////

RayCastData DomainUnion::RayCast(const Ray& ray)const {
	RayCastData r;
	RayCastData temp;

	auto iter(this->domain_.begin());

	for (auto end(this->domain_.end()); iter != end; ++iter) {
		temp = (*iter)->RayCast(ray);
		if (temp < r) { r = Move(temp); }
	}

	return r;
}

cntr::Vector<RayCastData> DomainUnion::
RayCastFull(const Ray& ray)const {
	/*cntr::Vector<RayCastData> r;
	cntr::Vector<RayCastData> rcdv;

	auto domain_i(this->domain_.begin());

	for (auto domain_end(this->domain_.end());
		 domain_i != domain_end; ++domain_i) {

		rcdv = (*domain_i)->RayCastFull(ray);

		auto rcdv_i(rcdv.begin());

		for (auto rcdv_end(rcdv.end()); rcdv_i != rcdv_end; ++rcdv_i) {
			auto domain_j(this->domain_.begin());

			for (; domain_j != domain_i; ++domain_j)
				if ((*domain_j)->FullContain((*rcdv_i)->root_point)) {
					rcdv_i = nullptr;
					goto A;
				}

			for (++domain_j; domain_j != domain_end; ++domain_j)
				if ((*domain_j)->FullContain((*rcdv_i)->root_point)) {
					rcdv_i = nullptr;
					goto A;
				}

			r.Push(Move(*rcdv_i));
			A:;
		}
	}

	return r;*/
}

cntr::Vector<RayCastData*>
RayCastData__(
	cntr::Vector<RayCastData>& dst,
	cntr::Vector<RayCastData>& a,
	cntr::Vector<RayCastData>& b) {

	if (a.empty()) { return; }
	if (b.empty()) {
		dst = Move(a);
		return;
	}

	size_t i(0);
	size_t j(0);

	bool last_a_to(a[a.size() - 1]->type.to());
	bool last_b_to(b[b.size() - 1]->type.to());

	for (;;) {
		if (a[i] < b[j]) {
			if (!b[j]->type.fr())
				dst.Push(Move(a[i]));
			++i;
		} else if (b[j] < a[i]) {
			if (!a[i]->type.fr())
				dst.Push(Move(b[j]));
			++j;
		} else {
			a[i]->type.fr(a[i]->type.fr() || b[j]->type.fr());
			a[i]->type.to(a[i]->type.to() || b[j]->type.to());

			dst.Push(Move(a[i]));
			++i;
			++j;
		}

		if (i == a.size()) {
			if (!last_a_to) {
				for (; j != b.size(); ++j)
					dst.Push(Move(b[j]));
			}

			return;
		}

		if (j == b.size()) {
			if (!last_b_to) {
				for (; i != a.size(); ++i)
					dst.Push(Move(a[j]));
			}

			return;
		}
	}
}

DomainUnion::RayCastTemp*
DomainUnion::RayCast_(const Ray& ray)const {
	auto rct(New<RayCastTemp>(0, this->domain_.size()));

	for (size_t i(0); i != this->domain_.size(); ++i)
		rct->rcdvv.Push(this->domain_[i]->RayCastFull(ray));

	/*

	.a..
	++b.
	F

	*/
	/*
	for (size_t a(0); a != this->domain_.size(); ++a) {
		for (size_t b(0); b != this->domain_.size(); ++b) {
			if (a == b) { continue; }

			size_t i(0);
			size_t j(0);

			for (size_t)
		}
	}*/
}

}