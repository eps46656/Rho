#include "hip/hip_runtime.h"
#include "MassPoint.cuh"

namespace rho {

void Collide(Num& dst_x_vel, Num& dst_y_vel, Num x_mass, Num x_vel, Num y_mass,
			 Num y_vel, Num e) {
	dst_x_vel =
		((x_vel) * (x_mass - y_mass * e) + (y_vel) * (x_mass * (1 + e))) /
		(x_mass + y_mass);

	dst_y_vel =
		((x_vel) * (y_mass * (1 + e)) + (y_vel) * (y_mass - x_mass * e)) /
		(x_mass + y_mass);
}

void Collide(Vector& dst_x_vel, Vector& dst_y_vel, Num x_mass,
			 const Vector& x_vel, Num y_mass, const Vector& y_vel,
			 const Vector& orth, Num e) {
	RHO__debug_if(x_vel.size() != orth.size() || y_vel.size() != orth.size()) {
		RHO__throw(, __func__, "dim error");
	}

	size_t dim(orth.size());

	Num x_vel_sq(0);
	Num x_vel_dot_orth(0);

	Num y_vel_sq(0);
	Num y_vel_dot_orth(0);

	Num orth_sq(0);

	for (size_t i(0); i != dim; ++i) {
		x_vel_sq += sq(x_vel[i]);
		x_vel_dot_orth += x_vel[i] * orth[i];

		y_vel_sq += sq(y_vel[i]);
		y_vel_dot_orth += y_vel[i] * orth[i];

		orth_sq = sq(orth[i]);
	}

	Num orth_l(sqrt(orth_sq));

	Num x_orth_vel_l_;
	Num y_orth_vel_l_;

	Collide(x_orth_vel_l_, y_orth_vel_l_, x_mass, x_vel_dot_orth / orth_l,
			y_mass, y_vel_dot_orth / orth_l, e);

	Vec x_orth_vel_;
	Vec y_orth_vel_;

#pragma unroll
	for (size_t i(0); i != RHO__max_dim; ++i) {
		dst_x_vel[i] = x_vel[i] - orth[i] * (x_vel_dot_orth / orth_sq +
											 x_orth_vel_l_ / orth_l);
		dst_y_vel[i] = y_vel[i] - orth[i] * (y_vel_dot_orth / orth_sq +
											 y_orth_vel_l_ / orth_l);
	}
}

/*
void Collide(
	Vector& dst_x, Vector& dst_y,
	MassPoint& x, MassPoint& y) {

	dst_x=
}*/

void Collide(Vector& dst_x, Vector& dst_y, MassPoint& x, MassPoint& y,
			 const Vector& orth, Num e) {}

}