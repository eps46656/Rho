#include "hip/hip_runtime.h"
#include "ForceFieldCentral.cuh"

namespace rho {

Space* ForceFieldCentral::ref() const { return this->ref_; }
Num ForceFieldCentral::force() const { return this->force_; }

void ForceFieldCentral::set_ref(Space* ref) { this->ref_ = ref; }
void ForceFieldCentral::set_force(Num force) { this->force_ = force; }

#////////////////////////////////////////////////

Vector ForceFieldCentral::GetForce(const Vector& point) const {
	RHO__debug_if(this->ref_->dim_r() != point.size())
		RHO__throw(ForceFieldCentral, __func__, "dim error");

	Vector direct(this->ref_->dim_r());
	Num direct_sq(0);

	for (size_t i(0); i != this->ref_->dim_r(); ++i) {
		direct[i] = point[i] - this->ref_->root_origin_r()[i];
		direct_sq += sq(point[i] - this->ref_->root_origin_r()[i]);
	}

	return direct *=
		   this->force_ / pow(direct_sq, Num(this->ref_->dim_r()) / 2);
}

}