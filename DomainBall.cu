#include "hip/hip_runtime.h"
#include "define.cuh"
#include "DomainBall.cuh"

#define RHO__throw__local(desc) RHO__throw(DomainBall, __func__, desc)

namespace rho {

DomainBall::DomainBall(Space* parent): DomainSole(parent) {}

#///////////////////////////////////////////////////////////////////////////////

bool DomainBall::Refresh() const { return this->ref()->RefreshSelf(); }

#///////////////////////////////////////////////////////////////////////////////

bool DomainBall::Contain_s(const Num* point) const {
	return sq(this->dim_s(), point).le<1>();
}

#///////////////////////////////////////////////////////////////////////////////

bool DomainBall::RayCastB(const Ray& ray) const {
	RayCastTemp rct;

	if (!this->RayCast_(ray, rct)) { return false; }
	if (rct.t[0].ne<0>()) { return rct.t[0].lt<1>(); }
	if (rct.t[1].ne<0>()) { return rct.t[1].lt<1>(); }

	return false;
}

RayCastData DomainBall::RayCast(const Ray& ray) const {
	RayCastTemp rct;

	if (this->RayCast_(ray, rct)) {
		if (rct.t[0].ne<0>()) {
			auto rcd(New<RayCastDataCore_>());
			rcd->domain = this;
			rcd->t = rct.t[0];
			rcd->phase.set(false, rct.t[0] != rct.t[1]);
			line<RHO__max_dim>(rcd->point, rct.t[0], rct.direct, rct.origin);

			return RayCastData(rcd);
		}

		if (rct.t[0] != rct.t[1]) {
			auto rcd(New<RayCastDataCore_>());
			rcd->domain = this;
			rcd->t = rct.t[1];
			rcd->phase.set(true, false);
			line<RHO__max_dim>(rcd->point, rct.t[1], rct.direct, rct.origin);

			return RayCastData(rcd);
		}
	}

	return RayCastData();
}

void DomainBall::RayCastForRender(RayCastDataPair& rcdp,
								  ComponentCollider* cmpt_collider,
								  const Ray& ray) const {
	RayCastTemp rct;
	if (!this->RayCast_(ray, rct)) { return; }

	if (rct.t[0].ne<0>()) {
		if (rcdp[1] < rct.t[0]) { return; }

		auto rcd(New<RayCastDataCore_>());
		rcd->cmpt_collider = cmpt_collider;
		rcd->domain = this;
		rcd->t = rct.t[0];
		rcd->phase.set(false, rct.t[0] != rct.t[1]);
		line<RHO__max_dim>(rcd->point, rct.t[0], rct.direct, rct.origin);

		if (rct.t[0] < rcdp[0]) {
			rcdp[1] = Move(rcdp[0]);
			rcdp[0] = rcd;
		} else {
			rcdp[1] = rcd;
			return;
		}
	}

	if (rct.t[0] != rct.t[1] && rct.t[1] < rcdp[1]) {
		auto rcd(New<RayCastDataCore_>());
		rcd->cmpt_collider = cmpt_collider;
		rcd->domain = this;
		rcd->t = rct.t[1];
		rcd->phase.set(true, false);
		line<RHO__max_dim>(rcd->point, rct.t[1], rct.direct, rct.origin);

		if (rct.t[0] < rcdp[0]) {
			rcdp[1] = Move(rcdp[0]);
			rcdp[0] = rcd;
		} else {
			rcdp[1] = rcd;
		}
	}
}

bool DomainBall::RayCastFull(RayCastDataVector& dst, const Ray& ray) const {
	RayCastTemp rct;

	if (this->RayCast_(ray, rct)) {
		if (rct.t[0].ne<0>()) {
			auto rcd(New<RayCastDataCore_>());
			rcd->domain = this;
			rcd->t = rct.t[0];
			rcd->phase.set(false, rct.t[0] != rct.t[1]);
			line<RHO__max_dim>(rcd->point, rct.t[0], rct.direct, rct.origin);

			dst.Push(rcd);
		}

		if (rct.t[0] != rct.t[1]) {
			auto rcd(New<RayCastDataCore_>());
			rcd->domain = this;
			rcd->t = rct.t[1];
			rcd->phase.set(true, false);
			line<RHO__max_dim>(rcd->point, rct.t[1], rct.direct, rct.origin);

			dst.Push(rcd);
		}
	}

	return false;
}

bool DomainBall::RayCast_(const Ray& ray, RayCastTemp& rct) const {
	this->ref()->MapPointFromRoot_rr(rct.origin, ray.origin);
	this->ref()->MapVectorFromRoot_rr(rct.direct, ray.direct);

#///////////////////////////////////////////////////////////////////////////////

	{
		Num a(0);
		Num b(0);
		Num c(-1);

		for (size_t i(0); i != this->dim_s(); ++i) {
			a += sq(rct.direct[i]);
			b -= rct.origin[i] * rct.direct[i];
			c += sq(rct.origin[i]);
		}

		if (a.eq<0>()) {
			if (c.gt<0>()) { return false; }
		} else {
			if ((c = sq(b) - a * c).lt<0>()) { return false; }
			c = sqrt(c);

			rct.t[1] = (b + c) / a;
			if (rct.t[1].lt<0>()) { return false; }

			rct.t[0] = (b - c) / a;
			if (rct.t[0].lt<0>()) { rct.t[0] = 0; }
		}
	}

#///////////////////////////////////////////////////////////////////////////////

	for (size_t i(this->dim_s()); i != this->dim_r(); ++i) {
		if (rct.direct[i].eq<0>()) {
			if (rct.origin[i].eq<0>()) { continue; }
			return false;
		}

		Num t(-rct.origin[i] / rct.direct[i]);
		if (t < rct.t[0] || rct.t[1] < t) { return false; }
		rct.t[0] = rct.t[1] = t;
	}

	return true;
}

#///////////////////////////////////////////////////////////////////////////////

Matrix DomainBall::GetParallelVector_s(const Vector& point) const {
	RHO__debug_if(this->dim_r() != point.dim() &&
				  this->dim_s() != point.dim()) {
		RHO__throw__local("dim error");
	}

	Num a(sq(this->dim_s(), point));

	if (a.ne<1>()) {
		Matrix r(this->dim_s(), this->dim_r());
		Matrix::identity(r, this->dim_r());

		return r;
	}

	Matrix orth(1, this->dim_s());
	Copy(this->dim_s(), orth, point);

	Complement(orth);

	Matrix tan(this->dim_s() - 1, this->dim_r());
	dot(this->dim_s() - 1, this->dim_s(), this->dim_r(), tan,
		orth + this->dim_s(), this->ref()->root_axis());

	return tan;
}

#///////////////////////////////////////////////////////////////////////////////

void DomainBall::GetTodTan(Num* dst, const RayCastData& rcd,
						   const Num* root_direct) const {
	RHO__debug_if(this != rcd->domain) RHO__throw__local("domain sole error");

	Num* point(rcd.Get<RayCastDataCore_*>()->point);

	Mat m;
	Mat temp;

	if (sq(this->dim_s(), point).ne<1>()) {
		Matrix::identity(temp, this->dim_r());
		Tod::TanMatrix(this->dim_s(), this->dim_r(), m, temp);
	} else {
		Copy<RHO__max_dim>(m, point);
		Complement(1, this->dim_s(), m);

		dot(this->dim_s() - 1, this->dim_s(), this->dim_r(), temp,
			m + RHO__max_dim, this->ref()->root_axis());

		Tod::TanMatrix(this->dim_s() - 1, this->dim_r(), m, temp);
	}

	dot(this->dim_r(), this->dim_r(), dst, root_direct, m);
}

#//////////////////////////////////////////////////////////////////////////////

size_t DomainBall::Complexity() const {
	return 15 * this->dim_s() + 5 * this->dim_cr();
}

}
