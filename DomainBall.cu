#include "hip/hip_runtime.h"
#include "define.cuh"
#include "DomainBall.cuh"

#define RHO__throw__local(desc) RHO__throw(DomainBall, __func__, desc)

namespace rho {

DomainBall::DomainBall(Space* ref): DomainSole(ref) {}

#///////////////////////////////////////////////////////////////////////////////

const Domain* DomainBall::Refresh() const {
	if (!this->ref_) { return nullptr; }
	this->ref_->Refresh();
	return this;
}

#///////////////////////////////////////////////////////////////////////////////

bool DomainBall::Contain_s(const Num* point) const {
	return sq(this->dim(), point).le<1>();
}

#///////////////////////////////////////////////////////////////////////////////

size_t DomainBall::RayCastComplexity() const {
	return 15 * this->dim() + 5 * this->root_codim();
}

RayCastData DomainBall::RayCast(const Ray& ray) const {
	RayCastTemp rct;

	if (this->RayCast_(ray, rct)) {
		if (rct.t[0].ne<0>()) {
			auto rcd(New<RayCastDataCore_>());
			rcd->domain = this;
			rcd->t = rct.t[0];
			rcd->phase.set(false, rct.t[0] != rct.t[1]);
			line<RHO__max_dim>(rcd->point, rct.t[0], rct.direct, rct.origin);

			return RayCastData(rcd);
		}

		if (rct.t[0] != rct.t[1]) {
			auto rcd(New<RayCastDataCore_>());
			rcd->domain = this;
			rcd->t = rct.t[1];
			rcd->phase.set(true, false);
			line<RHO__max_dim>(rcd->point, rct.t[1], rct.direct, rct.origin);

			return RayCastData(rcd);
		}
	}

	return RayCastData();
}

bool DomainBall::RayCastB(const Ray& ray) const {
	RayCastTemp rct;

	if (!this->RayCast_(ray, rct)) { return false; }
	if (rct.t[0].ne<0>()) { return rct.t[0].lt<1>(); }
	if (rct.t[1].ne<0>()) { return rct.t[1].lt<1>(); }

	return false;
}

void DomainBall::RayCastPair(RayCastDataPair& rcdp, const Ray& ray) const {
	RayCastTemp rct;
	if (!this->RayCast_(ray, rct)) { return; }

	if (rct.t[0].ne<0>()) {
		if (rcdp[1] < rct.t[0]) { return; }

		auto rcd(New<RayCastDataCore_>());
		rcd->domain = this;
		rcd->t = rct.t[0];
		rcd->phase.set(false, rct.t[0] != rct.t[1]);
		line<RHO__max_dim>(rcd->point, rct.t[0], rct.direct, rct.origin);

		if (rct.t[0] < rcdp[0]) {
			rcdp[1] = Move(rcdp[0]);
			rcdp[0] = rcd;
		} else {
			rcdp[1] = rcd;
			return;
		}
	}

	if (rct.t[0] != rct.t[1] && rct.t[1] < rcdp[1]) {
		auto rcd(New<RayCastDataCore_>());
		rcd->domain = this;
		rcd->t = rct.t[1];
		rcd->phase.set(true, false);
		line<RHO__max_dim>(rcd->point, rct.t[1], rct.direct, rct.origin);

		if (rct.t[0] < rcdp[0]) {
			rcdp[1] = Move(rcdp[0]);
			rcdp[0] = rcd;
		} else {
			rcdp[1] = rcd;
		}
	}
}

size_t DomainBall::RayCastFull(RayCastData* dst, const Ray& ray) const {
	RayCastTemp rct;
	if (!this->RayCast_(ray, rct)) { return 0; }

	size_t size(0);

	if (rct.t[0].ne<0>()) {
		auto rcd(New<RayCastDataCore_>());
		rcd->domain = this;
		rcd->t = rct.t[0];
		rcd->phase.set(false, rct.t[0] != rct.t[1]);
		line<RHO__max_dim>(rcd->point, rct.t[0], rct.direct, rct.origin);
		dst[size] = rcd;
		++size;
	}

	if (rct.t[0] != rct.t[1]) {
		auto rcd(New<RayCastDataCore_>());
		rcd->domain = this;
		rcd->t = rct.t[1];
		rcd->phase.set(true, false);
		line<RHO__max_dim>(rcd->point, rct.t[1], rct.direct, rct.origin);
		dst[size] = rcd;
		++size;
	}

	return size;
}

bool DomainBall::RayCast_(const Ray& ray, RayCastTemp& rct) const {
	this->ref_->MapPointFromRoot_rr(rct.origin, ray.origin);
	this->ref_->MapVectorFromRoot_rr(rct.direct, ray.direct);

#///////////////////////////////////////////////////////////////////////////////

	Num a(0);
	Num b(0);
	Num c(-1);

	for (dim_t i(0); i != this->dim(); ++i) {
		a += sq(rct.direct[i]);
		b -= rct.origin[i] * rct.direct[i];
		c += sq(rct.origin[i]);
	}

	if (a.eq<0>()) {
		if (c.gt<0>()) { return false; }
	} else {
		if ((c = sq(b) - a * c).lt<0>()) { return false; }
		c = sqrt(c);

		rct.t[1] = (b + c) / a;
		if (rct.t[1].lt<0>()) { return false; }

		rct.t[0] = (b - c) / a;
		if (rct.t[0].lt<0>()) { rct.t[0] = 0; }
	}

#///////////////////////////////////////////////////////////////////////////////

	for (dim_t i(this->dim()); i != this->root_dim(); ++i) {
		if (rct.direct[i].eq<0>()) {
			if (rct.origin[i].eq<0>()) { continue; }
			return false;
		}

		Num t(-rct.origin[i] / rct.direct[i]);
		if (t < rct.t[0] || rct.t[1] < t) { return false; }
		rct.t[0] = rct.t[1] = t;
	}

	return true;
}

#///////////////////////////////////////////////////////////////////////////////

Matrix DomainBall::GetParallelVector_s(const Vector& point) const {
	RHO__debug_if(this->root_dim() != point.dim() &&
				  this->dim() != point.dim()) {
		RHO__throw__local("dim error");
	}

	Num a(sq(this->dim(), point));

	if (a.ne<1>()) {
		Matrix r(this->dim(), this->root_dim());
		Matrix::identity(r, this->root_dim());

		return r;
	}

	Matrix orth(1, this->dim());
	Copy(this->dim(), orth, point);

	Complement(orth);

	Matrix tan(this->dim() - 1, this->root_dim());
	dot(this->dim() - 1, this->dim(), this->root_dim(), tan, orth + this->dim(),
		this->ref_->root_axis());

	return tan;
}

#///////////////////////////////////////////////////////////////////////////////

void DomainBall::GetTodTan(Num* dst, const RayCastData& rcd,
						   const Num* root_direct) const {
	RHO__debug_if(this != rcd->domain) {
		RHO__throw__local("domain sole error");
	}

	Num* point(rcd.Get<RayCastDataCore_*>()->point);

	Mat m;
	Mat temp;

	if (sq(this->dim(), point).ne<1>()) {
		Matrix::identity(temp, this->root_dim());
		Tod::TanMatrix(this->dim(), this->root_dim(), m, temp);
	} else {
		Copy<RHO__max_dim>(m, point);
		Complement(1, this->dim(), m);

		dot(this->dim() - 1, this->dim(), this->root_dim(), temp,
			m + RHO__max_dim, this->ref_->root_axis());

		Tod::TanMatrix(this->dim() - 1, this->root_dim(), m, temp);
	}

	dot(this->root_dim(), this->root_dim(), dst, root_direct, m);
}

}