#include "hip/hip_runtime.h"
#include "define.cuh"
#include "DomainBall.cuh"

#define RHO__throw__local(desc) RHO__throw(DomainBall, __func__, desc)

namespace rho {

DomainBall::DomainBall(Space* parent): DomainSole(parent) {}

#///////////////////////////////////////////////////////////////////////////////

bool DomainBall::Refresh() const { return this->ref()->RefreshSelf(); }

#///////////////////////////////////////////////////////////////////////////////

bool DomainBall::Contain_s(const Num* point) const {
	return sq(this->dim(), point).le<1>();
}

#///////////////////////////////////////////////////////////////////////////////

size_t DomainBall::RayCastComplexity() const {
	return 15 * this->dim() + 5 * this->dim_cr();
}

RayCastData DomainBall::RayCast(const Ray& ray) const {
	RayCastTemp rct;

	if (this->RayCast_(ray, rct)) {
		if (rct.t[0].ne<0>()) {
			auto rcd(New<RayCastDataCore_>());
			rcd->domain = this;
			rcd->t = rct.t[0];
			rcd->phase.set(false, rct.t[0] != rct.t[1]);
			line<RHO__max_dim>(rcd->point, rct.t[0], rct.direct, rct.origin);

			return RayCastData(rcd);
		}

		if (rct.t[0] != rct.t[1]) {
			auto rcd(New<RayCastDataCore_>());
			rcd->domain = this;
			rcd->t = rct.t[1];
			rcd->phase.set(true, false);
			line<RHO__max_dim>(rcd->point, rct.t[1], rct.direct, rct.origin);

			return RayCastData(rcd);
		}
	}

	return RayCastData();
}

bool DomainBall::RayCastB(const Ray& ray) const {
	RayCastTemp rct;

	if (!this->RayCast_(ray, rct)) { return false; }
	if (rct.t[0].ne<0>()) { return rct.t[0].lt<1>(); }
	if (rct.t[1].ne<0>()) { return rct.t[1].lt<1>(); }

	return false;
}

void DomainBall::RayCastPair(RayCastDataPair& rcdp, const Ray& ray) const {
	RayCastTemp rct;
	if (!this->RayCast_(ray, rct)) { return; }

	if (rct.t[0].ne<0>()) {
		if (rcdp[1] < rct.t[0]) { return; }

		auto rcd(New<RayCastDataCore_>());
		rcd->domain = this;
		rcd->t = rct.t[0];
		rcd->phase.set(false, rct.t[0] != rct.t[1]);
		line<RHO__max_dim>(rcd->point, rct.t[0], rct.direct, rct.origin);

		if (rct.t[0] < rcdp[0]) {
			rcdp[1] = Move(rcdp[0]);
			rcdp[0] = rcd;
		} else {
			rcdp[1] = rcd;
			return;
		}
	}

	if (rct.t[0] != rct.t[1] && rct.t[1] < rcdp[1]) {
		auto rcd(New<RayCastDataCore_>());
		rcd->domain = this;
		rcd->t = rct.t[1];
		rcd->phase.set(true, false);
		line<RHO__max_dim>(rcd->point, rct.t[1], rct.direct, rct.origin);

		if (rct.t[0] < rcdp[0]) {
			rcdp[1] = Move(rcdp[0]);
			rcdp[0] = rcd;
		} else {
			rcdp[1] = rcd;
		}
	}
}

size_t DomainBall::RayCastFull(RayCastData* dst, const Ray& ray) const {
	RayCastTemp rct;
	if (!this->RayCast_(ray, rct)) { return 0; }

	size_t size(0);

	if (rct.t[0].ne<0>()) {
		auto rcd(New<RayCastDataCore_>());
		rcd->domain = this;
		rcd->t = rct.t[0];
		rcd->phase.set(false, rct.t[0] != rct.t[1]);
		line<RHO__max_dim>(rcd->point, rct.t[0], rct.direct, rct.origin);
		dst[size] = rcd;
		++size;
	}

	if (rct.t[0] != rct.t[1]) {
		auto rcd(New<RayCastDataCore_>());
		rcd->domain = this;
		rcd->t = rct.t[1];
		rcd->phase.set(true, false);
		line<RHO__max_dim>(rcd->point, rct.t[1], rct.direct, rct.origin);
		dst[size] = rcd;
		++size;
	}

	return size;
}

bool DomainBall::RayCast_(const Ray& ray, RayCastTemp& rct) const {
	this->ref()->MapPointFromRoot_rr(rct.origin, ray.origin);
	this->ref()->MapVectorFromRoot_rr(rct.direct, ray.direct);

#///////////////////////////////////////////////////////////////////////////////

	{
		Num a(0);
		Num b(0);
		Num c(-1);

		for (size_t i(0); i != this->dim(); ++i) {
			a += sq(rct.direct[i]);
			b -= rct.origin[i] * rct.direct[i];
			c += sq(rct.origin[i]);
		}

		if (a.eq<0>()) {
			if (c.gt<0>()) { return false; }
		} else {
			if ((c = sq(b) - a * c).lt<0>()) { return false; }
			c = sqrt(c);

			rct.t[1] = (b + c) / a;
			if (rct.t[1].lt<0>()) { return false; }

			rct.t[0] = (b - c) / a;
			if (rct.t[0].lt<0>()) { rct.t[0] = 0; }
		}
	}

#///////////////////////////////////////////////////////////////////////////////

	for (size_t i(this->dim()); i != this->dim_r(); ++i) {
		if (rct.direct[i].eq<0>()) {
			if (rct.origin[i].eq<0>()) { continue; }
			return false;
		}

		Num t(-rct.origin[i] / rct.direct[i]);
		if (t < rct.t[0] || rct.t[1] < t) { return false; }
		rct.t[0] = rct.t[1] = t;
	}

	return true;
}

#///////////////////////////////////////////////////////////////////////////////

Matrix DomainBall::GetParallelVector_s(const Vector& point) const {
	RHO__debug_if(this->dim_r() != point.dim() && this->dim() != point.dim()) {
		RHO__throw__local("dim error");
	}

	Num a(sq(this->dim(), point));

	if (a.ne<1>()) {
		Matrix r(this->dim(), this->dim_r());
		Matrix::identity(r, this->dim_r());

		return r;
	}

	Matrix orth(1, this->dim());
	Copy(this->dim(), orth, point);

	Complement(orth);

	Matrix tan(this->dim() - 1, this->dim_r());
	dot(this->dim() - 1, this->dim(), this->dim_r(), tan, orth + this->dim(),
		this->ref()->root_axis());

	return tan;
}

#///////////////////////////////////////////////////////////////////////////////

void DomainBall::GetTodTan(Num* dst, const RayCastData& rcd,
						   const Num* root_direct) const {
	RHO__debug_if(this != rcd->domain) RHO__throw__local("domain sole error");

	Num* point(rcd.Get<RayCastDataCore_*>()->point);

	Mat m;
	Mat temp;

	if (sq(this->dim(), point).ne<1>()) {
		Matrix::identity(temp, this->dim_r());
		Tod::TanMatrix(this->dim(), this->dim_r(), m, temp);
	} else {
		Copy<RHO__max_dim>(m, point);
		Complement(1, this->dim(), m);

		dot(this->dim() - 1, this->dim(), this->dim_r(), temp, m + RHO__max_dim,
			this->ref()->root_axis());

		Tod::TanMatrix(this->dim() - 1, this->dim_r(), m, temp);
	}

	dot(this->dim_r(), this->dim_r(), dst, root_direct, m);
}

}
