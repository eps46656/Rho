#include "hip/hip_runtime.h"
#include "define.cuh"
#include "DomainBall.cuh"

#define RHO__throw__local(desc) RHO__throw(DomainBall, __func__, desc)

namespace rho {

DomainBall::DomainBall(Space* ref): DomainSole(ref) {}

#///////////////////////////////////////////////////////////////////////////////

const Domain* DomainBall::Refresh() const {
	if (!this->ref_) { return nullptr; }
	this->ref_->Refresh();
	return this;
}

#///////////////////////////////////////////////////////////////////////////////

bool DomainBall::Contain_s(const Num* point) const {
	return sq(this->dim(), point).le<1>();
}

#///////////////////////////////////////////////////////////////////////////////

size_t DomainBall::RayCastComplexity() const {
	return 15 * this->dim() + 5 * this->root_codim();
}

bool DomainBall::RayCast(RayCastData& dst, const Ray& ray) const {
	RayCastTemp rct;

	if (this->RayCast_(ray, rct)) {
		if (rct.t[0].ne<0>()) {
			dst.domain = this;
			dst.t = rct.t[0];
			dst.phase.set(false, rct.t[0] != rct.t[1]);
			line<RHO__max_dim>(reinterpret_cast<Num*>(dst.spare), rct.t[0],
							   rct.direct, rct.origin);

			return true;
		}

		if (rct.t[0] != rct.t[1]) {
			dst.domain = this;
			dst.t = rct.t[1];
			dst.phase.set(true, false);
			line<RHO__max_dim>(reinterpret_cast<Num*>(dst.spare), rct.t[1],
							   rct.direct, rct.origin);

			return true;
		}
	}

	return false;
}

bool DomainBall::RayCastB(const Ray& ray) const {
	RayCastTemp rct;

	if (!this->RayCast_(ray, rct)) { return false; }
	if (rct.t[0].ne<0>()) { return rct.t[0].lt<1>(); }
	if (rct.t[1].ne<0>()) { return rct.t[1].lt<1>(); }

	return false;
}

void DomainBall::RayCastPair(RayCastDataPair& dst, const Ray& ray) const {
	RayCastTemp rct;
	if (!this->RayCast_(ray, rct)) { return; }

	if (rct.t[0].ne<0>()) {
		if (dst[1] <= rct.t[0]) { return; }

		if (dst[0] <= rct.t[0]) {
			dst[1].Destroy();

			dst[1].domain = this;
			dst[1].t = rct.t[0];
			dst[1].phase.set(false, rct.t[0] != rct.t[1]);
			line<RHO__max_dim>(reinterpret_cast<Num*>(dst[1].spare), rct.t[0],
							   rct.direct, rct.origin);

			return;
		}

		dst[1] = dst[0];

		dst[0].domain = this;
		dst[0].t = rct.t[0];
		dst[0].phase.set(false, rct.t[0] != rct.t[1]);
		line<RHO__max_dim>(reinterpret_cast<Num*>(dst[0].spare), rct.t[0],
						   rct.direct, rct.origin);
	}

	if (rct.t[0] == rct.t[1] || dst[1] <= rct.t[1]) { return; }

	dst[1].Destroy();

	dst[1].domain = this;
	dst[1].t = rct.t[0];
	dst[1].phase.set(true, false);
	line<RHO__max_dim>(reinterpret_cast<Num*>(dst[1].spare), rct.t[1],
					   rct.direct, rct.origin);
}

size_t DomainBall::RayCastFull(RayCastData* dst, const Ray& ray) const {
	RayCastTemp rct;
	if (!this->RayCast_(ray, rct)) { return 0; }

	size_t size(0);

	if (rct.t[0].ne<0>()) {
		dst[size].domain = this;
		dst[size].t = rct.t[0];
		dst[size].phase.set(false, rct.t[0] != rct.t[1]);
		line<RHO__max_dim>(reinterpret_cast<Num*>(dst[size].spare), rct.t[0],
						   rct.direct, rct.origin);

		++size;
	}

	if (rct.t[0] != rct.t[1]) {
		dst[size].domain = this;
		dst[size].t = rct.t[1];
		dst[size].phase.set(true, false);
		line<RHO__max_dim>(reinterpret_cast<Num*>(dst[size].spare), rct.t[1],
						   rct.direct, rct.origin);

		++size;
	}

	return size;
}

bool DomainBall::RayCast_(const Ray& ray, RayCastTemp& rct) const {
	this->ref_->MapPointFromRoot_rr(rct.origin, ray.origin);
	this->ref_->MapVectorFromRoot_rr(rct.direct, ray.direct);

#///////////////////////////////////////////////////////////////////////////////

	Num a(0);
	Num b(0);
	Num c(-1);

	for (dim_t i(0); i != this->dim(); ++i) {
		a += sq(rct.direct[i]);
		b -= rct.origin[i] * rct.direct[i];
		c += sq(rct.origin[i]);
	}

	if (a.eq<0>()) {
		if (c.gt<0>()) { return false; }
	} else {
		if ((c = sq(b) - a * c).lt<0>()) { return false; }
		c = sqrt(c);

		rct.t[1] = (b + c) / a;
		if (rct.t[1].lt<0>()) { return false; }

		rct.t[0] = (b - c) / a;
		if (rct.t[0].lt<0>()) { rct.t[0] = 0; }
	}

#///////////////////////////////////////////////////////////////////////////////

	if (this->ref_->root_codim() != 0) {
		dim_t i(this->dim());
		do {
			if (rct.direct[i].eq<0>()) {
				if (rct.origin[i].eq<0>()) { continue; }
				return false;
			}

			Num t(-rct.origin[i] / rct.direct[i]);
			if (t < rct.t[0] || rct.t[1] < t) { return false; }
			rct.t[0] = rct.t[1] = t;
		} while (++i != this->ref_->root_dim());
	}

	return true;
}

#///////////////////////////////////////////////////////////////////////////////

Matrix DomainBall::GetParallelVector_s(const Vector& point) const {
	RHO__debug_if(this->root_dim() != point.dim() &&
				  this->dim() != point.dim()) {
		RHO__throw__local("dim error");
	}

	Num a(sq(this->dim(), point));

	if (a.ne<1>()) {
		Matrix r(this->dim(), this->root_dim());
		Matrix::identity(r, this->root_dim());

		return r;
	}

	Matrix orth(1, this->dim());
	Copy(this->dim(), orth, point);

	Complement(orth);

	Matrix tan(this->dim() - 1, this->root_dim());
	dot(this->dim() - 1, this->dim(), this->root_dim(), tan, orth + this->dim(),
		this->ref_->root_axis());

	return tan;
}

#///////////////////////////////////////////////////////////////////////////////

void DomainBall::GetTodTan(Num* dst, const RayCastData& rcd,
						   const Num* root_direct) const {
	RHO__debug_if(this != rcd.domain) {
		RHO__throw__local("domain sole error");
	}

	const Num* point(reinterpret_cast<const Num*>(rcd.spare));

	Mat m;
	Mat temp;

	if (sq(this->dim(), point).ne<1>()) {
		Matrix::identity(temp, this->root_dim());
		Tod::TanMatrix(this->dim(), this->root_dim(), m, temp);
	} else {
		Copy<RHO__max_dim>(m, point);
		Complement(1, this->dim(), m);

		dot(this->dim() - 1, this->dim(), this->root_dim(), temp,
			m + RHO__max_dim, this->ref_->root_axis());

		Tod::TanMatrix(this->dim() - 1, this->root_dim(), m, temp);
	}

	dot(this->root_dim(), this->root_dim(), dst, root_direct, m);
}

}