#include "DomainParallelotopeTiling.cuh"
#include "define.cuh"

#define RHO__throw__local(desc)                                                \
	RHO__throw(DomainParallelotopeTiling, __func__, desc);

namespace rho {

DomainParallelotopeTiling::DomainParallelotopeTiling(Space* ref):
	DomainSole(ref) {}

#///////////////////////////////////////////////////////////////////////////////

bool DomainParallelotopeTiling::Refresh() const {
	if (!this->ref()->RefreshSelf()) { return false; }

	this->tod_matrix_.set_dim(this->dim_s(), this->dim_r());
	Copy<RHO__max_dim_sq>(this->tod_matrix_, this->ref()->root_axis());
	Tod::TanMatrix(this->tod_matrix_);

	return true;
}

#///////////////////////////////////////////////////////////////////////////////

bool DomainParallelotopeTiling::Contain_s(const Num* point) const {
	return true;
}

#///////////////////////////////////////////////////////////////////////////////

bool DomainParallelotopeTiling::RayCastB(const Ray& ray) const {
	Num t(this->RayCast_(ray));
	return t.ne<0>() && t.lt<1>();
}

RayCastData DomainParallelotopeTiling::RayCast(const Ray& ray) const {
	Num t(this->RayCast_(ray));
	RayCastData r;

	if (t.gt<0>()) {
		r = New<RayCastDataCore>();
		r->domain = this;
		r->t = t;
	}

	return r;
}

bool DomainParallelotopeTiling::RayCastFull(RayCastDataVector& dst,
											const Ray& ray) const {
	Num t(this->RayCast_(ray));

	if (t.eq<-1>()) { return true; }

	if (t.ne<0>()) {
		auto rcd(New<RayCastDataCore>());
		rcd->domain = this;
		rcd->t = t;

		dst.Push(rcd);
	}

	return false;
}

void DomainParallelotopeTiling::RayCastForRender(
	RayCastDataPair& rcdp, ComponentCollider* cmpt_collider,
	const Ray& ray) const {
	Num t(this->RayCast_(ray));

	if (t.gt<0>() && t < rcdp[1]) {
		auto rcd(New<RayCastDataCore>());
		rcd->cmpt_collider = cmpt_collider;
		rcd->domain = this;
		rcd->t = t;

		if (t < rcdp[0]) {
			rcdp[1] = Move(rcdp[0]);
			rcdp[0] = rcd;
		} else {
			rcdp[1] = rcd;
		}
	}
}

Num DomainParallelotopeTiling::RayCast_(const Ray& ray) const {
	NumVector origin;
	NumVector direct;

	this->ref()->MapPointFromRoot_rr(origin, ray.origin);
	this->ref()->MapVectorFromRoot_rr(direct, ray.direct);

#///////////////////////////////////////////////////////////////////////////////

	for (size_t i(this->dim_s()); i != this->dim_r(); ++i) {
		if (direct[i].eq<0>()) {
			if (origin[i].eq<0>()) { continue; }
			return 0;
		}

		Num t(-origin[i] / direct[i]);

		if (t.le<0>()) { return 0; }

		for (++i; i != this->dim_r(); ++i) {
			if (origin[i] != t * direct[i]) { return 0; }
		}

		return t;
	}

	return -1;
}

#///////////////////////////////////////////////////////////////////////////////

void DomainParallelotopeTiling::GetTodTan(Num* dst, const RayCastData& rcd,
										  const Num* root_direct) const {
	dot(this->dim_r(), this->dim_r(), dst, root_direct, this->tod_matrix_);
}

#///////////////////////////////////////////////////////////////////////////////

size_t DomainParallelotopeTiling::Complexity() const {
	return this->dim_cr() * 5;
}

} // namespace rho
