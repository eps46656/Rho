#include "hip/hip_runtime.h"
#include"define.cuh"
#include"DomainSimplex.h"

#define RHO__throw__local(description) \
	RHO__throw(DomainSimplex, __func__, description);

namespace rho {

DomainSimplex::DomainSimplex(Space* ref) :DomainSole(ref) {}

#////////////////////////////////////////////////

void DomainSimplex::Refresh()const {}

bool DomainSimplex::ReadyForRendering()const {
	ContainFlag flag(0);
	ContainFlag flag_end(1); --(flag <<= (this->dim_s() + 1));
	ContainFlag reader_end(1); reader_end <<= this->dim_s();

	this->tod_matrix_.Resize(flag_end);

	for (; flag != flag_end; ++flag) {
		Matrix m(this->dim_r(), this->dim_r());

		auto a_i(this->ref()->root_axis_rr().begin());
		auto m_i(m.begin());

		ContainFlag reader(1);

		for (; reader != reader_end; reader <<= 1, a_i += this->dim_r()) {
			if (!(flag & reader)) {
				Memcpy(sizeof(Num)*this->dim_r(), m_i, a_i);
				m_i += this->dim_r();
			}
		}

		if (flag & reader) {
			auto end(m_i - this->dim_r());

			for (m_i = m.begin(); m_i != end; m_i += this->dim_r()) {
				for (size_t i(0); i != this->dim_r(); ++i)
					m_i[i] -= m_i[i + this->dim_r()];
			}
		}

		m.resize((m_i - m.begin()) / this->dim_r(), this->dim_r());

		TodMatrixTan(m);

		this->tod_matrix_[flag] = Move(m);
	}

	return true;
}

#////////////////////////////////////////////////

bool DomainSimplex::
Contain_s(const Vector& point)const {
	RHO__debug_if(this->dim_s() != point.size() &&
				  this->dim_r() != point.size()) {

		RHO__throw__local("dim error");
	}

	Num r(Num::zero());
	auto iter(point.begin());
	auto end(point.begin() + this->dim_s());

	for (; iter != end; ++iter) {
		if (iter->lt<0>()) { return false; }
		r += *iter;
	}

	if (r.gt<1>()) { return false; }

	for (end = point.end(); iter != end; ++iter)
		if (iter->ne<0>()) { return false; }

	return true;
}

bool DomainSimplex::
EdgeContain_s(const Vector& point)const {
	RHO__debug_if(this->dim_s() != point.size() &&
				  this->dim_r() != point.size()) {

		RHO__throw__local("dim error");
	}

	if (this->dim_cr()) { return this->Contain_s(point); }

	Num r(Num::zero());
	auto iter(point.begin());
	auto end(point.begin() + this->dim_s());

	for (; iter != end; ++iter) {
		if (iter->lt<0>()) { return false; }
		if (iter->eq<0>()) { goto A; }
		r += *iter;
	}

	return r.eq<1>();

A:;

	for (; iter != end; ++iter) {
		if (iter->lt<0>()) { return false; }
		r += *iter;
	}

	return r.le<1>();
}

bool DomainSimplex::
FullContain_s(const Vector& point)const {
	RHO__debug_if(this->dim_s() != point.size() &&
				  this->dim_r() != point.size()) {

		RHO__throw__local("dim error");
	}

	if (this->dim_cr()) { return false; }

	Num r(Num::zero());
	auto iter(point.begin());

	for (auto end(point.begin() + this->dim_s());
		 iter != end; ++iter) {

		if (iter->lt<0>()) { return false; }
		r += *iter;
	}

	return r.lt<1>();
}

Domain::ContainType DomainSimplex::
GetContainType_s(const Vector& point)const {
	RHO__debug_if(this->dim_s() != point.size() &&
				  this->dim_r() != point.size()) {

		RHO__throw__local("dim error");
	}

	if (this->dim_cr())
		return this->Contain_s(point) ?
		ContainType::edge : ContainType::none;

	Num r(Num::zero());
	auto iter(point.begin());
	auto end(point.end());

	for (; iter != end; ++iter) {
		if (iter->lt<0>()) { return ContainType::none; }
		if (iter->eq<0>()) { goto A; }
		r += *iter;
	}

	if (r.lt<1>()) { return ContainType::full; }
	if (r.gt<1>()) { return ContainType::none; }
	return ContainType::edge;

A:;

	isum(r, iter, end);
	return r.gt<1>() ? ContainType::none : ContainType::edge;
}

#////////////////////////////////////////////////

DomainSimplex::ContainFlag DomainSimplex::
GetContainFlag(const Vector& point) const {
	RHO__debug_if(this->dim_s() != point.size() &&
				  this->dim_r() != point.size()) {

		RHO__throw__local("dim error");
	}

	ContainFlag r(contain_flag_header);
	ContainFlag writer(1);

	Num sum(Num::zero());
	auto iter(point.begin());
	auto end(point.begin() + this->dim_s());

	for (; iter != end; ++iter, writer <<= 1) {
		if (iter->lt<0>()) { return 0; }
		if (iter->eq<-1>() || iter->eq<1>()) { r |= writer; }
		sum += *iter;
	}

	for (end = point.end(); iter != end; ++iter)
		if (iter->ne<0>()) { return 0; }

	if (sum.gt<1>()) { return 0; }
	if (sum.eq<1>()) { r |= writer; }

	return r;
}

#////////////////////////////////////////////////

RayCastData DomainSimplex::
RayCast(const Ray& ray)const {
	auto rct(this->RayCast_(ray));

	if (!rct) { return RayCastData(); }

	if (rct->t[0].ne<0>()) {
		auto rcd(New<RayCastDataCore_>());
		rcd->domain = this;
		rcd->t = rct->t[0];
		rcd->root_point = ray.root_point(rct->t[0]);
		rcd->contain_flag = rct->contain_flag[0];

		Delete(rct); return RayCastData(rcd);
	}

	if (rct->t[1].ne<0>()) {
		auto rcd(New<RayCastDataCore_>());
		rcd->domain = this;
		rcd->t = rct->t[1];
		rcd->root_point = ray.root_point(rct->t[1]);
		rcd->contain_flag = rct->contain_flag[1];

		Delete(rct); return RayCastData(rcd);
	}

	Delete(rct); return RayCastData();
}

cntr::Vector<RayCastData> DomainSimplex::
RayCastFull(const Ray& ray)const {
	RayCastTemp* rct(this->RayCast_(ray));

	cntr::Vector<RayCastData> r;

	if (!rct) { return r; }

	if (rct->t[0].ne<0>()) {
		auto rcd(New<RayCastDataCore_>());
		rcd->domain = this;
		rcd->t = rct->t[0];
		rcd->root_point = ray.root_point(rct->t[0]);
		rcd->contain_flag = rct->contain_flag[0];

		r.Push(rcd);
	}

	if (rct->t[0] != rct->t[1]) {
		auto rcd(New<RayCastDataCore_>());
		rcd->domain = this;
		rcd->t = rct->t[1];
		rcd->root_point = ray.root_point(rct->t[1]);
		rcd->contain_flag = rct->contain_flag[1];

		r.Push(rcd);
	}

	Delete(rct); return r;
}

void DomainSimplex::RayCastForRender(
	pair<RayCastData>& rcd_p,
	ComponentCollider* cmpt_collider,
	const Ray& ray)const {

	RayCastTemp* rct(this->RayCast_(ray));

	if (!rct) { return; }

	if (rct->t[0].ne<0>()) {
		if (rcd_p.second < rct->t[0]) { return; }

		auto rcd(New<RayCastDataCore_>());
		rcd->cmpt_collider = cmpt_collider;
		rcd->domain = this;
		rcd->t = rct->t[0];
		rcd->root_point = ray.root_point(rct->t[0]);
		rcd->contain_flag = rct->contain_flag[0];

		if (rct->t[0] < rcd_p.first) {
			rcd_p.second = Move(rcd_p.first);
			rcd_p.first = rcd;
		} else {
			rcd_p.second = rcd;
			return;
		}
	}

	if (rct->t[0] != rct->t[1] && rct->t[1] < rcd_p.second) {
		auto rcd(New<RayCastDataCore_>());
		rcd->cmpt_collider = cmpt_collider;
		rcd->domain = this;
		rcd->t = rct->t[1];
		rcd->root_point = ray.root_point(rct->t[1]);
		rcd->contain_flag = rct->contain_flag[1];

		if (rct->t[1] < rcd_p.first) {
			rcd_p.second = Move(rcd_p.first);
			rcd_p.first = rcd;
		} else {
			rcd_p.second = rcd;
		}
	}

	Delete(rct);
}

#////////////////////////////////////////////////

bool DomainSimplex::IsTanVector(
	const Vector& root_point, const Vector& root_vector)const {

	RHO__debug_if(this->dim_r() != root_point.size() ||
				  this->dim_r() != root_vector.size()) {

		RHO__throw__local("dim error");
	}

	Vector point(this->dim_r());
	this->ref()->MapPointFromRoot_rr(
		point.begin(), root_point.begin());

	auto contain_flag(this->GetContainFlag(root_point));

	if (!contain_flag) { return false; }
	if (!(contain_flag & (contain_flag - 1))) { return true; }

	return root_vector == root_vector * this->tod_matrix_[contain_flag];
}

#////////////////////////////////////////////////

DomainSimplex::RayCastTemp* DomainSimplex::
RayCast_(const Ray& ray) const {
	RHO__debug_if(this->root() != ray.root())
		RHO__throw__local("root space error");

#define RHO__fail {Delete(rct); return nullptr; }

	auto rct(New<RayCastTemp>());

	rct->origin.resize(this->dim_r());
	rct->direct.resize(this->dim_r());

	auto origin_i(rct->origin.begin());
	auto direct_i(rct->direct.begin());

	this->ref()->MapPointFromRoot_rr(
		origin_i, ray.root_origin_r().begin());

	this->ref()->MapVectorFromRoot_rr(
		direct_i, ray.root_direct_r().begin());

	rct->t[0] = Num::zero();
	rct->t[1] = Num::inf();
	rct->contain_flag[0] = rct->contain_flag[1] = 0;

#////////////////////////////////////////////////

	if (this->dim_cr()) {
		origin_i += this->dim_s();
		direct_i += this->dim_s();

		for (auto end(rct->origin.end());
			 origin_i != end; ++origin_i, ++direct_i) {

			if (direct_i->eq<0>()) {
				if (origin_i->ne<0>()) { RHO__fail }
				continue;
			}

			Num a(-(*origin_i) / (*direct_i));

			if (a < rct->t[0] || rct->t[1] < a) { RHO__fail }
			rct->t[0] = rct->t[1] = a;
		}

		origin_i = rct->origin.begin();
		direct_i = rct->direct.begin();
	}

#////////////////////////////////////////////////

	Num origin_sum(Num::one());
	Num direct_sum(Num::zero());

	ContainFlag writer(1);

	for (ContainFlag end(ContainFlag(1) << this->dim_s());
		 writer != end; writer <<= 1, ++origin_i, ++direct_i) {

		if (direct_i->eq<0>()) {
			if (origin_i->lt<0>() || origin_i->gt<1>()) { RHO__fail }
			continue;
		}

		Num a(-(*origin_i) / (*direct_i));

		if (direct_i->lt<0>()) {
			if (a < rct->t[0]) { RHO__fail }
			if (a < rct->t[1]) {
				rct->t[1] = a;
				rct->contain_flag[1] = writer;
			} else if (a == rct->t[1]) {
				rct->contain_flag[1] |= writer;
			}
		} else {
			if (rct->t[1] < a) { RHO__fail }
			if (rct->t[0] < a) {
				rct->t[0] = a;
				rct->contain_flag[0] = writer;
			} else if (rct->t[0] == a) {
				rct->contain_flag[0] |= writer;
			}
		}

		origin_sum -= (*origin_i);
		direct_sum += (*direct_i);
	}

	if (direct_sum.eq<0>()) {
		if (origin_sum.lt<0>()) { RHO__fail }
		return rct;
	}

	Num a(origin_sum / direct_sum);

	if (direct_sum.lt<0>()) {
		if (rct->t[1] < a) { RHO__fail }
		if (rct->t[0] < a) {
			rct->t[0] = a;
			rct->contain_flag[0] = writer;
		} else if (rct->t[0] == a) {
			rct->contain_flag[0] |= writer;
		}
	} else {
		if (a < rct->t[0]) { RHO__fail }
		if (a < rct->t[1]) {
			rct->t[1] = a;
			rct->contain_flag[1] = writer;
		} else if (a == rct->t[1]) {
			rct->contain_flag[1] |= writer;
		}
	}

	return rct;
}

#////////////////////////////////////////////////

TodData DomainSimplex::
Tod(const RayCastData& rcd, const Vector& root_direct)const {
	TodData r;

	r.orth = root_direct -
		(r.tan = root_direct *
		 this->tod_matrix_[
			 rcd.Get<RayCastDataCore_*>()->contain_flag]);

	return r;
}

}