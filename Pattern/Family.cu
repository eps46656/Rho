#include"Family.cuh"

namespace rho {
namespace pattern {

Family* Family::child_begin() { return this->next_child; }
Family* Family::child_end() { return this; }

Family* Family::descendant_begin() { return this->next_descendant(); }
Family* Family::descendant_end() { return this->parent; }

const Family* Family::child_begin()const { return this->next_child; }
const Family* Family::child_end()const { return this; }

const Family* Family::descendant_begin()const { return this->next_descendant(); }
const Family* Family::descendant_end()const { return this->parent; }

#////////////////////////////////////////////////

Family::Family() :
	parent(nullptr),
	prev_sibling(nullptr), next_sibling(nullptr),
	prev_child(this), next_child(this) {}

Family::Family(Family* p) :
	parent(p), prev_child(this), next_child(this) {

	if (p == p->prev_child) {
		this->prev_sibling = this->next_sibling = p;
		p->prev_child = p->next_child = this;
	} else {
		this->prev_sibling = p->prev_child;
		p->prev_child->next_sibling = this;

		this->next_sibling = p;
		p->prev_child = this;
	}
}

Family::~Family() { this->Pop(); }

#////////////////////////////////////////////////

Family* Family::next_descendant() {
	if (this != this->prev_child) { return this->next_child; }

	Family* n(this);
	Family* m(this->next_sibling);

	while (m && n == m->prev_child)
		m = (n = m)->next_sibling;

	return m;
}

const Family* Family::next_descendant()const
{ return const_cast<Family*>(this)->next_descendant(); }

#////////////////////////////////////////////////

void Family::Push(Family* child) {
	for (Family* i(this); i; i = i->parent) {
		if (i == child) {
			// child is this or this's ancestor, Push failed
			return;
		}
	}

	child->Pop()->parent = this;

	if (this == this->prev_child) {
		child->prev_sibling = child->next_sibling = this;
		this->prev_child = this->next_child = child;
	} else {
		child->prev_sibling = this->prev_child;
		this->prev_child->next_sibling = child;

		child->next_sibling = this;
		this->prev_child = child;
	}
}

Family* Family::Pop() {
	if (this->parent) {
		if (this->parent == this->prev_sibling) {
			if (this->parent == this->next_sibling) {
				this->parent->prev_child = this->parent->next_child = this->parent;
			} else {
				this->parent->next_child = this->next_sibling;
				this->next_sibling->prev_sibling = this->parent;
			}
		} else if (this->parent == this->next_sibling) {
			this->prev_sibling->next_sibling = this->parent;
			this->parent->prev_child = this->prev_sibling;
		} else {
			this->prev_sibling->next_sibling = this->next_sibling;
			this->next_sibling->prev_sibling = this->prev_sibling;
		}

		this->parent = nullptr;
		this->prev_sibling = this->next_sibling = nullptr;
		this->prev_child = this->next_child = this;
	}

	return this;
}

}
}