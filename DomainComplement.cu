#include "define.cuh"
#include "DomainComplement.cuh"

namespace rho {

Domain* DomainComplement::domain() const { return this->domain_; }

void DomainComplement::domain(Domain* domain) { this->domain_ = domain; }

#///////////////////////////////////////////////////////////////////////////////

DomainComplement::DomainComplement(Domain* domain):
	DomainComplex(domain->root()), domain_(domain) {}

#///////////////////////////////////////////////////////////////////////////////

void DomainComplement::Refresh() const {}

bool DomainComplement::ReadyForRendering() const {
	return this->root() == this->domain_->root();
}

#///////////////////////////////////////////////////////////////////////////////

bool DomainComplement::Contain(const Vector& root_point) const {
	return !this->domain_->Contain(root_point);
}

#///////////////////////////////////////////////////////////////////////////////

RayCastData DomainComplement::RayCast(const Ray& ray) const {
	RayCastData rcd(this->domain_->RayCast(ray));

	if (rcd) { rcd->type.set(!rcd->type.fr(), !rcd->type.to()); }

	return rcd;
}

cntr::Vector<RayCastData> DomainComplement::RayCastFull(const Ray& ray) const {
	cntr::Vector<RayCastData> rcdv(this->domain_->RayCastFull(ray));

	for (size_t i(0); i != rcdv.size(); ++i) rcdv[i]->type.reverse();

	return rcdv;
}

void DomainComplement::RayCastForRender(pair<RayCastData>& rcdp,
										ComponentCollider* cmpt_collider,
										const Ray& ray) const {
	RayCastDataCore* a[2] = { rcdp.first, rcdp.second };

	this->domain_->RayCastForRender(rcdp, cmpt_collider, ray);

	if (a[1] == rcdp.second) { return; }

	if (a[0] == rcdp.second) {
		a[0]->type.reverse();
	} else {
		if (a[0] != rcdp.first) { a[0]->type.reverse(); }
		if (a[1] != rcdp.second) { a[1]->type.reverse(); }
	}
}

#///////////////////////////////////////////////////////////////////////////////
/*
bool DomainComplement::IsTanVector(
	const Vector& root_point, const Vector& root_vector)const {

	switch (this->domain_->GetContainType(root_point)) {
		case ContainType::none:return true;
		case ContainType::full:return false;
	}

	return this->domain_->IsTanVector(root_point, root_vector);
}*/

}