#include "define.cuh"
#include "DomainComplement.cuh"

#define RHO__throw__local(desc) RHO__throw(DomainComplement, __func__, desc)

namespace rho {

Domain* DomainComplement::domain() const { return this->domain_; }

void DomainComplement::domain(Domain* domain) { this->domain_ = domain; }

#///////////////////////////////////////////////////////////////////////////////

DomainComplement::DomainComplement(Space* root):
	DomainComplex(root), domain_(nullptr) {
	RHO__debug_if(!root->is_root()) RHO__throw__local("root error");
}

DomainComplement::DomainComplement(Domain* domain):
	DomainComplex(domain->root()), domain_(domain) {}

#///////////////////////////////////////////////////////////////////////////////

bool DomainComplement::Refresh() const {
	return this->domain_ && this->root() == this->domain_->root() &&
		   this->domain_->Refresh();
}

#///////////////////////////////////////////////////////////////////////////////

bool DomainComplement::Contain(const Num* root_point) const {
	return !this->domain_->Contain(root_point);
}

#///////////////////////////////////////////////////////////////////////////////

void DomainComplement::RayCastForRender(RayCastDataPair& rcdp,
										ComponentCollider* cmpt_collider,
										const Ray& ray) const {
	RayCastDataCore* a[2]{ rcdp[0], rcdp[1] };

	this->domain_->RayCastForRender(rcdp, cmpt_collider, ray);

	if (a[1] == rcdp[1]) { return; }

	if (a[0] == rcdp[1]) {
		a[0]->phase.reverse();
	} else {
		if (a[0] != rcdp[0]) { a[0]->phase.reverse(); }
		if (a[1] != rcdp[1]) { a[1]->phase.reverse(); }
	}
}

RayCastData DomainComplement::RayCast(const Ray& ray) const {
	RayCastData rcd(this->domain_->RayCast(ray));
	if (rcd) { rcd->phase.reverse(); }
	return rcd;
}

bool DomainComplement::RayCastFull(RayCastDataVector& rcdv,
								   const Ray& ray) const {
	size_t i(rcdv.size());
	bool phase(this->domain_->RayCastFull(rcdv, ray));
	for (; i != rcdv.size(); ++i) { rcdv[i]->phase.reverse(); }

	return !phase;
}

#///////////////////////////////////////////////////////////////////////////////

void DomainComplement::GetTodTan(Num* dst, const RayCastData& rcd,
								 const Num* root_direct) const {
	RHO__throw__local("call error");
}

#///////////////////////////////////////////////////////////////////////////////

size_t DomainComplement::Complexity() const {
	return this->domain_->Complexity();
}

}