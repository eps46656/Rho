#include "define.cuh"
#include "DomainComplement.cuh"

#define RHO__throw__local(desc) RHO__throw(DomainComplement, __func__, desc)

namespace rho {

Domain* DomainComplement::domain() const { return this->domain_; }

void DomainComplement::domain(Domain* domain) { this->domain_ = domain; }

#///////////////////////////////////////////////////////////////////////////////

DomainComplement::DomainComplement(Space* root):
	DomainComplex(root), domain_(nullptr) {
	RHO__debug_if(!root->is_root()) RHO__throw_local("root error");
}

DomainComplement::DomainComplement(Domain* domain):
	DomainComplex(domain->root()), domain_(domain) {}

#///////////////////////////////////////////////////////////////////////////////

bool DomainComplement::Refresh() const {
	return this->domain_ && this->root() == this->domain_->root() &&
		   this->domain_->Refresh();
}

#///////////////////////////////////////////////////////////////////////////////

bool DomainComplement::Contain(const Num* root_point) const {
	return !this->domain_->Contain(root_point);
}

#///////////////////////////////////////////////////////////////////////////////

RayCastData DomainComplement::RayCast(const Ray& ray) const {
	RayCastData rcd(this->domain_->RayCast(ray));

	if (rcd) { rcd->phase.reverse(); }

	return rcd;
}

bool DomainComplement::RayCastFull(RayCastDataVector& rcdv,
								   const Ray& ray) const {
	size_t i(rcdv.size());
	this->domain_->RayCastFull(rcdv, ray);
	for (; i != rcdv.size(); ++i) { rcdv[i]->phase.reverse(); }

	return rcdv;
}

void DomainComplement::RayCastForRender(pair<RayCastData>& rcdp,
										ComponentCollider* cmpt_collider,
										const Ray& ray) const {
	RayCastDataCore* a[2] = { rcdp.first, rcdp.second };

	this->domain_->RayCastForRender(rcdp, cmpt_collider, ray);

	if (a[1] == rcdp.second) { return; }

	if (a[0] == rcdp.second) {
		a[0]->phase.reverse();
	} else {
		if (a[0] != rcdp.first) { a[0]->phase.reverse(); }
		if (a[1] != rcdp.second) { a[1]->phase.reverse(); }
	}
}

#///////////////////////////////////////////////////////////////////////////////

void DomainComplement::GetTodTan(Num* dst, const RayCastData& rcd,
								 const Num* root_direct) const {
	RHO__throw__local("call error");
}

#///////////////////////////////////////////////////////////////////////////////

size_t DomainComplement::Complexity() const {
	return this->domain_->Complexity();
}

/*
bool DomainComplement::IsTanVector(
	const Vector& root_point, const Vector& root_vector)const {

	switch (this->domain_->GetContainType(root_point)) {
		case ContainType::none:return true;
		case ContainType::full:return false;
	}

	return this->domain_->IsTanVector(root_point, root_vector);
}*/

}