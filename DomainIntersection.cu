#include "define.cuh"
#include "DomainIntersection.cuh"

#define RHO__throw__local(desc) RHO__throw(DomainIntersection, __func__, desc)

namespace rho {

RBT<Domain*>& DomainIntersection::domain() { return this->domain_; }
const RBT<Domain*>& DomainIntersection::domain() const { return this->domain_; }

#///////////////////////////////////////////////////////////////////////////////

DomainIntersection::DomainIntersection(Space* root): DomainComplex(root) {}

#///////////////////////////////////////////////////////////////////////////////

bool DomainIntersection::Refresh() const {
	auto iter(this->domain_.begin());

	for (auto end(this->domain_.end()); iter != end; ++iter) {
		if (this->root() != (*iter)->root() || !(*iter)->Refresh()) {
			return false;
		}
	}

	return true;
}

#///////////////////////////////////////////////////////////////////////////////

bool DomainIntersection::Contain(const Num* root_point) const {
	if (this->domain_.empty()) { return false; }

	auto iter(this->domain_.begin());

	for (auto end(this->domain_.end()); iter != end; ++iter) {
		if (!(*iter)->Contain(root_point)) { return false; }
	}

	return true;
}

#///////////////////////////////////////////////////////////////////////////////

bool DomainIntersection::RayCastFull(RayCastDataVector& dst,
									 const Ray& ray) const {
	switch (this->domain_.size()) {
		case 0: return false;
		case 1: return (*this->domain_.begin())->RayCastFull(dst, ray);
	}

	cntr::Vector<RayCastDataVector> rcdvv;
	rcdvv.Reserve(this->domain_.size());
	rcdvv.Resize(1);

	{
		auto iter(this->domain_.begin());
		size_t size(0);

		for (auto end(this->domain_.end()); iter != end; ++iter) {
			if (size == rcdvv.size()) { rcdvv.Push(); }

			bool phase((*iter)->RayCastFull(rcdvv.back(), ray));

			if (rcdvv.back().empty()) {
				if (!phase) { return false; }
			} else {
				++size;
			}
		}

		rcdvv.Resize(size);
	}

	if (rcdvv.size() == 2) {
		RayCast_(dst, rcdvv[0], rcdvv[1]);
	} else {
		RayCastDataVector temp;
		RayCast_(temp, rcdvv[0], rcdvv[1]);

		for (size_t i(2); i != rcdvv.size() - 1; ++i) {
			rcdvv[0] = Move(temp);
			RayCast_(temp, rcdvv[0], rcdvv[i]);
		}

		RayCast_(dst, temp, rcdvv.back());
	}

	return false;
}

void DomainIntersection::RayCast_(RayCastDataVector& dst, RayCastDataVector& a,
								  RayCastDataVector& b) {
	if (a.empty()) {
		if (b.size()) { dst = Move(b); }
		return;
	}

	if (b.empty()) {
		dst = Move(a);
		return;
	}

	size_t i(0);
	size_t j(0);

	bool a_to(a.back()->phase.to());
	bool b_to(b.back()->phase.to());

	for (;;) {
		if (a[i] < b[j]) {
			if (b[j]->phase.fr()) { dst.Push(Move(a[i])); }
			++i;
		} else if (b[j] < a[i]) {
			if (a[i]->phase.fr()) { dst.Push(Move(b[j])); }
			++j;
		} else {
			bool fr(a[i]->phase.fr() && b[j]->phase.fr());
			bool to(a[i]->phase.to() && b[j]->phase.to());

			if (fr || to) {
				a[i]->phase.set(fr, to);
				dst.Push(Move(a[i]));
			}

			++i;
			++j;
		}

		if (i == a.size()) {
			if (a_to) {
				for (; j != b.size(); ++j) { dst.Push(Move(b[j])); }
			}

			return;
		}

		if (j == b.size()) {
			if (b_to) {
				for (; i != a.size(); ++i) { dst.Push(Move(a[i])); }
			}

			return;
		}
	}
}

#///////////////////////////////////////////////////////////////////////////////

void DomainIntersection::GetTodTan(Num* dst, const RayCastData& rcd,
								   const Num* root_direct) const {
	RHO__throw__local("call error");
}

}