#include "define.cuh"
#include "DomainIntersection.h"

#define RHO__throw__local(description)                                         \
	RHO__throw(DomainIntersection, __func__, description);

namespace rho {

const cntr::Vector<Domain*>& DomainIntersection::domain() const {
	return this->domain_;
}

void DomainIntersection::add_domain(Domain* domain) {
	this->domain_.Push(domain);
}

#////////////////////////////////////////////////

DomainIntersection::DomainIntersection(const cntr::Vector<Domain*>& domain):
	DomainComplex(domain[0]->root()), domain_(domain) {}

DomainIntersection::DomainIntersection(std::initializer_list<Domain*> domain):
	DomainComplex((*domain.begin())->root()),
	domain_(domain.begin(), domain.end()) {}

#///////////////////////////////////////////////////////////////////////////////

void DomainIntersection::Refresh() const {
	auto iter(this->domain_.begin());
	auto end(this->domain_.end());

	Sort(iter, end);

	for (; iter != end; ++iter) {
		RHO__debug_if(std::count(iter + 1, end, *iter))
			RHO__throw__local("domain error");

		(*iter)->Refresh();
	}
}

bool DomainIntersection::ReadyForRendering() const {
	auto iter(this->domain_.begin());
	auto end(this->domain_.end());

	Sort(iter, end);

	for (; iter != end; ++iter) {
		RHO__debug_if(rho::Contain(iter + 1, end, *iter)) return false;

		if (!(*iter)->ReadyForRendering()) { return false; }
	}

	return true;
}

#///////////////////////////////////////////////////////////////////////////////

bool DomainIntersection::Contain(const Vector& root_point) const {
	if (this->domain_.empty()) { return false; }

	auto iter(this->domain_.begin());

	for (auto end(this->domain_.end()); iter != end; ++iter)
		if (!(*iter)->Contain(root_point)) { return false; }

	return true;
}

bool DomainIntersection::EdgeContain(const Vector& root_point) const {
	if (this->domain_.empty()) { return false; }

	auto iter(this->domain_.begin());
	auto end(this->domain_.end());

	for (; iter != end; ++iter) {
		ContainType contain_type = (*iter)->GetContainType(root_point);
		if (contain_type == ContainType::none) { return false; }
		if (contain_type == ContainType::edge) { break; }
	}

	for (end = this->domain_.end(); iter != end; ++iter)
		if (!(*iter)->EdgeContain(root_point)) { return false; }

	return true;
}

bool DomainIntersection::FullContain(const Vector& root_point) const {
	if (this->domain_.empty()) { return false; }

	auto iter(this->domain_.begin());

	for (auto end(this->domain_.end()); iter != end; ++iter)
		if (!(*iter)->EdgeContain(root_point)) { return false; }

	return true;
}

Domain::ContainType DomainIntersection::GetContainType(
	const Vector& root_point) const {
	ContainType cont(ContainType::none);

	auto iter(this->domain_.begin());

	for (auto end(this->domain_.end()); iter != end; ++iter) {
		cont = (*iter)->GetContainType(root_point);
		if (cont == ContainType::none) { return ContainType::none; }

		if (cont == ContainType::edge) {
			for (++iter; iter != end; ++iter) {
				if (!(*iter)->Contain(root_point)) return ContainType::none;
			}
		}
	}

	return cont;
}

#///////////////////////////////////////////////////////////////////////////////

RayCastData DomainIntersection::RayCast(const Ray& ray) const {
	switch (this->domain_.size()) {
		case 0: return RayCastData(false);
		case 1: return this->domain_[0]->RayCast(ray);
	}

	RayCastData r;

	auto domain_i(this->domain_.begin());

	for (auto domain_end(this->domain_.end()); domain_i != domain_end;
		 ++domain_i) {
		cntr::Vector<RayCastData> rcdv((*domain_i)->RayCastFull(ray));

		if (rcdv.empty()) { continue; }

		auto rcdv_iter(rcdv.begin());
		auto rcdv_end(rcdv.end());

		Sort(rcdv_iter, rcdv_end);

		for (; rcdv_iter != rcdv_end && (*rcdv_iter) < r; ++rcdv_iter) {
			auto domain_j(this->domain_.begin());

			for (; domain_j != domain_i; ++domain_j)
				if (!(*domain_j)->Contain((*rcdv_iter)->root_point)) goto A;

			for (++domain_j; domain_j != domain_end; ++domain_j)
				if (!(*domain_j)->Contain((*rcdv_iter)->root_point)) goto A;

			r = Move(*rcdv_iter);
			break;
		A:;
		}
	}

	return r;
}

cntr::Vector<RayCastData> DomainIntersection::RayCastFull(
	const Ray& ray) const {
	switch (this->domain_.size()) {
		case 0: return {};
		case 1: return this->domain_[0]->RayCastFull(ray);
	}

	cntr::Vector<RayCastData> r;

	auto domain_i(this->domain_.begin());

	for (auto domain_end(this->domain_.end()); domain_i != domain_end;
		 ++domain_i) {
		cntr::Vector<RayCastData> rcdv((*domain_i)->RayCastFull(ray));

		if (rcdv.empty()) { continue; }

		auto rcd_i(rcdv.begin());

		for (auto rcd_end(rcdv.end()); rcd_i != rcd_end; ++rcd_i) {
			auto domain_j(this->domain_.begin());

			for (; domain_j != domain_i; ++domain_j)
				if (!(*domain_j)->Contain((*rcd_i)->root_point)) goto A;

			for (++domain_j; domain_j != domain_end; ++domain_j)
				if (!(*domain_j)->Contain((*rcd_i)->root_point)) goto A;

			r.Push(Move(*rcd_i));
		A:;
		}
	}

	return r;
}

#///////////////////////////////////////////////////////////////////////////////

bool DomainIntersection::IsTanVector(
	const Vector& root_point, const Vector& root_vector) const {
	RHO__debug_if(this->dim_r() != root_point.size() ||
				  this->dim_r() != root_vector.size()) {
		RHO__throw__local("dim error");
	}

	auto iter(this->domain_.begin());

	for (auto end(this->domain_.end()); iter != end; ++iter) {
		switch ((*iter)->GetContainType(root_point)) {
			case ContainType::none: return false;
			case ContainType::edge:
				return (*iter)->IsTanVector(root_point, root_vector);
		}
	}

	return true;
}

}