#include "DomainDifference.cuh"
#include "define.cuh"

#define RHO__throw__local(description)                                         \
	RHO__throw(DomainDefference, __func__, description);

namespace rho {

Domain* DomainDifference::domain_a() const { return this->domain_a_; }
Domain* DomainDifference::domain_b() const { return this->domain_b_; }

#///////////////////////////////////////////////////////////////////////////////

void DomainDifference::doamin_a(Domain* domain_a) {
	this->domain_a_ = domain_a;
}
void DomainDifference::doamin_b(Domain* domain_b) {
	this->domain_b_ = domain_b;
}

#///////////////////////////////////////////////////////////////////////////////

DomainDifference::DomainDifference(Domain* domain_a, Domain* domain_b):
	DomainComplex(domain_a->root()), domain_a_(domain_a), domain_b_(domain_b) {
	RHO__debug_if(domain_a->root() != domain_b->root())
		RHO__throw__local("root error");
}

#///////////////////////////////////////////////////////////////////////////////

bool DomainDifference::Refresh() const {
	return this->domain_a_->Refresh() && this->domain_b_->Refresh();
}

#///////////////////////////////////////////////////////////////////////////////

bool DomainDifference::Contain(const Num* root_point) const {
	return this->domain_a_->Contain(root_point) &&
		   !this->domain_b_->Contain(root_point);
}

#///////////////////////////////////////////////////////////////////////////////

RayCastData DomainDifference::RayCast(const Ray& ray) const {
	RayCastDataVector rcdv_b;
	bool phase_b(this->domain_b_->RayCastFull(rcdv_b, ray));

	if (rcdv_b.empty())
		return phase_b ? RayCastData() : this->domain_a_->RayCast(ray);

	RayCastDataVector rcdv_a;
	this->domain_a_->RayCastFull(rcdv_a, ray);

	if (rcdv_a.empty()) { return RayCastData(); }

	size_t i(0);
	size_t j(0);

	bool last_rcdv_b_to(rcdv_b.back()->type.to());

	while (i != rcdv_a.size()) {
		if (rcdv_a[i] < rcdv_b[j]) {
			if (!rcdv_b[j]->type.fr()) return Move(rcdv_a[i]);
			++i;
		} else if (rcdv_b[j] < rcdv_a[i]) {
			if (rcdv_a[i]->type.fr()) {
				rcdv_b[j]->type.set(!rcdv_b[j]->type.fr(), true);
				return Move(rcdv_b[j]);
			}

			++j;
		} else {
			bool fr(rcdv_a[i]->type.fr() && !rcdv_b[j]->type.fr());
			bool to(rcdv_a[i]->type.to() && rcdv_b[j]->type.to());

			if (fr || to) {
				rcdv_a[i]->type.set(fr, to);
				return Move(rcdv_a[i]);
			}

			++i;
			++j;
		}

		if (j == rcdv_b.size()) {
			if (!last_rcdv_b_to && i != rcdv_a.size()) return Move(rcdv_a[i]);

			break;
		}
	}

	return RayCastData();
}

void DomainDifference::RayCastForRender(RayCastDataPair& rcdp,
	ComponentCollider* cmpt_collider, const Ray& ray) const {
	RayCastDataVector rcdv_b;
	this->domain_b_->RayCastFull(rcdv_b, ray);

	if (rcdv_b.empty()) {
		this->domain_a_->RayCastForRender(rcdp, cmpt_collider, ray);
		return;
	}

	RayCastDataVector rcdv_a;
	this->domain_a_->RayCastFull(rcdv_a, ray);

	if (rcdv_a.empty()) { return; }

	size_t i(0);
	size_t j(0);

	bool last_rcdv_b_to(rcdv_b.back()->type.to());

	while (i != rcdv_a.size()) {
		if (rcdv_a[i] < rcdv_b[j]) {
			if (rcdp[1] <= rcdv_a[i]) { return; }

			if (!rcdv_b[j]->type.fr()) {
				if (rcdv_a[i] < rcdp[0]) {
					rcdp[1] = Move(rcdp[0]);
					rcdp[0] = Move(rcdv_a[i]);
				} else {
					rcdp[1] = Move(rcdv_a[i]);
				}
			}

			++i;
		} else if (rcdv_b[j] < rcdv_a[i]) {
			if (rcdp[1] <= rcdv_b[j]) { return; }

			if (rcdv_a[i]->type.fr()) {
				rcdv_b[j]->type.set(!rcdv_b[j]->type.fr(), true);

				if (rcdv_b[j] < rcdp[0]) {
					rcdp[1] = Move(rcdp[0]);
					rcdp[0] = Move(rcdv_b[j]);
				} else {
					rcdp[1] = Move(rcdv_b[j]);
				}
			}

			++j;
		} else {
			if (rcdp[1] <= rcdv_a[i]) { return; }

			bool fr(rcdv_a[i]->type.fr() && !rcdv_b[j]->type.fr());
			bool to(rcdv_a[i]->type.to() && rcdv_b[j]->type.to());

			if (fr || to) {
				rcdv_a[i]->type.set(fr, to);

				if (rcdv_a[i] < rcdp[0]) {
					rcdp[1] = Move(rcdp[0]);
					rcdp[0] = Move(rcdv_a[i]);
				} else {
					rcdp[1] = Move(rcdv_a[i]);
				}
			}

			++i;
			++j;
		}

		if (j == rcdv_b.size()) {
			if (!last_rcdv_b_to) {
				for (; i != rcdv_a.size(); ++i) {
					if (rcdp[1] <= rcdv_a[i]) { return; }

					if (rcdp[0] <= rcdv_a[i]) {
						rcdp[1] = Move(rcdv_a[i]);
						return;
					}

					rcdp[1] = Move(rcdp[0]);
					rcdp[0] = Move(rcdv_a[i]);
				}
			}

			return;
		}
	}

	return;
}

bool DomainDifference::RayCastFull(
	RayCastDataVector& dst, const Ray& ray) const {
	RayCastDataVector rcdv_a;
	bool phase_a(this->domain_a_->RayCastFull(rcdv_a, ray));

	if (rcdv_a.empty()) { return phase_a; }

	RayCastDataVector rcdv_b;
	this->domain_b_->RayCastFull(rcdv_b, ray);

	if (rcdv_b.empty()) {
		dst.Reserve(dst.size() + rcdv_a.size());

		for (size_t i(0); i != rcdv_a.size(); ++i) dst.Push(Move(rcdv_a[i]));

		return phase_a;
	}

	dst.MoreReserve(rcdv_a.size() + rcdv_b.size());

	size_t i(0);
	size_t j(0);

	bool rcdv_b_to(rcdv_b.back()->type.to());

	while (i != rcdv_a.size()) {
		if (rcdv_a[i] < rcdv_b[j]) {
			if (!rcdv_b[j]->type.fr()) dst.Push(Move(rcdv_a[i]));
			++i;
		} else if (rcdv_b[j] < rcdv_a[i]) {
			if (rcdv_a[i]->type.fr()) {
				rcdv_b[j]->type.set(
					!rcdv_b[j]->type.fr(), !rcdv_b[j]->type.to());
				dst.Push(Move(rcdv_b[j]));
			}

			++j;
		} else {
			/*
			+------+------+-------+--------+
			| a_fr | b_fr | !b_fr | result |
			+------+------+-------+--------+
			| 0    | 0    | 1     | 0      |
			| 0    | 1    | 0     | 0      |
			| 1    | 0    | 1     | 1      |
			| 1    | 1    | 0     | 0      |
			+------+------+-------+--------+
			*/

			bool fr(rcdv_a[i]->type.fr() && !rcdv_b[j]->type.fr());
			bool to(rcdv_a[i]->type.to() && !rcdv_b[j]->type.to());

			if (fr || to) {
				rcdv_a[i]->type.set(fr, to);
				dst.Push(Move(rcdv_a[i]));
			}

			++i;
			++j;
		}

		if (j == rcdv_b.size()) {
			if (!rcdv_b_to) {
				for (; i != rcdv_a.size(); ++i) dst.Push(Move(rcdv_a[i]));
			}

			break;
		}
	}

	return phase_a;
}

#///////////////////////////////////////////////////////////////////////////////

void DomainDifference::GetTodTan(
	Num* dst, const RayCastData& rcd, const Num* root_direct) const {
	::printf("error\n");
	assert(false);
}

#///////////////////////////////////////////////////////////////////////////////

size_t DomainDifference::Complexity() const {
	return this->domain_a_->Complexity() + this->domain_b_->Complexity();
}

}